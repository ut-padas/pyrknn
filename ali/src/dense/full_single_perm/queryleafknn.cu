#include "hip/hip_runtime.h"

#define MAX_BLOCK_SIZE 1024
#define SM_SIZE_1 1024
#define SM_SIZE_2 2048
 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "queryleafknn.h"
#include "norm.h"
#include "merge.h"




static const char *cudaGetErrorEnum(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "<unknown>";
}


#define CHECK_CUBLAS(ans) { cublasAssert((ans), __FILE__, __LINE__); }
inline void cublasAssert(hipblasStatus_t code, const char *file, int line, bool abort=true) {
   if (code != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr,"CUBLAS assert: %s %s %d\n", cudaGetErrorEnum(code), file, line);
      if (abort) exit(code);
   }
}



void query_leafknn(float *X_ref, float *X_q, int *QId, int const ppl, int const leaves, int const k, float *NDist, int *NId, int const deviceId, int const verbose, int const nq, int *glob_pointIds, int num_search_leaves, int* local_leafIds, int dim){


  float dt_dist, dt_tot, dt_norms_q, dt_norms_ref, dt_tmp, dt_mem, dt_merge;
  size_t free, total;
  
  hipMemGetInfo(&free, &total);
  if (verbose) printf(" Available Memory : %.4f MB from %.4f \n", free/1e6, total/1e6);
  
  checkCudaErrors(hipSetDevice(deviceId));
  hipEvent_t t_start, t_end, t_dist, t_norms_ref, t_norms_q, t_memalloc, t_merge;
 
  checkCudaErrors(hipEventCreate(&t_start));
  checkCudaErrors(hipEventCreate(&t_end));
  checkCudaErrors(hipEventCreate(&t_dist));
  checkCudaErrors(hipEventCreate(&t_norms_ref));
  checkCudaErrors(hipEventCreate(&t_norms_q));
  checkCudaErrors(hipEventCreate(&t_memalloc));
  checkCudaErrors(hipEventCreate(&t_merge));
 
  
  checkCudaErrors(hipEventRecord(t_start, 0));
  

  hipblasStatus_t status;
  hipblasHandle_t handle;
  status = hipblasCreate(&handle);
  float oneFloat = 1.0;
  float zeroFloat = 0.0;

  if (verbose) printf("----------------------------- start leaf queries in pyrknn -----------------------------------\n");
  
  size_t tmp_NDist_size = sizeof(float) * ppl * nq;
  size_t norm_q_size = sizeof(float) * nq;
  size_t norm_ref_size = sizeof(float) * num_search_leaves * ppl; 
  if (verbose){
  printf("==========================\n");
  printf("ppl = %d \n", ppl);
  printf("leaves = %d \n", leaves);
  printf("k = %d \n", k);
  printf("nq = %d \n", nq);
  printf("num_search_leaves = %d \n", num_search_leaves);
 
  printf("Require %.4f (GB) for tmp NDists\n", tmp_NDist_size/1e9);
  printf("Require %.4f (GB) for norm refs\n", norm_ref_size/1e9);
  printf("Require %.4f (GB) for norm queries\n", norm_q_size/1e9);
  }
  float *tmp_NDist, *Norms_ref, *Norms_q;
  int *SortInd, *StepLen, *StepStart, *tidIdMap, *tidSortDir;
  

  checkCudaErrors(hipMalloc((void **) &tmp_NDist, tmp_NDist_size));
  checkCudaErrors(hipMalloc((void **) &Norms_ref, norm_ref_size));
  checkCudaErrors(hipMalloc((void **) &Norms_q, norm_q_size));

  checkCudaErrors(hipMalloc((void **) &SortInd, sizeof(int) * SM_SIZE_1));
  checkCudaErrors(hipMalloc((void **) &StepLen, sizeof(int) * 12 * SM_SIZE_1));
  checkCudaErrors(hipMalloc((void **) &StepStart, sizeof(int) * 12* SM_SIZE_1));
  checkCudaErrors(hipMalloc((void **) &tidIdMap, sizeof(int) * 12* SM_SIZE_1));
  checkCudaErrors(hipMalloc((void **) &tidSortDir, sizeof(int) * 12* SM_SIZE_1));


  checkCudaErrors(hipMemset(tmp_NDist, 0, tmp_NDist_size));  
 
  checkCudaErrors(hipEventRecord(t_memalloc, 0));
  checkCudaErrors(hipEventSynchronize(t_memalloc));
  checkCudaErrors(hipEventElapsedTime(&dt_mem, t_start, t_memalloc));

	int size_part = ppl + k;
	int size_sort = size_part;

	while (size_sort > SM_SIZE_2) size_sort = ceil((size_sort+k)/2.0);

	float tmp = size_sort/2.0;
	int blocksize = ceil(tmp);
	float tmp_f = 2 * blocksize;
	int N_pow2 = pow(2, ceil(log2(tmp_f)));
	tmp_f = N_pow2;
	int steps = log2(tmp_f);


  
  int t_b = ppl;
  while (t_b > MAX_BLOCK_SIZE) t_b /= ceil(t_b/2.0);
 
  //int t_b = (ppl > MAX_BLOCK_SIZE) ? MAX_BLOCK_SIZE : ppl;
 
  dim3 BlockDist(t_b, 1, 1);
  dim3 GridDist(nq, 1, 1);
  
  dim3 BlockNorm_ref(1, 1, 1);
  dim3 GridNorm_ref(ppl, num_search_leaves, 1);
  
  dim3 BlockNorm_q(1, 1,1);
  dim3 GridNorm_q(nq, 1, 1);
 
  dim3 BlockMerge(blocksize, 1, 1);
  dim3 GridMerge(nq, 1, 1);
 
  ComputeNorms <<< GridNorm_q, BlockNorm_q >>> (X_q, Norms_q, dim);

  checkCudaErrors(hipDeviceSynchronize());  
  checkCudaErrors(hipEventRecord(t_norms_q, 0));
  checkCudaErrors(hipEventSynchronize(t_norms_q));
  checkCudaErrors(hipEventElapsedTime(&dt_norms_q, t_memalloc, t_norms_q));
  
  ComputeNorms_ref <<< GridNorm_ref, BlockNorm_ref >>> (X_ref, Norms_ref, local_leafIds, dim);

  checkCudaErrors(hipDeviceSynchronize()); 
  checkCudaErrors(hipEventRecord(t_norms_ref, 0));
  checkCudaErrors(hipEventSynchronize(t_norms_ref));
  checkCudaErrors(hipEventElapsedTime(&dt_norms_ref, t_norms_q, t_norms_ref));

  int num_gemms = nq;
  
  CHECK_CUBLAS( hipblasSgemmStridedBatched( handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                           1, ppl, dim, 
                                           &oneFloat, X_q, dim, dim,
                                           X_ref, dim, ppl*dim,
                                           &zeroFloat, tmp_NDist, 1, ppl, num_gemms) );
   

  checkCudaErrors(hipDeviceSynchronize());  
  checkCudaErrors(hipEventRecord(t_dist, 0));
  checkCudaErrors(hipEventSynchronize(t_dist));
  checkCudaErrors(hipEventElapsedTime(&dt_dist, t_norms_ref, t_dist));

  S_PrecompMergeNP2 <<< 1, blocksize >>> (SortInd, StepLen, StepStart, tidIdMap, tidSortDir, steps);
  checkCudaErrors(hipDeviceSynchronize());  
  
  S_MergeHorizNP2<<< GridMerge, BlockMerge >>> (NDist, NId, k, ppl, tmp_NDist, glob_pointIds, steps, QId, SortInd, StepLen, StepStart, tidIdMap, tidSortDir, local_leafIds, Norms_q, Norms_ref);
  
  checkCudaErrors(hipDeviceSynchronize());  
  checkCudaErrors(hipEventRecord(t_merge, 0));
  checkCudaErrors(hipEventSynchronize(t_merge));
  checkCudaErrors(hipEventElapsedTime(&dt_merge, t_dist, t_merge));


  checkCudaErrors(hipFree(Norms_ref));
  checkCudaErrors(hipFree(Norms_q));
  checkCudaErrors(hipFree(tmp_NDist));
  checkCudaErrors(hipFree(SortInd));
  checkCudaErrors(hipFree(StepLen));
  checkCudaErrors(hipFree(StepStart));
  checkCudaErrors(hipFree(tidIdMap));
  checkCudaErrors(hipFree(tidSortDir));

  checkCudaErrors(hipEventRecord(t_end, 0));
  checkCudaErrors(hipEventSynchronize(t_end));
  checkCudaErrors(hipEventElapsedTime(&dt_tot, t_start, t_end));

  hipMemGetInfo(&free, &total);
  if (verbose) printf(" Available Memory : %.4f MB from %.4f \n", free/1e6, total/1e6);
  if (verbose){
		printf("----------------- Timings ------------\n");
		printf(" Memory : %.4f (%.f %%) \n", dt_mem/1000, 100*dt_mem/dt_tot);
		printf(" Norms queries : %.4f (%.f %%) \n", dt_norms_q/1000, 100*dt_norms_q/dt_tot);
		printf(" Norms ref : %.4f (%.f %%) \n", dt_norms_ref/1000, 100*dt_norms_ref/dt_tot);
		printf(" Dist : %.4f (%.f %%) \n", dt_dist/1000, 100*dt_dist/dt_tot);
		printf(" Merge : %.4f (%.f %%) \n", dt_merge/1000, 100*dt_merge/dt_tot);
		printf("\n Total : %.4f \n", dt_tot/1000);
		printf("-----------------------------------\n");
  }



}
