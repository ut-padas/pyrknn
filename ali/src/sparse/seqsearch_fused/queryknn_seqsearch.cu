#include "hip/hip_runtime.h"

#define MAX_BLOCK_SIZE 1024
#define SM_SIZE_1 1024
#define SM_SIZE_2 2048
 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "queryknn_seqsearch.h"
#include "SeqBinSearch.h"
#include "Norms.h"
#include "merge.h"


void query_leafknn_seqsearch(int *R_ref, int *C_ref, float *V_ref, int *R_q,  int * C_q, float *V_q, int *QId, int const ppl, int const leaves, int const k, float *NDist, int *NId, int const deviceId, int const verbose, int const nq, int const dim, int const avgnnz, int *glob_leafIds, int num_search_leaves, int *local_leafIds){


  float dt_dist, dt_tot, dt_norms_q, dt_norms_ref, dt_tmp, dt_mem, dt_merge;
  size_t free, total;
  
  hipMemGetInfo(&free, &total);
  if (verbose) printf(" Available Memory : %.4f MB from %.4f \n", free/1e6, total/1e6);
  
  checkCudaErrors(hipSetDevice(deviceId));
  hipEvent_t t_start, t_end, t_dist, t_norms_ref, t_norms_q, t_memalloc, t_merge;
 
  checkCudaErrors(hipEventCreate(&t_start));
  checkCudaErrors(hipEventCreate(&t_end));
  checkCudaErrors(hipEventCreate(&t_dist));
  checkCudaErrors(hipEventCreate(&t_norms_ref));
  checkCudaErrors(hipEventCreate(&t_norms_q));
  checkCudaErrors(hipEventCreate(&t_memalloc));
  checkCudaErrors(hipEventCreate(&t_merge));
 
  
  checkCudaErrors(hipEventRecord(t_start, 0));
  
  if (verbose) printf("----------------------------- start leaf queries in pyrknn -----------------------------------\n");
  
  size_t tmp_NDist_size = sizeof(float) * ppl * nq;
  size_t norm_q_size = sizeof(float) * nq;
  size_t norm_ref_size = sizeof(float) * num_search_leaves * ppl; 

  printf("==========================\n");
  printf("ppl = %d \n", ppl);
  printf("leaves = %d \n", leaves);
  printf("k = %d \n", k);
  printf("nq = %d \n", nq);
  printf("dim = %d \n", dim);
  printf("avgnnz = %d \n", avgnnz);
  printf("num_search_leaves = %d \n", num_search_leaves);
 
  printf("Require %.4f (GB) for tmp NDists\n", tmp_NDist_size/1e9);
  printf("Require %.4f (GB) for norm refs\n", norm_ref_size/1e9);
  printf("Require %.4f (GB) for norm queries\n", norm_q_size/1e9);

  float *Norms_ref, *Norms_q;
  int *SortInd, *StepLen, *StepStart, *tidIdMap, *tidSortDir;
  

  checkCudaErrors(hipMalloc((void **) &Norms_ref, norm_ref_size));
  checkCudaErrors(hipMalloc((void **) &Norms_q, norm_q_size));

  checkCudaErrors(hipMalloc((void **) &SortInd, sizeof(int) * SM_SIZE_1));
  checkCudaErrors(hipMalloc((void **) &StepLen, sizeof(int) * 12 * SM_SIZE_1));
  checkCudaErrors(hipMalloc((void **) &StepStart, sizeof(int) * 12* SM_SIZE_1));
  checkCudaErrors(hipMalloc((void **) &tidIdMap, sizeof(int) * 12* SM_SIZE_1));
  checkCudaErrors(hipMalloc((void **) &tidSortDir, sizeof(int) * 12* SM_SIZE_1));


  checkCudaErrors(hipEventRecord(t_memalloc, 0));
  checkCudaErrors(hipEventSynchronize(t_memalloc));
  checkCudaErrors(hipEventElapsedTime(&dt_mem, t_start, t_memalloc));

  
  int t_b = ppl;
  while (t_b > MAX_BLOCK_SIZE) t_b = ceil(t_b/2.0);

  //int t_b = (ppl > MAX_BLOCK_SIZE) ? MAX_BLOCK_SIZE : ppl;
  

  int size_sort = t_b + k;
	float tmp = size_sort/2.0;
	int blocksize = ceil(tmp);
	float tmp_f = 2 * blocksize;
	int N_pow2 = pow(2, ceil(log2(tmp_f)));
	tmp_f = N_pow2;
	int steps = log2(tmp_f);
  

 
  dim3 BlockDist(t_b, 1, 1);
  dim3 GridDist(nq, 1, 1);
  
  dim3 BlockNorm_ref(1, 1, 1);
  dim3 GridNorm_ref(ppl, num_search_leaves, 1);
  
  dim3 BlockNorm_q(1, 1,1);
  dim3 GridNorm_q(nq, 1, 1);
 
 
  ComputeNorms <<< GridNorm_q, BlockNorm_q >>> (R_q, C_q, V_q, Norms_q);

  checkCudaErrors(hipDeviceSynchronize());  
  checkCudaErrors(hipEventRecord(t_norms_q, 0));
  checkCudaErrors(hipEventSynchronize(t_norms_q));
  checkCudaErrors(hipEventElapsedTime(&dt_norms_q, t_memalloc, t_norms_q));
  
  ComputeNorms_ref <<< GridNorm_ref, BlockNorm_ref >>> (R_ref, C_ref, V_ref, Norms_ref, local_leafIds);

  checkCudaErrors(hipDeviceSynchronize()); 
  checkCudaErrors(hipEventRecord(t_norms_ref, 0));
  checkCudaErrors(hipEventSynchronize(t_norms_ref));
  checkCudaErrors(hipEventElapsedTime(&dt_norms_ref, t_norms_q, t_norms_ref));

  S_PrecompMergeNP2 <<< 1, blocksize >>> (SortInd, StepLen, StepStart, tidIdMap, tidSortDir, steps);
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(t_merge, 0));
  checkCudaErrors(hipEventSynchronize(t_merge));
  checkCudaErrors(hipEventElapsedTime(&dt_merge, t_norms_ref, t_merge));
 
  FusedLeafKNN_seq <<< GridDist, BlockDist >>> (R_ref, C_ref, V_ref, R_q, C_q, V_q, local_leafIds, Norms_q, Norms_ref, k, ppl, dim, QId, NDist, NId, glob_leafIds, steps, SortInd, StepLen, StepStart, tidIdMap, tidSortDir);

  checkCudaErrors(hipDeviceSynchronize());  
  checkCudaErrors(hipEventRecord(t_dist, 0));
  checkCudaErrors(hipEventSynchronize(t_dist));
  checkCudaErrors(hipEventElapsedTime(&dt_dist, t_merge, t_dist));


  checkCudaErrors(hipFree(Norms_ref));
  checkCudaErrors(hipFree(Norms_q));
  checkCudaErrors(hipFree(SortInd));
  checkCudaErrors(hipFree(StepLen));
  checkCudaErrors(hipFree(StepStart));
  checkCudaErrors(hipFree(tidIdMap));
  checkCudaErrors(hipFree(tidSortDir));

  checkCudaErrors(hipEventRecord(t_end, 0));
  checkCudaErrors(hipEventSynchronize(t_end));
  checkCudaErrors(hipEventElapsedTime(&dt_tot, t_start, t_end));

  hipMemGetInfo(&free, &total);
  if (verbose) printf(" Available Memory : %.4f MB from %.4f \n", free/1e6, total/1e6);
  if (verbose){
		printf("----------------- Timings ------------\n");
		printf(" Memory : %.4f (%.f %%) \n", dt_mem/1000, 100*dt_mem/dt_tot);
		printf(" Norms queries : %.4f (%.f %%) \n", dt_norms_q/1000, 100*dt_norms_q/dt_tot);
		printf(" Norms ref : %.4f (%.f %%) \n", dt_norms_ref/1000, 100*dt_norms_ref/dt_tot);
		printf(" Dist : %.4f (%.f %%) \n", dt_dist/1000, 100*dt_dist/dt_tot);
		printf(" Merge : %.4f (%.f %%) \n", dt_merge/1000, 100*dt_merge/dt_tot);
		printf("\n Total : %.4f \n", dt_tot/1000);
		printf("-----------------------------------\n");
  }



}
