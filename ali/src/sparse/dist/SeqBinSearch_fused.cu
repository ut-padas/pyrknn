#include "hip/hip_runtime.h"
#define SM_SIZE_11 2048
#define SM_SIZE_12 4096

#include "SeqBinSearch.h"


__device__ int BinSearch(int* arr, int elem, int start, int stop){

  int ret = start;
  int testInd;

  for (int l = stop - start; l > 1; l -= floorf(l/2.0)){

    testInd = (ret + l < stop - 1) ? ret + l : stop -1;
    ret = (arr[testInd] <= elem) ? testInd : ret;

  }
  
  testInd = (ret + 1 < stop - 1) ? ret + 1 : stop - 1;
  ret = (arr[testInd] <= elem) ? testInd : ret;
  
  return ret;
}


__global__ void ComputeDists_seq(int* R_ref, int* C_ref, float* V_ref, int* R_q, int* C_q, float* V_q, int* leafIds, float* Norms_q, float* Norms_ref, int const k_nn, float* KNN_tmp, int const ppl, int const d, int* QId){
  

  __shared__ int SM_C_q [SM_SIZE_11];

  int tid = threadIdx.x;

  int q = blockIdx.x;

  int ind0_q = R_q[q];
  int ind1_q = R_q[q+1];
  int nnz_q = ind1_q - ind0_q;
  float norm_q = Norms_q[q];
   

  for (int n = tid; n < nnz_q; n += blockDim.x) SM_C_q[n] = C_q[ind0_q + n];
  __syncthreads(); 


  int leafId = leafIds[q];
  int nq = gridDim.x;
  float c_tmp = 0.0; 
  
  for (int pt = tid; pt < ppl; pt += blockDim.x){
    
    c_tmp = 0.0;
    int ptId = leafId * ppl + pt;
    
    int ind0_pt = R_ref[ptId];
    int nnz_pt = R_ref[ptId+1] - ind0_pt;
    int ret = 0;
  
    for (int pos_k = 0; pos_k < nnz_pt; pos_k++){
      int k = C_ref[ind0_pt + pos_k];
      
      ret = BinSearch(SM_C_q, k, ret, nnz_q);
      int ind_jk = (SM_C_q[ret] == k) ? ret : -1;

      c_tmp += (ind_jk != -1) ? V_ref[ind0_pt + pos_k] * V_q[ind0_q + ret] : 0.0;
    }
    c_tmp = -2 * c_tmp + norm_q + Norms_ref[ptId];
    if (c_tmp < 1e-8) c_tmp = 0.0;
    int write_ind = q * ppl + pt; 
    KNN_tmp[write_ind] = c_tmp;
    //if (q == 0 && pt < 100) printf("D[%d] = %.4f , write at %d \n", pt, c_tmp, write_ind); 
     
  }
  



}


