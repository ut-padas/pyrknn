#include <iostream>
#include <stdio.h>
#include <cublasXt.h>
#include <hiprand.h>

#include "timer.hpp"

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)

void createRandomMatrix(hiprandGenerator_t &gen, int n, int d, void **matrix_gpu)
{
    CUDA_CALL(hipMalloc(matrix_gpu, n*d*sizeof(float)));
    CURAND_CALL(hiprandGenerateUniform(gen,(float *) *matrix_gpu,n*d));
    
}

int main()
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hiprandGenerator_t gen;
    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));

    int d = 100;
    int n;
    float *matrix, *matrix_gpu, *vector_gpu;
    float *vector = (float *) malloc(d*sizeof(float));
    createRandomMatrix(gen,d,1,(void **) &vector_gpu);
    CUDA_CALL(hipblasSetVector(d,sizeof(float),vector,1,vector_gpu,1));

    Timer timer = Timer();
   
    float alpha = 1.;
    float beta = 1.;

    for (int i = 1; i < 10; i++)
    {
        n = i * 1e5;
        matrix = (float *) calloc(n*d, sizeof(float));
        createRandomMatrix(gen,n,d,(void **) &matrix_gpu);
        
        CUDA_CALL(hipblasSetMatrix(n,d,sizeof(float),matrix,n,matrix_gpu,n));
        
        float *result = (float *) calloc(n,sizeof(float));
        float *result_gpu;
        CUDA_CALL(hipMalloc((void**) &result_gpu, n*sizeof(float)));
        CUDA_CALL(hipblasSetVector(n,sizeof(float),result,1,result_gpu,1));

        timer.start();
        CUDA_CALL(hipblasSgemv(handle, HIPBLAS_OP_N,
            n, d,
            &alpha,
            matrix_gpu, n,
            vector_gpu, 1,
            &beta,
            result_gpu, 1));
        hipDeviceSynchronize();
        timer.stop();
        timer.show_elapsed_time();

        CUDA_CALL(hipblasGetVector(n,sizeof(float),result_gpu,1,result,1));

        CUDA_CALL(hipFree(matrix_gpu));
        CUDA_CALL(hipFree(result_gpu));
        free(matrix);
        free(result);
    }
    CUDA_CALL(hipblasDestroy(handle));
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(vector_gpu));
    free(vector);
    return 0;
}