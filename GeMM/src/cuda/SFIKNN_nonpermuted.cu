#include "hip/hip_runtime.h"

#define SM_SIZE_1 1024
#define SM_SIZE_2 2048
#define SM_SIZE_SORT 8192

#include "FIKNN_sparse.h"

__global__ void ComputeNorms(int* R, int* C, float* V, int* G_Id, float* Norms, int ppl) {

  //int row = threadIdx.x + blockIdx.x * blockDim.x;
  int ind = threadIdx.x;
  int leafId_g = blockIdx.z * blockDim.y + blockIdx.y;
  for (int row = ind; row < ppl; row += blockDim.x){
    int g_rowId = leafId_g * ppl + row;
    //changed
  
    int g_Id = G_Id[g_rowId];

   
    int ind0_i = R[g_Id];
 
    int nnz = R[g_Id + 1] - ind0_i;
    float norm_i = 0.0;
   
    for (int n_i = 0; n_i < nnz; n_i += 1) {
      norm_i += V[ind0_i + n_i] * V[ind0_i + n_i];
    }
    Norms[g_Id] = norm_i;
  }
}

__global__ void ComputeTriDists(int* R, int* C, float* V, int* G_Id, float* Norms , int k_nn, float* KNN_dist_tmp, int ppl, int bl, int sizebleaves, int partsize) {

  int ind = threadIdx.x;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;
  int block = blockIdx.x;


  //int size_block = k_nn * (k_nn + 1) /2;
  int size_block = partsize * (partsize + 1) /2;
  
  for (int elem = ind; elem < size_block; elem += blockDim.x){

    //float tmp = -8 * elem + 4 * k_nn * (k_nn+1) - 7;
    float tmp = -8 * elem + 4 * partsize * (partsize + 1) - 7;
    int rowId = sqrt(tmp)/2.0 - 0.5;
    //rowId = k_nn - 1 - rowId;
    rowId = partsize - 1 - rowId;
    //int colId = elem + rowId - k_nn * (k_nn + 1) / 2 + (k_nn - rowId) * ((k_nn - rowId) + 1)/2;
    int colId = elem + rowId - partsize * (partsize + 1) / 2 + (partsize - rowId) * ((partsize - rowId) + 1)/2;

    float c_tmp = 0.0;

    //int g_rowId = leafId_g * ppl + block * k_nn + rowId;
    //int g_colId = leafId_g * ppl + block * k_nn + colId;
    int g_rowId = leafId_g * ppl + block * partsize + rowId;
    int g_colId = leafId_g * ppl + block * partsize + colId;
   
    //changed 
    int perm_i = G_Id[g_rowId];
    int perm_j = G_Id[g_colId];

    int ind0_i = R[perm_i];
    int ind1_i = R[perm_i + 1];

    int ind0_j = R[perm_j];
    int ind1_j = R[perm_j + 1];

    int nnz_i = ind1_i - ind0_i;
    int nnz_j = ind1_j - ind0_j;

    float norm_ij = Norms[perm_i] + Norms[perm_j];

    int tmp_0, tmp_1, ind_jk, k, ret, testInd;

    ret = 0;
    testInd = 0;


    if (nnz_i > 0 && nnz_j >0){
      for (int pos_k = 0; pos_k < nnz_j; pos_k++){
    
        k = C[ind0_j + pos_k];
        
        // Binary search
        for (int l = nnz_i - ret; l > 1; l -= floorf(l/2.0)){
          tmp_0 = ret + l;
          tmp_1 = nnz_i - 1;
          testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
          ret = (C[ind0_i + testInd] <= k) ? testInd : ret;
        }
      
        tmp_0 = ret + 1;
        tmp_1 = nnz_i - 1;
        testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;
      
        ret = (C[testInd + ind0_i] <= k) ? testInd : ret;
      
        ind_jk = (C[ret + ind0_i] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;
      
      }
    }
    c_tmp = -2 * c_tmp + norm_ij;
    c_tmp = (c_tmp > 2e-6) ? sqrt(c_tmp) : 0.0;
    
    // changed 
    //int gid_pt = leafId_g * ppl + block * k_nn + rowId;
    //int gid_pt_T = leafId_g * ppl + block * k_nn + colId;
    //int gid_pt = leafId_local * ppl + block * k_nn + rowId;
    //int gid_pt_T = leafId_local * ppl + block * k_nn + colId;
    //int ind_knn = gid_pt * k_nn + colId;
    //int ind_knn_T = gid_pt_T * k_nn + rowId;
    int gid_pt = leafId_local * ppl + block * partsize + rowId;
    int gid_pt_T = leafId_local * ppl + block * partsize + colId;
    int ind_knn = gid_pt * partsize + colId;
    int ind_knn_T = gid_pt_T * partsize + rowId;
    KNN_dist_tmp[ind_knn] = c_tmp;
    if (colId > rowId) KNN_dist_tmp[ind_knn_T] = c_tmp;
    
  }
  
}



__global__ void ComputeTriDists_last(int* R, int* C, float* V, int* G_Id, float* Norms , int k_nn, float* KNN_dist_tmp, int ppl, int rem_len , int blockId, int bl, int sizebleaves, int partsize) {




  int ind = threadIdx.x;
  //int leaf_id_g = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;
  int block = blockId;
  


  int size_block = rem_len * (rem_len + 1) /2;


  for (int elem = ind; elem < size_block; elem += blockDim.x){

    float tmp = -8 * elem + 4 * rem_len * (rem_len+1) - 7;
    int rowId = sqrt(tmp)/2.0 - 0.5;
    rowId = rem_len - 1 - rowId;
    int colId = elem + rowId - rem_len * (rem_len + 1) / 2 + (rem_len - rowId) * ((rem_len - rowId) + 1)/2;

    float c_tmp = 0.0;
    //if (block * k_nn + rowId < ppl && block * k_nn + colId < ppl){
    if (block * partsize + rowId < ppl && block * partsize + colId < ppl){

    //int g_rowId = leaf_id_g * ppl + block * k_nn + rowId;
    //int g_colId = leaf_id_g * ppl + block * k_nn + colId;
    //int g_rowId = leafId_g * ppl + block * k_nn + rowId;
    //int g_colId = leafId_g * ppl + block * k_nn + colId;
    int g_rowId = leafId_g * ppl + block * partsize + rowId;
    int g_colId = leafId_g * ppl + block * partsize + colId;

    //changed
    int perm_i = G_Id[g_rowId];
    int perm_j = G_Id[g_colId];

    int ind0_i = R[perm_i];
    int ind1_i = R[perm_i + 1];

    int ind0_j = R[perm_j];
    int ind1_j = R[perm_j + 1];

    int nnz_i = ind1_i - ind0_i;
    int nnz_j = ind1_j - ind0_j;

    float norm_ij = Norms[perm_i] + Norms[perm_j];

    int tmp_0, tmp_1, ind_jk, k, ret, testInd;

    ret = 0;
    testInd = 0;


    if (nnz_i > 0 && nnz_j >0){
      for (int pos_k = 0; pos_k < nnz_j; pos_k++){

        k = C[ind0_j + pos_k];

        // Binary search
        for (int l = nnz_i - ret; l > 1; l -= floorf(l/2.0)){
          tmp_0 = ret + l;
          tmp_1 = nnz_i - 1;
          testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
          ret = (C[ind0_i + testInd] <= k) ? testInd : ret;
        }

        tmp_0 = ret + 1;
        tmp_1 = nnz_i - 1;
        testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;

        ret = (C[testInd + ind0_i] <= k) ? testInd : ret;

        ind_jk = (C[ret + ind0_i] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;

      }
    }
    c_tmp = -2 * c_tmp + norm_ij;
    c_tmp = (c_tmp > 2e-6) ? sqrt(c_tmp) : 0.0;

    } else {
      c_tmp = 1e30;
    }


    // changed
    /*
    int gid_pt = leafId_local * ppl + block * k_nn + rowId;
    int gid_pt_T = leafId_local * ppl + block * k_nn + colId;
    int ind_knn = gid_pt * k_nn + colId;
    int ind_knn_T = gid_pt_T * k_nn + rowId;
    */
    int gid_pt = leafId_local * ppl + block * partsize + rowId;
    int gid_pt_T = leafId_local * ppl + block * partsize + colId;
    int ind_knn = gid_pt * partsize + colId;
    int ind_knn_T = gid_pt_T * partsize + rowId;

    KNN_dist_tmp[ind_knn] = c_tmp;
    if (colId > rowId) KNN_dist_tmp[ind_knn_T] = c_tmp;

    for (int row_tmp = 0; row_tmp<rem_len; row_tmp++){
      for (int q = ind + rem_len; q < k_nn; q += blockDim.x){
        //gid_pt = leafId_local * ppl + block * k_nn + row_tmp;
        //ind_knn = gid_pt * k_nn + q;
        gid_pt = leafId_local * ppl + block * partsize + row_tmp;
        ind_knn = gid_pt * partsize + q;
        KNN_dist_tmp[ind_knn] = 1e30;
      } 
    } 


  }

}


/*
__global__ void SortGIds(int* G_Id, int ppl, int ppl_pow2){
  int tid = threadIdx.x;
  
  int leafId_g = blockIdx.z* blockDim.y + blockIdx.y; 
  
  __shared__ int SM_GId[12000];
  for (int tid_seq = tid; tid_seq < ppl_pow2; tid_seq += blockDim.x) SM_GId[tid_seq] = (tid_seq < ppl) ? G_Id[leafId_g * ppl + tid_seq] : 1e30;
  
  __syncthreads();
  
  
  int tmp_i;
  for (int g = 2; g <= ppl_pow2; g *= 2){
    for (int l = g/2; l > 0; l /= 2){
      for (int tid_seq = tid; tid_seq < ppl_pow2; tid_seq += blockDim.x){
        
        int ixj = tid_seq ^ l;
        
        if (ixj > tid_seq){
          if ((tid_seq & g) == 0){
            if (SM_GId[tid_seq] > SM_GId[ixj]){
              tmp_i = SM_GId[tid_seq];
              SM_GId[tid_seq] = SM_GId[ixj];
              SM_GId[ixj] = tmp_i;
            }
          } else {
            if (SM_GId[tid_seq] < SM_GId[ixj]){
              tmp_i = SM_GId[tid_seq];
              SM_GId[tid_seq] = SM_GId[ixj];
              SM_GId[ixj] = tmp_i;
            }
          }
        }
      }
    __syncthreads();
    }
  }    
  for (int tid_seq = tid; tid_seq < ppl; tid_seq += blockDim.x) G_Id[leafId_g * ppl + tid_seq] = SM_GId[tid_seq];
    
  
}


*/





__global__ void ComputeRecDists(int* R, int* C, float* V, int* G_Id, float* Norms, int k_nn, int ppl, int blockInd, float* d_knn_temp, int bl, int sizebleaves, int partsize) {


  //__shared__ float SM_dist[SM_SIZE_2];
  //__shared__ int SM_Id[SM_SIZE_2];
  
  int row_l = blockIdx.x;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g =  bl * sizebleaves + leafId_local;
  int j = threadIdx.x;
  
  //int size_part = ppl - (k_nn) * (blockInd+1); 
  int size_part = ppl - (partsize) * (blockInd+1); 
  
  __shared__ int SM[SM_SIZE_1];

  //int rowId_leaf = k_nn * blockInd + row_l;
  int rowId_leaf = partsize * blockInd + row_l;
  int g_rowId_I = leafId_g * ppl + rowId_leaf;
  
  //changed 
  int perm_i = G_Id[g_rowId_I];
  //int perm_i = g_rowId_I;

  int ind0_i = R[perm_i];
  int ind1_i = R[perm_i+1];

  float norm_i = Norms[perm_i];
  int nnz_i = ind1_i - ind0_i;
  

  for (int n_i = j; n_i< nnz_i; n_i += blockDim.x) SM[n_i] = C[ind0_i + n_i];
   
  __syncthreads();

  int C_Y[1024];

  for (int j_tmp = j; j_tmp < size_part; j_tmp += blockDim.x){

    //int colId_leaf = k_nn * (blockInd+1) + j_tmp;
    int colId_leaf = partsize * (blockInd+1) + j_tmp;
      
    int g_rowId_J = leafId_g * ppl + colId_leaf;
      
    //changed 
    int perm_j = G_Id[g_rowId_J];
    //int perm_j = g_rowId_J;
        
    int ind0_j = R[perm_j];
    int ind1_j = R[perm_j+1];

    int nnz_j = ind1_j - ind0_j;

    for (int n_j = 0; n_j < nnz_j; n_j++) C_Y[n_j] = C[ind0_j + n_j];

    float norm_ij = norm_i + Norms[perm_j];
        
    float c_tmp = 0.0;
    int tmp_0, tmp_1, ind_jk, k, ret, testInd;
      
    ret = 0;
    testInd = 0;
      
    // loop over the elements of j
  
    if (nnz_i >0 && nnz_j > 0 && colId_leaf < ppl){
      for (int pos_k = 0; pos_k < nnz_j; pos_k++){
          
        //k = C[ind0_j + pos_k];
        k = C_Y[pos_k];
      
        // Binary search
      
        for (int l = nnz_i - ret; l > 1; l -= floorf(l/2.0)){
  
          tmp_0 = ret + l;
          tmp_1 = nnz_i - 1;
              
          testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
              
          ret = (SM[testInd] <= k) ? testInd : ret;
          //ret = (C[ind0_i + testInd] <= k) ? testInd : ret;
        }

        tmp_0 = ret + 1;
        tmp_1 = nnz_i - 1;
            
        testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;
        ret = (SM[testInd] <= k) ? testInd : ret;
        //ret = (C[ind0_i +testInd] <= k) ? testInd : ret;
            
        ind_jk = (SM[ret] == k) ? ret : -1;
        //ind_jk = (C[ind0_i +ret] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;

        } 
    }
         
    c_tmp = -2 * c_tmp + norm_ij;
    c_tmp = (c_tmp > 1e-8) ? sqrt(c_tmp) : 0.0;
        
        
    int size_tmp = size_part;
    //int ind_tmp = leafId_local * k_nn * size_tmp + row_l * size_tmp + colId_leaf - (k_nn) * (blockInd+1);
    int ind_tmp = leafId_local * partsize * size_tmp + row_l * size_tmp + colId_leaf - (partsize) * (blockInd+1);
    d_knn_temp[ind_tmp] = c_tmp;
  }


}
 



__global__ void MergeHoriz(float* KNN, int* KNN_Id, int k_nn, int ppl, int blockInd, float* d_temp_knn, int* sort_arr, int* sort_arr_part, int steps, int* G_Id, bool init, int bl, int sizebleaves, int partsize){

   
  __shared__ float SM_dist[SM_SIZE_2];
  __shared__ int SM_Id[SM_SIZE_2];



  int j = threadIdx.x;
  int row_l = blockIdx.x;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;
  

  //int size_part = ppl - (k_nn) * blockInd;
  //int size_part = ppl - (k_nn) * blockInd;
  int size_part = ppl - (partsize) * blockInd;
  int size_sort = 2 * blockDim.x;

  int rowId_leaf = k_nn * blockInd + row_l;
  
  for (int n=j; n < SM_SIZE_2; n += blockDim.x){
    SM_dist[n] = 1e30; 
    SM_Id[n] = -1;
  }

  float tmp_f;
  int tmp_i;
  int ind_sort;
   
  int num_batches = size_part / (size_sort - k_nn);
  
  for (int col_batch = 0; col_batch < num_batches; col_batch++){
    for (int j_tmp = j; j_tmp < size_sort; j_tmp += blockDim.x){
      
      int colId_leaf = k_nn * blockInd + col_batch * (size_sort - k_nn) + j_tmp;
      
      if (col_batch == 0 && j_tmp < k_nn){
        
        int ind_pt = G_Id[leafId_g * ppl + rowId_leaf];
        int ind_read = ind_pt * k_nn + j_tmp;
        SM_dist[j_tmp] = KNN[ind_read];
        SM_Id[j_tmp] = KNN_Id[ind_read];
      } else if (colId_leaf < ppl && j_tmp >= k_nn){

        int size_tmp = size_part - k_nn;
        int ind_tmp = leafId_local * k_nn * size_tmp + row_l * size_tmp + colId_leaf - (k_nn) * (blockInd+1);
        int g_colId_J = leafId_g * ppl + colId_leaf;
        
        SM_dist[j_tmp] = d_temp_knn[ind_tmp];
        SM_Id[j_tmp] = G_Id[g_colId_J];
      }
      //if (init == 0 && rowId_leaf == 1149 && leaf_id_g == 0 && col_batch == 0) printf("D[%d] = %.4f , at %d  \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);
    }

    __syncthreads();
       
    for (int j_tmp = j; j_tmp < size_sort; j_tmp += blockDim.x) {

      if (j_tmp >= k_nn){
        int index = SM_Id[j_tmp];
        for (int ind_check = 0; ind_check < k_nn; ind_check++){
          if (index == SM_Id[ind_check]){
            SM_Id[j_tmp] = -1;
            SM_dist[j_tmp] = 1e30;
            break;
          }
        }
      }

    }
    __syncthreads();
    


    for (int step = 0; step < steps; step++){
    
      int j_tmp = j;
      ind_sort = step * 2 * blockDim.x + j_tmp;

      int tid = sort_arr[ind_sort];
      int ixj = sort_arr_part[ind_sort];

      int min_max = (1 & tid);
      int coupled_flag = (1 & ixj);

      tid = tid >> 1;
      ixj = ixj >> 1;

      if (coupled_flag == 1){

        ind_sort += blockDim.x;

        int tid_1 = sort_arr[ step * 2 * blockDim.x + j_tmp + blockDim.x];
        int ixj_1 = sort_arr_part[step * 2 * blockDim.x + j_tmp + blockDim.x];
        int min_max_1 = (1 & tid_1);


        tid_1 = tid_1 >> 1;
        ixj_1 = ixj_1 >> 1;


        if (min_max_1 == 1 && SM_dist[tid_1] > SM_dist[ixj_1]){

          tmp_f = SM_dist[tid_1];
          SM_dist[tid_1] = SM_dist[ixj_1];
          SM_dist[ixj_1] = tmp_f;

          tmp_i = SM_Id[tid_1];
          SM_Id[tid_1] = SM_Id[ixj_1];
          SM_Id[ixj_1] = tmp_i;

        }

        if (min_max_1 == 0 && SM_dist[tid] < SM_dist[ixj]){

          tmp_f = SM_dist[tid_1];
          SM_dist[tid_1] = SM_dist[ixj_1];
          SM_dist[ixj_1] = tmp_f;

          tmp_i = SM_Id[tid_1];
          SM_Id[tid_1] = SM_Id[ixj_1];
          SM_Id[ixj_1] = tmp_i;

        }

      }

      if (min_max == 1){
        if (SM_dist[tid] > SM_dist[ixj]){
          tmp_f = SM_dist[tid];
          SM_dist[tid] = SM_dist[ixj];
          SM_dist[ixj] = tmp_f;

          tmp_i = SM_Id[tid];
          SM_Id[tid] = SM_Id[ixj];
          SM_Id[ixj] = tmp_i;
        }
      } else {
        if (SM_dist[tid] < SM_dist[ixj]){
          tmp_f = SM_dist[tid];
          SM_dist[tid] = SM_dist[ixj];
          SM_dist[ixj] = tmp_f;

          tmp_i = SM_Id[tid];
          SM_Id[tid] = SM_Id[ixj];
          SM_Id[ixj] = tmp_i;
        }
      }
    
      __syncthreads();
    }


  }
  for (int j_tmp = j; j_tmp < k_nn; j_tmp += blockDim.x){ 
    if (j_tmp < k_nn){
      int ind_pt = leafId_g * ppl + rowId_leaf;
      int write_ind = G_Id[ind_pt] * k_nn + j_tmp;
      KNN[write_ind] = SM_dist[j_tmp];
      KNN_Id[write_ind] = SM_Id[j_tmp];
      //if (init == 0 && rowId_leaf == 1149 && leaf_id_g == 0) printf("sorted D[%d] = %.4f , at %d  \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);
    }
  } 
}



__global__ void MergeVer(float* KNN, int* KNN_Id, int k_nn, int ppl, int blockInd, float* d_temp_knn, int* sort_arr, int* sort_arr_part, int steps, int* G_Id, bool init, int bl, int sizebleaves, int partsize){

  __shared__ float SM_dist[SM_SIZE_1];
  __shared__ int SM_Id[SM_SIZE_1];


  int j = threadIdx.x;

  int col = blockIdx.x;
  int leafId_local = blockIdx.z* blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;
  //int colId_leaf = (init) ? col : col + k_nn * (blockInd + 1);
  //int size_part = (init) ? ppl : ppl - (blockInd + 1) * (k_nn);
  int colId_leaf = (init) ? col : col + partsize * (blockInd + 1);
  int size_part = (init) ? ppl : ppl - (blockInd + 1) * partsize;


  //int ind_tmp = (init) ? leafId_local * ppl * k_nn + col * k_nn + j : leafId_local * k_nn * size_part + j * size_part + col;
  for (int j_tmp = j; j_tmp < partsize; j_tmp += blockDim.x){
		int ind_tmp = (init) ? leafId_local * ppl * partsize + col * partsize + j_tmp : leafId_local * partsize * size_part + j_tmp * size_part + col;
		SM_dist[j_tmp] = d_temp_knn[ind_tmp];
		int block = col / k_nn;
  //int rowId_g = (init) ? leafId_g * ppl + block * k_nn + j : leafId_g * ppl + k_nn * blockInd + j;
		int rowId_g = (init) ? leafId_g * ppl + block * partsize + j_tmp : leafId_g * ppl + partsize * blockInd + j_tmp;
		SM_Id[j] = G_Id[rowId_g];

		int ind_pt_knn = leafId_g * ppl + colId_leaf;
		int ind_pt_knn_g = G_Id[ind_pt_knn];
 
		int ind_knn = ind_pt_knn_g * k_nn + j_tmp;
		//SM_dist[j + k_nn] = KNN[ind_knn];
		//SM_Id[j + k_nn] = KNN_Id[ind_knn];
		if (j_tmp < k_nn){ 
			SM_dist[j_tmp + partsize] = KNN[ind_knn];
			SM_Id[j_tmp + partsize] = KNN_Id[ind_knn];
		} else {
      SM_dist[j_tmp + partsize] = 1e30;
      SM_Id[j_tmp + partsize] = -1;
    }
  }
	__syncthreads();
  
  for (int j_tmp = j; j_tmp < partsize; j_tmp += blockDim.x){ 
    int index = SM_Id[j_tmp];
    for (int ind_check = 0; ind_check < k_nn; ind_check++){
      if (index == SM_Id[ind_check + partsize]){
        SM_dist[j_tmp] = 1e30;
        SM_Id[j_tmp] = -1;
        break;
      }
    }
  }
  __syncthreads();


  float tmp_f;
  int tmp_i;

  for (int step = 0 ; step < steps; step++){

    int ind_sort = step * 2 * blockDim.x + j;
    int tid = sort_arr[ind_sort];
    int ixj = sort_arr_part[ind_sort];
    int min_max = (1 & tid);
    int coupled_flag = (1 & ixj);

    tid = tid >> 1;
    ixj = ixj >> 1;
    if (coupled_flag == 1){
        
      ind_sort += blockDim.x;
      int tid_1 = sort_arr[ind_sort];
      int ixj_1 = sort_arr_part[ind_sort];
      int min_max_1 = (1 & tid_1);
      
      
      
      tid_1 = tid_1 >> 1;
      ixj_1 = ixj_1 >> 1;
      

      if (min_max_1 == 1 && SM_dist[tid_1] > SM_dist[ixj_1]){
      
        tmp_f = SM_dist[tid_1];
        SM_dist[tid_1] = SM_dist[ixj_1];
        SM_dist[ixj_1] = tmp_f;
       
        tmp_i = SM_Id[tid_1];
        SM_Id[tid_1] = SM_Id[ixj_1];
        SM_Id[ixj_1] = tmp_i;
                
      }

      if (min_max_1 == 0 && SM_dist[tid] < SM_dist[ixj]){
      
        tmp_f = SM_dist[tid_1];
        SM_dist[tid_1] = SM_dist[ixj_1];
        SM_dist[ixj_1] = tmp_f;
       
        tmp_i = SM_Id[tid_1];
        SM_Id[tid_1] = SM_Id[ixj_1];
        SM_Id[ixj_1] = tmp_i;
                
      }
      
    } 
    
     
    if (min_max == 1){
      if (SM_dist[tid] > SM_dist[ixj]){
        tmp_f = SM_dist[tid];
        SM_dist[tid] = SM_dist[ixj];
        SM_dist[ixj] = tmp_f;
      
        tmp_i = SM_Id[tid];
        SM_Id[tid] = SM_Id[ixj];
        SM_Id[ixj] = tmp_i;
      } 
    } else {
      if (SM_dist[tid] < SM_dist[ixj]){
        tmp_f = SM_dist[tid];
        SM_dist[tid] = SM_dist[ixj];
        SM_dist[ixj] = tmp_f;
      
        tmp_i = SM_Id[tid];
        SM_Id[tid] = SM_Id[ixj];
        SM_Id[ixj] = tmp_i;
      }
    }



  __syncthreads();

  }

  
  if (j < k_nn){ 
		int ind_pt_knn = leafId_g * ppl + colId_leaf;
		int ind_pt_knn_g = G_Id[ind_pt_knn];
		int ind_knn = ind_pt_knn_g * k_nn + j;
		KNN[ind_knn] = SM_dist[j];
		KNN_Id[ind_knn] = SM_Id[j];
  } 
  

}

/*
__global__ void knn_kernel_B(float* KNN, int* KNN_Id, int k_nn, int ppl, int blockInd, float* d_temp_knn, int* G_Id, bool init, int bl, int sizebleaves){

  __shared__ float SM_dist[SM_SIZE_1];
  __shared__ int SM_Id[SM_SIZE_1];

  int tid = threadIdx.x;

  int col = blockIdx.x;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;
  
  int colId_leaf = (init) ? col : col + k_nn * (blockInd + 1);
  int size_part = (init) ? ppl : ppl - (blockInd + 1) * (k_nn);
  if (tid < k_nn){
    //changed 
    int ind_tmp = (init) ? leafId_local * ppl * k_nn + col * k_nn + tid : leafId_local * k_nn * size_part + tid * size_part + col;
    SM_dist[tid] = (colId_leaf < ppl) ? d_temp_knn[ind_tmp] : 1e30;
   
    int block = col / k_nn;
    int rowId_g = (init) ? leafId_g * ppl + block * k_nn + tid : leafId_g * ppl + k_nn * blockInd + tid;  
    SM_Id[tid] = (colId_leaf < ppl) ? G_Id[rowId_g] : -1;
  } else {

    int ind_pt_knn = leafId_g * ppl + colId_leaf;
    int ind_pt_knn_g = G_Id[ind_pt_knn];

    int ind_knn = ind_pt_knn_g * k_nn + tid - k_nn;
    SM_dist[tid] = (colId_leaf < ppl) ? KNN[ind_knn] : 1e30;
    SM_Id[tid] = (colId_leaf < ppl) ? KNN_Id[ind_knn] : -1;
      
  }


  
  __syncthreads();
  
  if (tid < k_nn){
    int index = SM_Id[tid];
    for (int ind_check = 0; ind_check < k_nn; ind_check++){
      if (index == SM_Id[ind_check + k_nn]){
        SM_dist[tid] = 1e30;
        SM_Id[tid] = -1;
        break;
      }
    }
    
  }
  
  
  __syncthreads(); 
 
 
  // sort 
  float tmp_f;
  int tmp_i;

  int size = 2 * k_nn;
  for (int g = 2; g <= size; g *= 2){
    for (int l = g/2; l > 0; l /= 2){

      int ixj = tid ^ l;
      
      if (ixj > tid){
        if ((tid & g) == 0){
          if (SM_dist[tid] > SM_dist[ixj]){

            tmp_f = SM_dist[ixj];
            SM_dist[ixj] = SM_dist[tid];
            SM_dist[tid] = tmp_f;

            tmp_i = SM_Id[ixj];
            SM_Id[ixj] = SM_Id[tid];
            SM_Id[tid] = tmp_i;

          }
        } else {
          if (SM_dist[tid] < SM_dist[ixj]){

            tmp_f = SM_dist[ixj];
            SM_dist[ixj] = SM_dist[tid];
            SM_dist[tid] = tmp_f;

            tmp_i = SM_Id[ixj];
            SM_Id[ixj] = SM_Id[tid];
            SM_Id[tid] = tmp_i;

          }
        }
      }
      __syncthreads();
      
    }
  }
  
  int ind_pt = leafId_g * ppl + colId_leaf;    
  int ind_pt_g = G_Id[ind_pt];
  int write_ind = ind_pt_g * k_nn + tid;
   
  if (tid < k_nn) {
    KNN[write_ind] = SM_dist[tid];
    KNN_Id[write_ind] = SM_Id[tid];
  }

}
*/

void PrecompSortIds(int* d_arr, int* d_arr_part, int N_true, int N_pow2, int steps, int copy_size){

  
  
  int min_max, elem, coupled_elem;
  int loc_len = ceil(N_true/2);
  int* tracker;
  tracker = (int *)malloc(sizeof(int) * N_pow2);
  
  for (int i = 0; i < N_pow2; i ++) tracker[i] = i;

  int step = 0;

  int *arr, *arr_part;
  arr = (int *)malloc(sizeof(int) * copy_size); 
  arr_part = (int *)malloc(sizeof(int) * copy_size);

  
  memset(arr, 0, sizeof(int) * copy_size);
  memset(arr_part, 0, sizeof(int) * copy_size);
 
  int first_pair = 1;
  int prev_elem = 0;
  int tmp2;
  for (int g = 2; g <= N_pow2; g *= 2){
    for (int l = g/2; l > 0; l /= 2){
      elem = 0;
      for (int i = 0; i < N_pow2; i++){
        int ixj = i ^ l;

        if (tracker[ixj] >= N_true && tracker[i] >= N_true) continue;

        if (ixj > i){

          min_max = ((i&g) == 0 ) ? 1 : 0;

          coupled_elem = 0;

          int write_loc = elem;
          
          if (tracker[ixj] >= N_true || tracker[i] >= N_true) {
            coupled_elem = 1;
            if (min_max == 0 && tracker[ixj] >= N_true) {
              tmp2 = tracker[ixj];
              tracker[ixj] = tracker[i];
              tracker[i] = tmp2;
            }
            if (min_max == 1 && tracker[i] >= N_true) {
              tmp2 = tracker[ixj];
              tracker[ixj] = tracker[i];
              tracker[i] = tmp2;
            }
            if (first_pair == 1){
              prev_elem = elem;
              write_loc = elem;
              first_pair = 0;
            } else {
              write_loc = prev_elem + loc_len;
              first_pair = 1;
              elem++;
            }
            
            arr[step * N_true + write_loc] = (i << 1) + min_max;
            arr_part[step * N_true + write_loc] = (ixj << 1) + coupled_elem;

          } else {
            write_loc = elem;
            arr[step * N_true + write_loc] = (i << 1) + min_max;
            arr_part[step * N_true + write_loc] = (ixj << 1) + coupled_elem;
            elem++;
          }




        }
      }    

      step++;
    }
  }
  checkCudaErrors(hipMemcpy(d_arr, arr, sizeof(int)*copy_size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_arr_part, arr_part, sizeof(int)*copy_size, hipMemcpyHostToDevice));
  //checkCudaErrors(hipDeviceSynchronize());
  
  free(arr);
  free(arr_part); 
  free(tracker); 
  
}



void sfi_leafknn(int *R, int *C, float *V, int *G_Id, int M, int leaves, int k, float *knn, int *knn_Id){



  float dt1, dt2, dt3, dt4, dt5, dt6, dt7, dt8, dt9, dt_tmp;
  hipEvent_t t0;
  hipEvent_t t1;
  hipEvent_t t2;
  hipEvent_t t3;
  hipEvent_t t4;
  hipEvent_t t5;
  hipEvent_t t6;
  hipEvent_t t7;
  hipEvent_t t8;
  hipEvent_t t9;

  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));
  checkCudaErrors(hipEventCreate(&t2));
  checkCudaErrors(hipEventCreate(&t3));
  checkCudaErrors(hipEventCreate(&t4));
  checkCudaErrors(hipEventCreate(&t5));
  checkCudaErrors(hipEventCreate(&t6));
  checkCudaErrors(hipEventCreate(&t7));
  checkCudaErrors(hipEventCreate(&t8));
  checkCudaErrors(hipEventCreate(&t9));

  checkCudaErrors(hipEventRecord(t0, 0));



  int ppl = M/leaves;


  int partsize = (k > 32) ? k : 32;
  int num_blocks_tri = ppl / partsize;
  //if (num_blocks_tri * k < ppl) num_blocks_tri += 1;
  //int rem_len = (num_blocks_tri * k < ppl) ? ppl - num_blocks_tri * k : 0;
  int rem_len = (num_blocks_tri * partsize < ppl) ? ppl - num_blocks_tri * partsize : 0;
 
  int C_len = R[M];


  int t_b = (ppl > SM_SIZE_1) ? SM_SIZE_1 : ppl;
  //float tmp = leaves / 65535;
  int num_splits = 1;
  while (leaves > num_splits * 65535) num_splits *= 2;
  //int num_splits = ceil(tmp);

  //printf("leaves = %d , tmp = %.4f , num_splits = %d \n ", leaves, tmp, num_splits);
  int batch_leaves_1 = (leaves > 64000) ? leaves / num_splits : leaves;
  int batch_leaves_2 = (leaves > 64000) ? num_splits : 1;

  int verbose = 1;

  dim3 BlockNorm(t_b, 1, 1);
  dim3 GridNorm(1, batch_leaves_1, batch_leaves_2);


  if (verbose) printf("----------------------------- Start of sfiknn ----------------------------- \n\n");

  float *d_Norms;


  int size_tri = partsize;
  int blockDim_tri = size_tri * (size_tri + 1)/2;
  if (blockDim_tri > SM_SIZE_1) blockDim_tri = SM_SIZE_1;

  int size_tri_last = (rem_len > 32) ? 32 : rem_len;
  int blockDim_tri_last = size_tri_last * (size_tri_last + 1)/2;
  if (blockDim_tri_last > SM_SIZE_1) blockDim_tri_last = SM_SIZE_1;

  dim3 BlockDistTri(blockDim_tri, 1, 1);
  dim3 GridDistTri(num_blocks_tri, batch_leaves_1, batch_leaves_2);
  dim3 BlockDistTri_last(blockDim_tri_last, 1, 1);
  dim3 GridDistTri_last(1, batch_leaves_1, batch_leaves_2);


  //dim3 dimGrid_sq(k, batch_leaves_1, batch_leaves_2);
  dim3 GridDistRec(partsize, batch_leaves_1, batch_leaves_2);
  dim3 GridMergeHoriz(partsize, batch_leaves_1, batch_leaves_2);

  int size_v_block_reduced = (k + partsize)/2;
  dim3 BlockMergeVer(size_v_block_reduced, 1, 1);
  
  printf("=======================\n");
  printf(" Num points = %d \n", M);
  printf(" pt/leaf = %d \n", ppl);
  printf(" Leaves = %d \n", leaves);
  printf(" K = %d \n", k);
  printf(" PartSize = %d \n", partsize);

  printf(" dim BlockThreads  Norms = (%d , %d, %d) \n", BlockNorm.x, BlockNorm.y, BlockNorm.z);
  printf(" dim GridThreads Norms = (%d , %d, %d) \n", GridNorm.x, GridNorm.y, GridNorm.z);
  printf(" dim BlockThreads Diagonal Distances = (%d , %d, %d) \n", BlockDistTri.x, BlockDistTri.y, BlockDistTri.z);
  printf(" dim GridThreads Diagonal Distances = (%d , %d, %d) \n", GridDistTri.x, GridDistTri.y, GridDistTri.z);
  printf(" dim BlockThreads Diagonal Distances last = (%d , %d, %d) \n", BlockDistTri_last.x, BlockDistTri_last.y, BlockDistTri_last.z);
  printf(" dim GridThreads Diagonal Distances last = (%d , %d, %d) \n", GridDistTri_last.x, GridDistTri_last.y, GridDistTri_last.z);
  printf(" dim GridThreads Distance Horiz = (%d , %d, %d) \n", partsize, batch_leaves_1, batch_leaves_2);
  printf(" dim GridThreads MergeVer = (%d , %d, %d) \n", GridMergeHoriz.x, GridMergeHoriz.y, GridMergeHoriz.z);
  printf(" dim BlockMerge MergeVer = (%d , %d, %d) \n", BlockMergeVer.x, BlockMergeVer.y, BlockMergeVer.z);
  



  int *d_arr, *d_arr_part, *d_arr_v, *d_arr_part_v;
  float SM_SIZE_2_f = SM_SIZE_2;
  int n_s = log2(SM_SIZE_2_f) *(log2(SM_SIZE_2_f)+1) /2;

  int copy_size = (ppl) * n_s;
  //float tmp = 2*k;
  //float tmp = 2*k;
  float tmp = 2*partsize;
  int n_s_v = log2(tmp) * (log2(tmp)+1)/2;
  //int copy_size_v = k * n_s;
  int copy_size_v = (2 * partsize) * n_s_v;

  size_t free, total, m1, m2, m3;

  int *d_R, *d_GId, *d_C, *d_knn_Id;
  float *d_V, *d_knn;  
  
  
  
  checkCudaErrors(hipMalloc((void **) &d_R, sizeof(int) * (M+1)));
  checkCudaErrors(hipMalloc((void **) &d_GId, sizeof(int) * M));
  checkCudaErrors(hipMalloc((void **) &d_C, sizeof(int) * C_len));
  checkCudaErrors(hipMalloc((void **) &d_V, sizeof(float) * C_len));

  checkCudaErrors(hipMalloc((void **) &d_knn_Id, sizeof(int) *M*k));
  checkCudaErrors(hipMalloc((void **) &d_knn, sizeof(float) *M*k));

  checkCudaErrors(hipMemcpy(d_R, R, sizeof(int) * (M+1), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_C, C, sizeof(int) * C_len, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_V, V, sizeof(float) * C_len, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_GId, G_Id, sizeof(int) * M, hipMemcpyHostToDevice)); 
  checkCudaErrors(hipMemcpy(d_knn, knn, sizeof(float) * M * k, hipMemcpyHostToDevice)); 
  checkCudaErrors(hipMemcpy(d_knn_Id, knn_Id, sizeof(int) * M * k, hipMemcpyHostToDevice)); 

  hipMemGetInfo(&free, &total);
  checkCudaErrors(hipMalloc((void **) &d_arr, sizeof(int) * copy_size));
  checkCudaErrors(hipMalloc((void **) &d_arr_part, sizeof(int) * copy_size));

  checkCudaErrors(hipMalloc((void **) &d_arr_v, sizeof(int) * copy_size_v));
  checkCudaErrors(hipMalloc((void **) &d_arr_part_v, sizeof(int) * copy_size_v));


  checkCudaErrors(hipMemset(d_arr, 0, sizeof(int) * copy_size));
  checkCudaErrors(hipMemset(d_arr_part, 0, sizeof(int) * copy_size));
  checkCudaErrors(hipMemset(d_arr_v, 0, sizeof(int) * copy_size_v));
  checkCudaErrors(hipMemset(d_arr_part_v, 0, sizeof(int) * copy_size_v));
  hipMemGetInfo(&m1, &total);



  checkCudaErrors(hipEventRecord(t1, 0));
   
  int size_sort_ver = k + partsize;
  int size_sort_ver_pow2 = 2*partsize;
  PrecompSortIds(d_arr_v, d_arr_part_v, size_sort_ver, size_sort_ver_pow2, n_s_v, copy_size_v);


  checkCudaErrors(hipEventRecord(t2, 0));

  float * d_temp_knn;
  checkCudaErrors(hipMalloc((void **) &d_Norms, sizeof(float) * M));

  hipMemGetInfo(&m2, &total);
  //float size_tmp = sizeof(float) * M * ;
  float size_tmp = sizeof(float) * M * partsize;
  int bleaves = (size_tmp > m2) ? log2(size_tmp / m2) : 0;
  int numbleaves = 1 << bleaves;
  int sizebleaves = leaves / numbleaves; 
  printf(" Num BatchLeaves = %d \n", numbleaves);
  printf(" Size BatchLeaves = %d \n", sizebleaves);
  printf("=======================\n");

  

  
  checkCudaErrors(hipMalloc((void **) &d_temp_knn, sizeof(float) * sizebleaves * ppl * partsize));
  hipMemGetInfo(&m3, &total);

  int steps;

  ComputeNorms <<< GridNorm, BlockNorm >>>(d_R, d_C, d_V, d_GId, d_Norms, ppl);
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipEventRecord(t3, 0));
  dt5 = 0.0; 
  dt6 = 0.0; 
  dt7 = 0.0; 
  for (int bl = 0; bl < numbleaves; bl++){

		ComputeTriDists <<< GridDistTri, BlockDistTri >>>(d_R, d_C, d_V, d_GId, d_Norms, k, d_temp_knn, ppl, bl, sizebleaves, partsize);
		checkCudaErrors(hipDeviceSynchronize());
  
		if (rem_len > 0) {
			ComputeTriDists_last <<< GridDistTri_last, BlockDistTri_last >>>(d_R, d_C, d_V, d_GId, d_Norms, k, d_temp_knn, ppl, rem_len, num_blocks_tri, bl, sizebleaves, partsize);
			checkCudaErrors(hipDeviceSynchronize());
		}


		int size_v = ppl;
		dim3 GridMergeVer(size_v, batch_leaves_1, batch_leaves_2);
    
		MergeVer <<< GridMergeVer, BlockMergeVer >>> (d_knn, d_knn_Id, k, ppl, 0, d_temp_knn, d_arr_v, d_arr_part_v, n_s_v, d_GId, true, bl, sizebleaves, partsize);
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipEventRecord(t4, 0));
  
		int num_iters = (rem_len > 0) ? num_blocks_tri : num_blocks_tri - 1;
		for (int blockInd = 0; blockInd < num_iters; blockInd++){

		  checkCudaErrors(hipEventRecord(t5, 0));	
			
      int size_part = ppl - blockInd *k;
			int size_sort = size_part;

			while (size_sort > SM_SIZE_2) size_sort = ceil((size_sort+k)/2);
			float tmp = size_sort/2.0;
			int blocksize = ceil(tmp);
			float tmp_f = 2 * blocksize;
			int N_pow2 = pow(2, ceil(log2(tmp_f)));
			tmp_f = N_pow2;
			steps = log2(tmp_f) * (log2(tmp_f) +1)/2;
			


			int real_size = 2 * blocksize;
			PrecompSortIds(d_arr, d_arr_part, real_size, N_pow2, steps, copy_size);

			int blocksize_dist = size_part - partsize;
			while(blocksize_dist > SM_SIZE_1) blocksize_dist = ceil(blocksize_dist / 2.0);


			dim3 BlockDistRec( blocksize_dist, 1, 1);
			dim3 BlockMergeHoriz( blocksize, 1, 1);

			int size_v2 = ppl - (blockInd + 1) * partsize;
			dim3 GridMergeVer(size_v2, batch_leaves_1, batch_leaves_2);

			ComputeRecDists <<< GridDistRec, BlockDistRec >>> (d_R, d_C, d_V, d_GId, d_Norms, k, ppl, blockInd, d_temp_knn, bl, sizebleaves, partsize);
    	checkCudaErrors(hipDeviceSynchronize());
		  checkCudaErrors(hipEventRecord(t6, 0));
			
			MergeHoriz <<< GridMergeHoriz, BlockMergeHoriz >>> (d_knn, d_knn_Id, k, ppl, blockInd, d_temp_knn, d_arr, d_arr_part, steps, d_GId, false, bl, sizebleaves, partsize); 
			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipEventRecord(t7, 0));
		
			MergeVer <<< GridMergeVer, BlockMergeVer >>> (d_knn, d_knn_Id, k, ppl, blockInd, d_temp_knn, d_arr_v, d_arr_part_v, n_s_v, d_GId, false,bl, sizebleaves, partsize);
			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipEventRecord(t8, 0));
      checkCudaErrors(hipEventElapsedTime(&dt_tmp, t5, t6));
      dt5 += dt_tmp;
      checkCudaErrors(hipEventElapsedTime(&dt_tmp, t6, t7));
      dt6 += dt_tmp;
      checkCudaErrors(hipEventElapsedTime(&dt_tmp, t7, t8));
      dt7 += dt_tmp;
    }
  
  }



  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventSynchronize(t9));
  checkCudaErrors(hipEventRecord(t9, 0));
  checkCudaErrors(hipEventElapsedTime(&dt1, t0, t1));
  checkCudaErrors(hipEventElapsedTime(&dt2, t1, t2));
  checkCudaErrors(hipEventElapsedTime(&dt3, t2, t3));
  checkCudaErrors(hipEventElapsedTime(&dt4, t3, t4));
  checkCudaErrors(hipEventElapsedTime(&dt8, t4, t9));
  checkCudaErrors(hipEventElapsedTime(&dt9, t0, t9));

  //checkCudaErrors(hipMemcpy(knn, d_knn, sizeof(float) * M * k, hipMemcpyDeviceToHost));
  //checkCudaErrors(hipMemcpy(knn_Id, d_knn_Id, sizeof(int) * M * k, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(knn, d_knn, sizeof(float) * M * k, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(knn_Id, d_knn_Id, sizeof(int) * M * k, hipMemcpyDeviceToHost));


  checkCudaErrors(hipFree(d_Norms));
  checkCudaErrors(hipFree(d_temp_knn));
  checkCudaErrors(hipFree(d_arr_part));
  checkCudaErrors(hipFree(d_arr));
  checkCudaErrors(hipFree(d_arr_part_v));
  checkCudaErrors(hipFree(d_arr_v));

  checkCudaErrors(hipEventDestroy(t0));
  checkCudaErrors(hipEventDestroy(t1));
  checkCudaErrors(hipEventDestroy(t2));
  checkCudaErrors(hipEventDestroy(t3));
  checkCudaErrors(hipEventDestroy(t4));
  checkCudaErrors(hipEventDestroy(t5));
  checkCudaErrors(hipEventDestroy(t6));
  checkCudaErrors(hipEventDestroy(t7));
  checkCudaErrors(hipEventDestroy(t8));
  checkCudaErrors(hipEventDestroy(t9));
  //hipMemGetInfo(&free, &total);
  printf("--------------- Timings ----------------\n");
  printf("Memory allocation = %.4f (%.4f %%) \n", dt1/1e3, dt1/dt9);
  printf("Precomp sortId (vertical) = %.4f (%.4f %%) \n", dt2/1e3, dt2/dt9);
  printf("Computing norms = %.4f (%.4f %%) \n", dt3/1e3, dt3/dt9);
  printf("Diagonal part = %.4f (%.4f %%) \n", dt4/1e3, dt4/dt9);
  printf("Iterative part = %.4f (%.4f %%) \n", dt8/1e3, dt8/dt9);
  printf("\tCompute Dists = %.4f (%.4f %%) \n", dt5/1e3, dt5/dt9);
  printf("\tMerge Horizontally = %.4f (%.4f %%) \n", dt6/1e3, dt6/dt9);
  printf("\tMerge Vertically  = %.4f (%.4f %%) \n", dt7/1e3, dt7/dt9);
  printf("Total = %.4f \n", dt9/1e3);
  printf("--------------- Memory usage ----------------\n");
  printf("Storing norms = %.4f GB \n", (m1-m2)/1e9);
  printf("Precomputing the sort indices = %.4f GB \n", (free-m1)/1e9);
  printf("Temporary storage = %.4f GB \n", (m2-m3)/1e9);
  printf("----------------------------- End of leaf-knn -----------------------------\n\n");

}











