#include "hip/hip_runtime.h"

#define SM_SIZE_1 1024
#define SM_SIZE_2 2048
#define SM_SIZE_SORT 8192

#include "sfiknn.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void ComputeNorms(int* R, int* C, float* V, int* G_Id, float* Norms, int ppl) {

  int ind = threadIdx.x;
  int leaf_id_g = blockIdx.z * blockDim.y + blockIdx.y;
  for (int row = ind; row < ppl; row += blockDim.x){
    int g_rowId = leaf_id_g * ppl + row;
    //changed
  
    int g_Id = g_rowId;

   
    int ind0_i = R[g_Id];
 
    int nnz = R[g_Id + 1] - ind0_i;
    float norm_i = 0.0;
    float v;
    for (int n_i = 0; n_i < nnz; n_i += 1) {
      v = V[ind0_i + n_i];
      norm_i += v * v;
      //if (G_Id[g_rowId] == 777478) printf("norm = %.4f, V[0] = %.4f , R = %d , nnz = %d \n", norm_i, v, ind0_i, nnz);
    }
    Norms[g_Id] = norm_i;
  }
}

__global__ void ComputeTriDists(int* R, int* C, float* V, int* G_Id, float* Norms , int k_nn, float* KNN_dist_tmp, int ppl, int bl, int sizebleaves, int partsize) {



  int ind = threadIdx.x;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;
;
  int block = blockIdx.x;



  int size_block = partsize * (partsize + 1) /2;
  
  for (int elem = ind; elem < size_block; elem += blockDim.x){

    float tmp = -8 * elem + 4 * partsize * (partsize+1) - 7;
    int rowId = sqrt(tmp)/2.0 - 0.5;
    rowId = partsize - 1 - rowId;
    int colId = elem + rowId - partsize * (partsize + 1) / 2 + (partsize - rowId) * ((partsize - rowId) + 1)/2;

    float c_tmp = 0.0;

    int g_rowId = leafId_g * ppl + block * partsize + rowId;
    int g_colId = leafId_g * ppl + block * partsize + colId;
   
    //changed 
    int perm_i = g_rowId;
    int perm_j = g_colId;

    int ind0_i = R[perm_i];
    int ind1_i = R[perm_i + 1];

    int ind0_j = R[perm_j];
    int ind1_j = R[perm_j + 1];

    int nnz_i = ind1_i - ind0_i;
    int nnz_j = ind1_j - ind0_j;

    float norm_ij = Norms[perm_i] + Norms[perm_j];

    int tmp_0, tmp_1, ind_jk, k, ret, testInd;

    ret = 0;
    testInd = 0;


    if (nnz_i > 0 && nnz_j >0){
      for (int pos_k = 0; pos_k < nnz_j; pos_k++){
    
        k = C[ind0_j + pos_k];
        
        // Binary search
        for (int l = nnz_i - ret; l > 1; l -= floorf(l/2.0)){
          tmp_0 = ret + l;
          tmp_1 = nnz_i - 1;
          testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
          ret = (C[ind0_i + testInd] <= k) ? testInd : ret;
        }
      
        tmp_0 = ret + 1;
        tmp_1 = nnz_i - 1;
        testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;
      
        ret = (C[testInd + ind0_i] <= k) ? testInd : ret;
      
        ind_jk = (C[ret + ind0_i] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;
      
      }
    }
    c_tmp = -2 * c_tmp + norm_ij;
    c_tmp = (c_tmp > 2e-6) ? sqrt(c_tmp) : 0.0;
    //if (G_Id[leafId_g * ppl + block * partsize + rowId] == 777478) printf("D[%d] = %.4f for %d , leaf = %d \n", colId , c_tmp, G_Id[leafId_g * ppl + block * partsize + colId], leafId_local); 
    //if (G_Id[leafId_g * ppl + block * partsize + colId] == 777478) printf("D[%d] = %.4f for %d , leaf = %d \n", rowId , c_tmp, G_Id[leafId_g * ppl + block * partsize + rowId], leafId_local); 
    
    // changed 
    int gid_pt = leafId_local * ppl + block * partsize + rowId;
    int gid_pt_T = leafId_local * ppl + block * partsize + colId;
    int ind_knn = gid_pt * partsize + colId;
    int ind_knn_T = gid_pt_T * partsize + rowId;
    //KNN_dist_tmp[ind_knn] = (iternum >0 && colId == rowId) ? 1e30 : c_tmp;
    
    //if (leafId_g ==0 && g_rowId == 0) printf("D[%d] = %.4f write at %d \n", colId, c_tmp, ind_knn);
    //if (ind_knn < 0 || ind_knn_T < 0 || ind_knn >= gridDim.y * ppl * partsize || ind_knn_T >= gridDim.y * ppl * partsize) printf("print err ind_knn = %d , ind_knn_T = %d \n", ind_knn, ind_knn_T);
    
    KNN_dist_tmp[ind_knn] = c_tmp;
    if (colId > rowId) KNN_dist_tmp[ind_knn_T] = c_tmp;
    
  }
  
}



__global__ void ComputeTriDists_last(int* R, int* C, float* V, int* G_Id, float* Norms , int k_nn, float* KNN_dist_tmp, int ppl, int rem_len , int blockId, int bl, int sizebleaves, int partsize) {




  int ind = threadIdx.x;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;
  int block = blockId;
  


  int size_block = rem_len * (rem_len + 1) /2;


  for (int elem = ind; elem < size_block; elem += blockDim.x){

    float tmp = -8 * elem + 4 * rem_len * (rem_len+1) - 7;
    int rowId = sqrt(tmp)/2.0 - 0.5;
    rowId = rem_len - 1 - rowId;
    int colId = elem + rowId - rem_len * (rem_len + 1) / 2 + (rem_len - rowId) * ((rem_len - rowId) + 1)/2;

    float c_tmp = 0.0;
    if (block * partsize + rowId < ppl && block * partsize + colId < ppl){

    int g_rowId = leafId_g * ppl + block * partsize + rowId;
    int g_colId = leafId_g * ppl + block * partsize + colId;

    //changed
    int perm_i = g_rowId;
    int perm_j = g_colId;

    int ind0_i = R[perm_i];
    int ind1_i = R[perm_i + 1];

    int ind0_j = R[perm_j];
    int ind1_j = R[perm_j + 1];

    int nnz_i = ind1_i - ind0_i;
    int nnz_j = ind1_j - ind0_j;

    float norm_ij = Norms[perm_i] + Norms[perm_j];

    int tmp_0, tmp_1, ind_jk, k, ret, testInd;

    ret = 0;
    testInd = 0;


    if (nnz_i > 0 && nnz_j >0){
      for (int pos_k = 0; pos_k < nnz_j; pos_k++){

        k = C[ind0_j + pos_k];

        // Binary search
        for (int l = nnz_i - ret; l > 1; l -= floorf(l/2.0)){
          tmp_0 = ret + l;
          tmp_1 = nnz_i - 1;
          testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
          ret = (C[ind0_i + testInd] <= k) ? testInd : ret;
        }

        tmp_0 = ret + 1;
        tmp_1 = nnz_i - 1;
        testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;

        ret = (C[testInd + ind0_i] <= k) ? testInd : ret;

        ind_jk = (C[ret + ind0_i] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;

      }
    }
    c_tmp = -2 * c_tmp + norm_ij;
    c_tmp = (c_tmp > 2e-6) ? sqrt(c_tmp) : 0.0;

    } else {
      c_tmp = 1e30;
    }


    // changed
    int gid_pt = leafId_local * ppl + block * partsize + rowId;
    int gid_pt_T = leafId_local * ppl + block * partsize + colId;
    int ind_knn = gid_pt * partsize + colId;
    int ind_knn_T = gid_pt_T * partsize + rowId;

    //KNN_dist_tmp[ind_knn] = (iternum >0 && colId == rowId) ? 1e30 : c_tmp;
    //if (G_Id[leafId_g * ppl + block * partsize + rowId] == 777478) printf("D[%d] = %.4f for %d , leaf = %d \n", colId , c_tmp, G_Id[leafId_g * ppl + block * partsize + colId], leafId_local); 
    //if (G_Id[leafId_g * ppl + block * partsize + colId] == 777478) printf("T D[%d] = %.4f for %d , leaf = %d \n", rowId , c_tmp, G_Id[leafId_g * ppl + block * partsize + rowId], leafId_local); 
    KNN_dist_tmp[ind_knn] = c_tmp;
    if (colId > rowId) KNN_dist_tmp[ind_knn_T] = c_tmp;
    /*
    for (int row_tmp = 0; row_tmp<rem_len; row_tmp++){
      for (int q = ind + rem_len; q < partsize; q += blockDim.x){
        gid_pt = leafId_local * ppl + block * partsize + row_tmp;
        ind_knn = gid_pt * partsize + q;
        if (ind_knn < leafId_local * ppl * partsize) KNN_dist_tmp[ind_knn] = 1e30;
      } 
    }
    */
    /*
    int max_write = gridDim.y * gridDim.z * ppl * partsize;
    int size_pad = (partsize - rem_len - 1) * rem_len;
    for (int elem = ind; elem < size_pad; elem += blockDim.x){
      int row_tmp = size_pad / (partsize - rem_len - 1);
      int col_tmp = size_pad - row_tmp * (partsize - rem_len - 1) + rem_len;
      //row_tmp += rem_len;
      gid_pt = leafId_local * ppl * partsize + block * partsize * partsize + row_tmp * partsize;
      ind_knn = gid_pt + col_tmp;
      if (G_Id[leafId_g * ppl + block * partsize + row_tmp] == 777478) printf("padding (%d, %d), write at %d , max = %d \n", row_tmp, col_tmp, ind_knn, max_write);
      if (ind_knn >= max_write) printf("leaf = %d , row_tmp = %d , col_tmp = %d , block = %d \n", leafId_local, row_tmp, col_tmp, block);
      KNN_dist_tmp[ind_knn] = 1e30;  
    }
    */

  }

}

__global__ void ComputeRecDists(int* R, int* C, float* V, int* G_Id, float* Norms, int k_nn, int ppl, int blockInd, float* d_knn_temp, int bl, int sizebleaves, int partsize) {


  __shared__ int SM[1000];
  
  int row_l = blockIdx.x;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;
  int j = threadIdx.x;
  
  int size_part = ppl - (partsize) * (blockInd+1); 
  


  int rowId_leaf = partsize * blockInd + row_l;
  int g_rowId_I = leafId_g * ppl + rowId_leaf;
  
  //changed 
  int perm_i = g_rowId_I;

  int ind0_i = R[perm_i];
  int ind1_i = R[perm_i+1];

  float norm_i = Norms[perm_i];
  int nnz_i = ind1_i - ind0_i;
  

  for (int n_i = j; n_i< nnz_i; n_i += blockDim.x) SM[n_i] = C[ind0_i + n_i];
   
  __syncthreads();


  for (int j_tmp = j; j_tmp < size_part; j_tmp += blockDim.x){

    int colId_leaf = partsize * (blockInd+1) + j_tmp;
      
    int g_rowId_J = leafId_g * ppl + colId_leaf;
      
    //changed 
    int perm_j = g_rowId_J;
        
    int ind0_j = R[perm_j];
    int ind1_j = R[perm_j+1];

    int nnz_j = ind1_j - ind0_j;

    float norm_ij = norm_i + Norms[perm_j];
        
    float c_tmp = 0.0;
    int tmp_0, tmp_1, ind_jk, k, ret, testInd;
      
    ret = 0;
    testInd = 0;
      
    // loop over the elements of j
  
    if (nnz_i >0 && nnz_j > 0 && colId_leaf < ppl){
      for (int pos_k = 0; pos_k < nnz_j; pos_k++){
          
        k = C[ind0_j + pos_k];
      
        // Binary search
      
        for (int l = nnz_i - ret; l > 1; l -= floorf(l/2.0)){
  
          tmp_0 = ret + l;
          tmp_1 = nnz_i - 1;
              
          testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
              
          ret = (SM[testInd] <= k) ? testInd : ret;
        }

        tmp_0 = ret + 1;
        tmp_1 = nnz_i - 1;
            
        testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;
        ret = (SM[testInd] <= k) ? testInd : ret;
            
        ind_jk = (SM[ret] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;

        } 
    }
         
    c_tmp = -2 * c_tmp + norm_ij;
    c_tmp = (c_tmp > 1e-8) ? sqrt(c_tmp) : 0.0;
        
        
    int size_tmp = size_part;
    int ind_tmp = leafId_local * partsize * size_tmp + row_l * size_tmp + colId_leaf - (partsize) * (blockInd+1);
    d_knn_temp[ind_tmp] = c_tmp;
    //if (G_Id[leafId_local * ppl + rowId_leaf] == 777478) printf("D[%d] = %.4f for %d , leaf = %d \n", j_tmp , c_tmp, G_Id[leafId_local * ppl + colId_leaf], leafId_local); 
  }


}
 



__global__ void MergeHoriz(float* KNN, int* KNN_Id, int k_nn, int ppl, int blockInd, float* d_temp_knn, int* sort_arr, int* sort_arr_part, int steps, int* G_Id, bool init, int bl, int sizebleaves, int partsize){

   
  __shared__ float SM_dist[SM_SIZE_2];
  __shared__ int SM_Id[SM_SIZE_2];



  int j = threadIdx.x;
  int row_l = blockIdx.x;
  int leafId_local = blockIdx.z * blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local; 

  int size_part = ppl - (partsize) * blockInd;
  int size_sort = 2 * blockDim.x;

  int rowId_leaf = partsize * blockInd + row_l;
  //int g_rowId_I = leaf_id_g * ppl + rowId_leaf;
  
  for (int n=j; n < SM_SIZE_2; n += blockDim.x){
    SM_dist[n] = 1e30; 
    SM_Id[n] = -1;
  }

  float tmp_f;
  int tmp_i;
  int ind_sort;
   
  int num_batches = size_part / (size_sort - k_nn);
  if (size_part > 0 && num_batches == 0) num_batches += 1;

  
  for (int col_batch = 0; col_batch < num_batches; col_batch++){
    for (int j_tmp = j; j_tmp < size_sort; j_tmp += blockDim.x){
      
      int colId_leaf = (col_batch == 0) ? partsize * (blockInd + 1) + j_tmp - k_nn : partsize * blockInd + col_batch * (size_sort - k_nn) + j_tmp;
      
      if (col_batch == 0 && j_tmp < k_nn){
        
        int ind_pt = G_Id[leafId_g * ppl + rowId_leaf];
        int ind_read = ind_pt * k_nn + j_tmp;
        SM_dist[j_tmp] = KNN[ind_read];
        SM_Id[j_tmp] = KNN_Id[ind_read];
        //if (ind_pt ==  777478) printf(" Neighbors , D[%d] = %.4f at %d \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);
      } else if (colId_leaf < ppl && j_tmp >= k_nn){

        int size_tmp = size_part - partsize;
        //int ind_tmp = leafId_local * partsize * size_tmp + row_l * size_tmp + colId_leaf - (k_nn) * (blockInd+1);
        int ind_tmp = leafId_local * partsize * size_tmp + row_l * size_tmp + colId_leaf - (partsize) * (blockInd+1);
        int g_colId_J = leafId_g * ppl + colId_leaf;
        
        SM_dist[j_tmp] = d_temp_knn[ind_tmp];
        SM_Id[j_tmp] = G_Id[g_colId_J];
        //if (G_Id[leafId_g * ppl + rowId_leaf] ==  777478) printf(" Dists tmp , D[%d] = %.4f at %d \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);
        //if (leafId_g == 0 && rowId_leaf == 782) printf(" Dists , D[%d] = %.4f at %d \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);
      }
      //if (init == 0 && rowId_leaf == 1149 && leaf_id_g == 0 && col_batch == 0) printf("D[%d] = %.4f , at %d  \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);
    }

    __syncthreads();
       
    for (int j_tmp = j; j_tmp < size_sort; j_tmp += blockDim.x) {

      if (j_tmp >= k_nn){
        int index = SM_Id[j_tmp];
        for (int ind_check = 0; ind_check < k_nn; ind_check++){
          if (index == SM_Id[ind_check]){
            SM_Id[j_tmp] = -1;
            SM_dist[j_tmp] = 1e30;
            break;
          }
        }
      }

    }
    __syncthreads();
    


    for (int step = 0; step < steps; step++){
    
      int j_tmp = j;
      ind_sort = step * 2 * blockDim.x + j_tmp;

      int tid = sort_arr[ind_sort];
      int ixj = sort_arr_part[ind_sort];

      int min_max = (1 & tid);
      int coupled_flag = (1 & ixj);

      tid = tid >> 1;
      ixj = ixj >> 1;

      if (coupled_flag == 1){

        ind_sort += blockDim.x;

        int tid_1 = sort_arr[ step * 2 * blockDim.x + j_tmp + blockDim.x];
        int ixj_1 = sort_arr_part[step * 2 * blockDim.x + j_tmp + blockDim.x];
        int min_max_1 = (1 & tid_1);


        tid_1 = tid_1 >> 1;
        ixj_1 = ixj_1 >> 1;


        if (min_max_1 == 1 && SM_dist[tid_1] > SM_dist[ixj_1]){

          tmp_f = SM_dist[tid_1];
          SM_dist[tid_1] = SM_dist[ixj_1];
          SM_dist[ixj_1] = tmp_f;

          tmp_i = SM_Id[tid_1];
          SM_Id[tid_1] = SM_Id[ixj_1];
          SM_Id[ixj_1] = tmp_i;

        }

        if (min_max_1 == 0 && SM_dist[tid] < SM_dist[ixj]){

          tmp_f = SM_dist[tid_1];
          SM_dist[tid_1] = SM_dist[ixj_1];
          SM_dist[ixj_1] = tmp_f;

          tmp_i = SM_Id[tid_1];
          SM_Id[tid_1] = SM_Id[ixj_1];
          SM_Id[ixj_1] = tmp_i;

        }

      }

      if (min_max == 1){
        if (SM_dist[tid] > SM_dist[ixj]){
          tmp_f = SM_dist[tid];
          SM_dist[tid] = SM_dist[ixj];
          SM_dist[ixj] = tmp_f;

          tmp_i = SM_Id[tid];
          SM_Id[tid] = SM_Id[ixj];
          SM_Id[ixj] = tmp_i;
        }
      } else {
        if (SM_dist[tid] < SM_dist[ixj]){
          tmp_f = SM_dist[tid];
          SM_dist[tid] = SM_dist[ixj];
          SM_dist[ixj] = tmp_f;

          tmp_i = SM_Id[tid];
          SM_Id[tid] = SM_Id[ixj];
          SM_Id[ixj] = tmp_i;
        }
      }
    
      __syncthreads();
    }


  }
  for (int j_tmp = j; j_tmp < k_nn; j_tmp += blockDim.x){ 
    if (j_tmp < k_nn){
      int ind_pt = leafId_g * ppl + rowId_leaf;
      int write_ind = G_Id[ind_pt] * k_nn + j_tmp;
      KNN[write_ind] = SM_dist[j_tmp];
      KNN_Id[write_ind] = SM_Id[j_tmp];
      //if (ind_pt ==  777478) printf(" Sorted Horiz Dists  , D[%d] = %.4f at %d \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);
      //if (leafId_g == 0 && rowId_leaf == 782) printf("Block = %d,  Sorted Neighbors , D[%d] = %.4f at %d , write at %d \n", blockInd, j_tmp, SM_dist[j_tmp], SM_Id[j_tmp], write_ind);
      //if (init == 0 && rowId_leaf == 1149 && leaf_id_g == 0) printf("sorted D[%d] = %.4f , at %d  \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);
    }
  } 
}


__global__ void MergeVer_v2(float* KNN, int* KNN_Id, int k_nn, int ppl, int blockInd, float* d_temp_knn, int* G_Id, bool init, int bl, int sizebleaves, int partsize){

  __shared__ float SM_dist[SM_SIZE_1];
  __shared__ int SM_Id[SM_SIZE_1];


  int j = threadIdx.x;
  float tmp_f;
  int tmp_i;

  int col = blockIdx.x;
  int leafId_local = blockIdx.z* blockDim.y + blockIdx.y;
  int leafId_g = bl * sizebleaves + leafId_local;

  int colId_leaf = (init) ? col : col + partsize * (blockInd + 1);
  int size_part = (init) ? ppl : ppl - (blockInd + 1) * partsize;


  //float norm_i = Norms[leafId_g * ppl + colId_leaf];

  int ind_pt_knn = leafId_g * ppl + colId_leaf;
  int ind_pt_knn_g = G_Id[ind_pt_knn];

  for (int ind = j; ind < 2 * partsize; ind += blockDim.x){
    SM_dist[ind] = 1e30;
    SM_Id[ind] = -1;
  }
  __syncthreads();


  for (int j_tmp = j; j_tmp < partsize; j_tmp += blockDim.x){

    //int ind_tmp = (init) ? leafId_local * ppl * partsize + col * partsize + j_tmp : leafId_local * partsize * size_part + j_tmp + col * partsize;
    int ind_tmp = (init) ? leafId_local * ppl * partsize + col * partsize + j_tmp : leafId_local * partsize * size_part + j_tmp * size_part + col;

    int block = col / partsize;
    int M = ppl * gridDim.y * gridDim.z;
    
    int rowId_g = (init) ? leafId_g * ppl + block * partsize + j_tmp : leafId_g * ppl + partsize * blockInd + j_tmp;

    int Max_blocks = ppl / partsize;
    int rem_len = ppl - Max_blocks * partsize;

    SM_Id[j_tmp] = (rowId_g < M) ? G_Id[rowId_g] : -1;
    SM_dist[j_tmp] = (Max_blocks == block && j_tmp >= rem_len && init == 1) ? 1e30 : d_temp_knn[ind_tmp];
    //tmp_f = (rowId_g < M) ? -2 * d_temp_knn[ind_tmp] + norm_i + Norms[rowId_g] : 1e30;
    //tmp_f = d_temp_knn[ind_tmp] + norm_i + Norms[rowId_g];

    /*
    if (init){
      tmp_f = (block * partsize + j_tmp < ppl) ? tmp_f : 1e30;
    }

    SM_dist[j_tmp] = (tmp_f > 0.0) ? sqrt(tmp_f) : 0.0;
    */
    //if (ind_pt_knn_g == 777478 && blockInd == 0 && init == 1) printf("Ver read tmp D[%d] = %.4f at %d \n", j_tmp, SM_dist[j_tmp], SM_Id[j_tmp]);

    int ind_knn = ind_pt_knn_g * k_nn + j_tmp;
    if (j_tmp < k_nn){
      SM_dist[j_tmp + partsize] = KNN[ind_knn];
      SM_Id[j_tmp + partsize] = KNN_Id[ind_knn];
      //if (ind_pt_knn_g == 777478 && blockInd == 0 && init == 1) printf("Ver read Nhbd D[%d] = %.4f at %d \n", j_tmp + partsize, SM_dist[j_tmp + partsize], SM_Id[j_tmp + partsize]);
     
    } else {
      SM_dist[j_tmp + partsize] = 1e30;
      SM_Id[j_tmp + partsize] = -1;
    }


  }

  __syncthreads();


  for (int j_tmp = j; j_tmp < partsize; j_tmp += blockDim.x){
    int index = SM_Id[j_tmp];
    for (int ind_check = 0; ind_check < k_nn; ind_check++){
      if (index == SM_Id[ind_check + partsize]){
        SM_dist[j_tmp] = 1e30;
        SM_Id[j_tmp] = -1;
        break;
      }
    }
  }
  __syncthreads();



  //int size_sort = 2 * partsize;


  for (int g = 2; g <= 2 * blockDim.x; g *= 2){
    for (int l = g/2; l > 0; l /= 2){
      int tid = j;
      int ixj = tid ^ l;
      if (ixj < tid){
        tid += blockDim.x;
        ixj = tid ^ l;
        if (ixj < tid){
          tid = ixj;
          ixj = tid ^ l;
        }
      }

      if ((tid & g) == 0){

        if (SM_dist[tid] > SM_dist[ixj]){
          tmp_f = SM_dist[ixj];
          SM_dist[ixj] = SM_dist[tid];
          SM_dist[tid] = tmp_f;
          tmp_i = SM_Id[ixj];
          SM_Id[ixj] = SM_Id[tid];
          SM_Id[tid] = tmp_i;
        }
      } else {
        if (SM_dist[tid] < SM_dist[ixj]){
          tmp_f = SM_dist[ixj];
          SM_dist[ixj] = SM_dist[tid];
          SM_dist[tid] = tmp_f;
          tmp_i = SM_Id[ixj];
          SM_Id[ixj] = SM_Id[tid];
          SM_Id[tid] = tmp_i;
        }
      }
      __syncthreads();
    }
  }

  if (j < k_nn){
    int ind_pt_knn = leafId_g * ppl + colId_leaf;
    int ind_pt_knn_g = G_Id[ind_pt_knn];
    int ind_knn = ind_pt_knn_g * k_nn + j;
    //if (ind_pt_knn_g == 777478 && blockInd == 0) printf("Sorted Ver D[%d] = %.4f at %d \n", j, SM_dist[j], SM_Id[j]);
    KNN[ind_knn] = SM_dist[j];
    KNN_Id[ind_knn] = SM_Id[j];
    //if (leafId_g == 0 && colId_leaf == 782) printf("BlockInd = %d , Sorted Ver D[%d] = %.4f , at %d , write at %d \n", blockInd, j, SM_dist[j], SM_Id[j], ind_knn); 
  }


}


void PrecompSortIdsSparse(int* d_arr, int* d_arr_part, int N_true, int N_pow2, int steps, int copy_size){

  
  
  int min_max, elem, coupled_elem;
  int loc_len = ceil(N_true/2);
  int* tracker;
  tracker = (int *)malloc(sizeof(int) * N_pow2);
  
  for (int i = 0; i < N_pow2; i ++) tracker[i] = i;

  int step = 0;

  int *arr, *arr_part;
  arr = (int *)malloc(sizeof(int) * copy_size); 
  arr_part = (int *)malloc(sizeof(int) * copy_size);

  
  memset(arr, 0, sizeof(int) * copy_size);
  memset(arr_part, 0, sizeof(int) * copy_size);
 
  int first_pair = 1;
  int prev_elem = 0;
  int tmp2;
  for (int g = 2; g <= N_pow2; g *= 2){
    for (int l = g/2; l > 0; l /= 2){
      elem = 0;
      for (int i = 0; i < N_pow2; i++){
        int ixj = i ^ l;

        if (tracker[ixj] >= N_true && tracker[i] >= N_true) continue;

        if (ixj > i){

          min_max = ((i&g) == 0 ) ? 1 : 0;

          coupled_elem = 0;

          int write_loc = elem;
          
          if (tracker[ixj] >= N_true || tracker[i] >= N_true) {
            coupled_elem = 1;
            if (min_max == 0 && tracker[ixj] >= N_true) {
              tmp2 = tracker[ixj];
              tracker[ixj] = tracker[i];
              tracker[i] = tmp2;
            }
            if (min_max == 1 && tracker[i] >= N_true) {
              tmp2 = tracker[ixj];
              tracker[ixj] = tracker[i];
              tracker[i] = tmp2;
            }
            if (first_pair == 1){
              prev_elem = elem;
              write_loc = elem;
              first_pair = 0;
            } else {
              write_loc = prev_elem + loc_len;
              first_pair = 1;
              elem++;
            }
            
            arr[step * N_true + write_loc] = (i << 1) + min_max;
            arr_part[step * N_true + write_loc] = (ixj << 1) + coupled_elem;

          } else {
            write_loc = elem;
            arr[step * N_true + write_loc] = (i << 1) + min_max;
            arr_part[step * N_true + write_loc] = (ixj << 1) + coupled_elem;
            elem++;
          }




        }
      }    

      step++;
    }
  }
  checkCudaErrors(hipMemcpy(d_arr, arr, sizeof(int)*copy_size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_arr_part, arr_part, sizeof(int)*copy_size, hipMemcpyHostToDevice));
  checkCudaErrors(hipDeviceSynchronize());
  
  free(arr);
  free(arr_part); 
  free(tracker); 
  
}



void sfi_leafknn(int *d_R, int *d_C, float *d_V, int *d_GId, int M, int leaves, int k, float *d_knn, int *d_knn_Id){


  printf("----------------------------- Starting SpIF knn ----------------------------- \n\n");

  float dt1, dt2, dt3, dt4, dt5, dt6, dt7, dt8, dt9, dt_tmp;
  hipEvent_t t0;
  hipEvent_t t1;
  hipEvent_t t2;
  hipEvent_t t3;
  hipEvent_t t4;
  hipEvent_t t5;
  hipEvent_t t6;
  hipEvent_t t7;
  hipEvent_t t8;
  hipEvent_t t9;

  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));
  checkCudaErrors(hipEventCreate(&t2));
  checkCudaErrors(hipEventCreate(&t3));
  checkCudaErrors(hipEventCreate(&t4));
  checkCudaErrors(hipEventCreate(&t5));
  checkCudaErrors(hipEventCreate(&t6));
  checkCudaErrors(hipEventCreate(&t7));
  checkCudaErrors(hipEventCreate(&t8));
  checkCudaErrors(hipEventCreate(&t9));

  checkCudaErrors(hipEventRecord(t0, 0));

  int verbose = 1;
	if (verbose) printf("----------------------------- Start of sfiknn ----------------------------- \n\n");

  size_t free, total, m1, m2, m3;

  int ppl = M/leaves;

  int partsize = (k > 32) ? k : 32;
  
  hipMemGetInfo(&free, &total);
  printf(" Available Memory : %.4f GB from %.4f \n", free/1e9, total/1e9);
  

  size_t size_req = sizeof(float) * partsize * M;
  int counter=0;
	while (size_req < free && partsize < 256 && counter < 6) {
    counter++;
    size_req *= 2;
    partsize *= 2;
    //printf("partsize = %d,  free = %.4f , size_req = %.4f,\n", partsize, free/1e9, size_req/1e9);
  }
  partsize /= 2;
  partsize = (partsize > ppl) ? ppl : partsize;
  partsize = (partsize < k) ? k : partsize;  


  int num_blocks_tri = ppl / partsize;
  int rem_len = (num_blocks_tri * partsize < ppl) ? ppl - num_blocks_tri * partsize : 0;
 

  int t_b = (ppl > SM_SIZE_1) ? SM_SIZE_1 : ppl;

  int num_splits = 1;
  while (leaves > num_splits * 65535) num_splits *= 2;

  int batch_leaves_1 = (leaves > 64000) ? leaves / num_splits : leaves;
  int batch_leaves_2 = (leaves > 64000) ? num_splits : 1;

  


  float *d_Norms;


  //int size_tri = (k > 32) ? 32 : k;
  int size_tri = partsize;
 
  int blockDim_tri = size_tri * (size_tri + 1)/2;
  if (blockDim_tri > SM_SIZE_1) blockDim_tri = SM_SIZE_1;

  int size_tri_last = (rem_len > 32) ? 32 : rem_len;
  int blockDim_tri_last = size_tri_last * (size_tri_last + 1)/2;
  if (blockDim_tri_last > SM_SIZE_1) blockDim_tri_last = SM_SIZE_1;

  dim3 BlockDistTri(blockDim_tri, 1, 1);
  dim3 GridDistTri(num_blocks_tri, batch_leaves_1, batch_leaves_2);

  dim3 BlockDistTri_last(blockDim_tri_last, 1, 1);
  dim3 GridDistTri_last(1, batch_leaves_1, batch_leaves_2);
  
  dim3 BlockNorm(t_b, 1, 1);
  dim3 GridNorm(1, batch_leaves_1, batch_leaves_2);

  dim3 GridDistRec(partsize, batch_leaves_1, batch_leaves_2);
  
  dim3 BlockMergeVer(partsize, 1, 1);
  
  dim3 GridMergeHoriz(partsize, batch_leaves_1, batch_leaves_2);


  printf("=======================\n");
  printf(" Num points = %d \n", M);
  printf(" pt/leaf = %d \n", ppl);
  printf(" Leaves = %d \n", leaves);
  printf(" K = %d \n", k);
  printf(" PartSize = %d \n", partsize);

  printf(" BlockThreads Norms = (%d , %d, %d) \n", BlockNorm.x, BlockNorm.y, BlockNorm.z);
  printf(" dim GridThreads Norms = (%d , %d, %d) \n", GridNorm.x, GridNorm.y, GridNorm.z);
  printf(" dim GridThreads MergeHoriz = (%d , %d, %d) \n", GridMergeHoriz.x, GridMergeHoriz.y, GridMergeHoriz.z);
  printf(" dim BlockThreads MergeVer = (%d , %d, %d) \n", BlockMergeVer.x, BlockMergeVer.y, BlockMergeVer.z);
  printf(" dim BlockThreads Diagonal Distances = (%d , %d, %d) \n", BlockDistTri.x, BlockDistTri.y, BlockDistTri.z);
  printf(" dim GridThreads Diagonal Distances = (%d , %d, %d) \n", GridDistTri.x, GridDistTri.y, GridDistTri.z);
  printf(" dim GridThreads RecDistances = (%d, %d, %d) \n", GridDistRec.x, GridDistRec.y, GridDistRec.z);

  /*
  if (verbose) printf(" dim GridThreads IterativePart = (%d , %d, %d) \n", k, batch_leaves_1, batch_leaves_2);
  if (verbose) printf(" dim BlockThreads  Norms = (%d , %d, %d) \n", t_b, 1, 1);
  if (verbose) printf(" dim GridThreads Norms = (%d , %d, %d) \n", 1, batch_leaves_1, batch_leaves_2);
  */



  int *d_arr, *d_arr_part;
  float SM_SIZE_2_f = SM_SIZE_2;
  int n_s = log2(SM_SIZE_2_f) *(log2(SM_SIZE_2_f)+1) /2;

  int copy_size = (ppl) * n_s;
  float tmp = 2*partsize;
  int n_s_v = log2(tmp) * (log2(tmp)+1)/2;

  //size_t free, total, m1, m2, m3;

  //int *d_R, *d_GId, *d_C, *d_knn_Id;
  //float *d_V, *d_knn;  
    

  hipMemGetInfo(&free, &total);
  checkCudaErrors(hipMalloc((void **) &d_arr, sizeof(int) * copy_size));
  checkCudaErrors(hipMalloc((void **) &d_arr_part, sizeof(int) * copy_size));

  checkCudaErrors(hipMemset(d_arr, 0, sizeof(int) * copy_size));
  checkCudaErrors(hipMemset(d_arr_part, 0, sizeof(int) * copy_size));
  hipMemGetInfo(&m1, &total);

  checkCudaErrors(hipEventRecord(t1, 0));
  checkCudaErrors(hipMalloc((void **) &d_Norms, sizeof(float) * M));

  hipMemGetInfo(&m2, &total);

  size_t size_tmp = sizeof(float) * M * partsize;
  printf(" Needed %.4f GB , avail : %.4f \n", size_tmp/1e9, m2/1e9);
  float tmp2 = ceil(size_tmp/m2);
  
  int bleaves = (size_tmp > m2) ? log2(tmp2) : 0;
  int numbleaves = 1 << bleaves;
  int sizebleaves = leaves / numbleaves;

  printf(" Num BatchLeaves = %d \n", numbleaves);
  printf(" Size BatchLeaves = %d \n", sizebleaves);
  printf("======================\n");


  float * d_temp_knn;
  checkCudaErrors(hipMalloc((void **) &d_temp_knn, sizeof(float) * M * partsize));
  hipMemGetInfo(&m3, &total);



  checkCudaErrors(hipEventRecord(t2, 0));


  int steps;

  ComputeNorms <<< GridNorm, BlockNorm >>>(d_R, d_C, d_V, d_GId, d_Norms, ppl);
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipEventRecord(t3, 0));

  for (int bl = 0; bl < numbleaves; bl++){

  
    ComputeTriDists <<< GridDistTri, BlockDistTri >>>(d_R, d_C, d_V, d_GId, d_Norms, k, d_temp_knn, ppl, bl, sizebleaves, partsize);
    checkCudaErrors(hipDeviceSynchronize());
  
    if (rem_len > 0) {
      ComputeTriDists_last <<< GridDistTri_last, BlockDistTri_last >>>(d_R, d_C, d_V, d_GId, d_Norms, k, d_temp_knn, ppl,  rem_len, num_blocks_tri, bl, sizebleaves, partsize);
      checkCudaErrors(hipDeviceSynchronize());
    }

    int size_v = ppl;
    dim3 GridMergeVer(size_v, batch_leaves_1, batch_leaves_2);    
		//MergeVer_v2 <<< GridMergeVer, BlockMergeVer >>> (d_knn, d_knn_Id, k, ppl, 0, d_temp_knn, d_arr_v, d_arr_part_v, n_s_v, d_GId, true, M);
		MergeVer_v2 <<< GridMergeVer, BlockMergeVer >>> (d_knn, d_knn_Id, k, ppl, 0, d_temp_knn, d_GId, true, bl, sizebleaves, partsize );
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipEventRecord(t4, 0));
  
		int num_iters = (rem_len > 0) ? num_blocks_tri : num_blocks_tri - 1;
 
		dt5 = 0.0;
		dt6 = 0.0;
		dt7 = 0.0;
 
		for (int blockInd = 0; blockInd < num_iters; blockInd++){

			checkCudaErrors(hipEventRecord(t5, 0));
			checkCudaErrors(hipEventSynchronize(t5));
			
			int size_part = ppl - blockInd *k;
			int size_sort = size_part;

			while (size_sort > SM_SIZE_2) size_sort = ceil((size_sort+k)/2);
		 
			float tmp = size_sort/2.0;
			int blocksize = ceil(tmp);
			float tmp_f = 2 * blocksize;
			int N_pow2 = pow(2, ceil(log2(tmp_f)));
			tmp_f = N_pow2;
			steps = log2(tmp_f) * (log2(tmp_f) +1)/2;  
			int real_size = 2 * blocksize;


			int blocksize_dist = size_part - k;
			while(blocksize_dist > SM_SIZE_1) blocksize_dist = ceil(blocksize_dist / 2.0);


			dim3 BlockDistRec( blocksize_dist, 1, 1);
			dim3 BlockMergeHoriz( blocksize, 1, 1);

			int size_v2 = ppl - (blockInd + 1) * partsize;
			dim3 GridMergeVer(size_v2, batch_leaves_1, batch_leaves_2);

			ComputeRecDists <<< GridDistRec, BlockDistRec >>> (d_R, d_C, d_V, d_GId, d_Norms, k, ppl, blockInd, d_temp_knn, bl, sizebleaves, partsize);
			checkCudaErrors(hipDeviceSynchronize());

			checkCudaErrors(hipEventRecord(t6, 0));
			checkCudaErrors(hipEventSynchronize(t6));
			
			PrecompSortIdsSparse(d_arr, d_arr_part, real_size, N_pow2, steps, copy_size);
			
			MergeHoriz <<< GridMergeHoriz, BlockMergeHoriz >>> (d_knn, d_knn_Id, k, ppl, blockInd, d_temp_knn, d_arr, d_arr_part, steps, d_GId, false, bl, sizebleaves, partsize); 
			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipEventRecord(t7, 0));
			checkCudaErrors(hipEventSynchronize(t7));
			
      MergeVer_v2 <<< GridMergeVer, BlockMergeVer >>> (d_knn, d_knn_Id, k, ppl, blockInd, d_temp_knn, d_GId, false, bl, sizebleaves, partsize);
			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipEventRecord(t8, 0));
			checkCudaErrors(hipEventSynchronize(t8));
			checkCudaErrors(hipEventElapsedTime(&dt_tmp, t5, t6));
			dt5 += dt_tmp; 
			checkCudaErrors(hipEventElapsedTime(&dt_tmp, t6, t7)); 
			dt6 += dt_tmp; 
			checkCudaErrors(hipEventElapsedTime(&dt_tmp, t7, t8)); 
			dt7 += dt_tmp; 

		}
  
  }

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(t9, 0));
  checkCudaErrors(hipEventSynchronize(t9));
  checkCudaErrors(hipEventElapsedTime(&dt1, t0, t1));
  checkCudaErrors(hipEventElapsedTime(&dt2, t1, t2));
  checkCudaErrors(hipEventElapsedTime(&dt3, t2, t3));
  checkCudaErrors(hipEventElapsedTime(&dt4, t3, t4));
  checkCudaErrors(hipEventElapsedTime(&dt8, t4, t9));
  checkCudaErrors(hipEventElapsedTime(&dt9, t0, t9));

  //checkCudaErrors(hipMemcpy(knn, d_knn, sizeof(float) * M * k, hipMemcpyDeviceToHost));
  //checkCudaErrors(hipMemcpy(knn_Id, d_knn_Id, sizeof(int) * M * k, hipMemcpyDeviceToHost));
  

  checkCudaErrors(hipFree(d_Norms));
  checkCudaErrors(hipFree(d_temp_knn));
  checkCudaErrors(hipFree(d_arr_part));
  checkCudaErrors(hipFree(d_arr));

  checkCudaErrors(hipEventDestroy(t0));
  checkCudaErrors(hipEventDestroy(t1));
  checkCudaErrors(hipEventDestroy(t2));
  checkCudaErrors(hipEventDestroy(t3));
  checkCudaErrors(hipEventDestroy(t4));
  checkCudaErrors(hipEventDestroy(t5));
  checkCudaErrors(hipEventDestroy(t6));
  checkCudaErrors(hipEventDestroy(t7));
  checkCudaErrors(hipEventDestroy(t8));
  checkCudaErrors(hipEventDestroy(t9));
  
  printf("--------------- Timings ----------------\n");
  printf("\t Memory allocation :%.4f (%.4f %%) \n", dt1/1000, dt1/dt9);
  printf("\t Precomp sortId (vertical)  = %.4f (%.4f %%) \n", dt2/1000, dt2/dt9);
  printf("\t Computing norms = %.4f (%.4f %%) \n", dt3/1000, dt3/dt9);
  printf("\t Diagonal part = %.4f (%.4f %%) \n", dt4/1000, dt4/dt9);
  printf("\t Iterative part = %.4f (%.4f %%) \n", dt8/1000, dt8/dt9);
  printf("\t\t	 Compute Dists = %.4f (%.4f %%) \n", dt5/1000, dt5/dt9);
  printf("\t\t	 Merge Horizontally = %.4f (%.4f %%) \n", dt6/1000, dt6/dt9);
  printf("\t\t	 Merge Vertically  = %.4f (%.4f %%) \n", dt7/1000, dt7/dt9);
  printf("\t Total = %.4f \n", dt9/1000);
  printf("--------------- Memory usage ----------------\n");
  printf("\t Storing norms = %.4f GB \n", (m1-m2)/1e9);
  printf("\t Precomputing the sort indices = %.4f GB \n", (free-m1)/1e9);
	printf("\t Temporary storage = %.4f GB \n", (m2-m3)/1e9); 
  printf("----------------------------- End of leaf-knn -----------------------------\n\n");
  
}











