#include "hip/hip_runtime.h"

#include <stdio.h> 
#include <stdlib.h>
//#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "knn_seq.cpp"
#include <limits.h>


__global__ void compute_norm(int* R, int* C, float* V, int* G_Id, float* Norms, int ppl) {

  int row = threadIdx.x;
  int leaf_id_g = blockIdx.x;
  
  int g_rowId = leaf_id_g * ppl + row;

  int g_Id = G_Id[g_rowId]; 
  int ind0_i = R[g_Id];
 
  int nnz = R[g_Id + 1] - ind0_i;
  float norm_i = 0.0;
  
  for (int n_i = 0; n_i < nnz; n_i++) norm_i += V[ind0_i + n_i] * V[ind0_i + n_i];
  int ind_write = blockIdx.x * ppl + row;
  Norms[ind_write] = norm_i;

}

__global__ void knn_kernel_tri(int* R, int* C, float* V, int* G_Id, float* Norms , int k_nn, float* KNN_dist, int* KNN_Id, int ppl, int max_nnz, int m) {

    __shared__ int SM[4096];
    __shared__ float SM_dist[2048];
    
      int ind = threadIdx.x;
      int leaf_id_g = blockIdx.y;  
      int block = blockIdx.x;

      
      int i = ind / m;
      int j = ind - i * m;

      int g_rowId = leaf_id_g * ppl + block * m + i;
      int g_colId = leaf_id_g * ppl + block * m + j;

      int perm_i = G_Id[g_rowId];
      int perm_j = G_Id[g_colId];

      int ind0_i = R[perm_i];
      int ind1_i = R[perm_i + 1];

      int ind0_j = R[perm_j];
      int ind1_j = R[perm_j + 1];
     
      int nnz_i = ind1_i - ind0_i;
      int nnz_j = ind1_j - ind0_j;
     
      
      float norm_ij = 0.0;
      if (nnz_i > 256 || nnz_j > 256) printf("Exceeding the max nnz/pt \n");  

      
      norm_ij += Norms[g_rowId] + Norms[g_colId];

      int shift_i = max_nnz * i;
       
      // TODO: this reading should be balanced among the threads
      // above problem is fixed
      
      for (int n_i = j; n_i < nnz_i; n_i += m) SM[shift_i + n_i] = C[ind0_i + n_i];
      __syncthreads();
      
      float c_tmp = 0.0;
      int tmp_0, tmp_1, ind_jk, k, ret, testInd;
 
      ret = 0;
      testInd = 0;
      if (j >= i){
      // loop over the elements of j
      for (int pos_k = 0; pos_k < nnz_j; pos_k++){
        
        k = SM[max_nnz * j + pos_k];

        // Binary search
        for (int l = nnz_i - ret; l > 1; l /= 2){
          tmp_0 = ret + l;
          tmp_1 = nnz_i - 1;
          testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
          ret = (SM[testInd + shift_i] <= k) ? testInd : ret;
        }
        
        tmp_0 = ret + 1;
        tmp_1 = nnz_i - 1;
        testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;
        ret = (SM[testInd + shift_i] <= k) ? testInd : ret; 
        ind_jk = (SM[ret + shift_i] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;
      }
      c_tmp = -2 * c_tmp + norm_ij;
      c_tmp = ( c_tmp > 0) ? sqrt(c_tmp) : 0.0;
      }
      __syncthreads();

      if (j >= i) SM_dist[i * 2 * m + j] = c_tmp;
      if (j >= i) SM[i * 2 * m + j] = G_Id[leaf_id_g * ppl + block * m + j];
      
      if (j > i) SM_dist[j * 2 * m + i] = c_tmp;
      if (j > i) SM[j * 2 * m + i] = G_Id[leaf_id_g * ppl + block * m + i];
       
           
      __syncthreads();
           
      // TODO : should fix the issue for the initial value
      //int ind_knn = leaf_id_g * ppl * k_nn + (block * m + row_read) * k_nn + col_read;
      //SM_dist[row_read * 2 * m + col_read + m] = (col_read < k_nn) ? KNN_dist[ind_knn] : 1e30;
      //SM[row_read * 2 * m + col_read + m] = (col_read < k_nn) ? KNN_Id[ind_knn] :  0;
      
      SM_dist[i * 2 * m + j + m] = (j < k_nn) ? 1e30 : 1e30;
      SM[i * 2 * m + j + m] = (j < k_nn) ? 0 :  0;
      
      //int w = block * m + row_read;
      __syncthreads();
      

      // bitonic sort 

            

      for (int batch = 0; batch < 2; batch ++){
       
      int row = ind / (2 * m);
      int col = ind - row * 2 * m;

      //if (leaf_id_g == 1000 && w == 1000 ) printf("val = %.4f , id = %d \n", SM_dist[col], SM[col]);
      float tmp_f; 
      int tmp_i;
      int size = 2 *m;
      int col_tmp = (row + batch * m / 2) * 2 * m + col;
      for (int g = 2; g <= size; g *= 2){
        for (int l = g/2; l > 0; l /= 2){

          int ixj = col ^ l;
          int ixj_tmp = (row + batch * m / 2) * 2 * m + ixj;

          if (ixj > col){
            if(( col & g) == 0){
              if (SM_dist[col_tmp] > SM_dist[ixj_tmp]){
                
                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[col_tmp];
                SM_dist[col_tmp] = tmp_f;
                
                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[col_tmp];
                SM[col_tmp] = tmp_i;
              }
           } else {
              if (SM_dist[col_tmp] < SM_dist[ixj_tmp]){
                
                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[col_tmp];
                SM_dist[col_tmp] = tmp_f;
                
                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[col_tmp];
                SM[col_tmp] = tmp_i;
              }
           }
         }
       __syncthreads();
       }
     }

   } 
   
   int ind_knn = leaf_id_g * ppl * k_nn + (block * m + i) * k_nn + j;
   if (j < k_nn){
     KNN_dist[ind_knn] = SM_dist[ i * 2 * m + j];
     KNN_Id[ind_knn] = SM[i * 2 * m + j]; 

   }

}





__global__ void knn_kernel_sq(int* R, int* C, float* V, int* G_Id, float* Norms , int k_nn, float* KNN_dist, int* KNN_Id, int ppl, int max_nnz, int m, int blockInd) {

   // square partitions

    __shared__ int SM[4096];
    __shared__ float SM_dist[2048];

   int i = threadIdx.x;
   int j = threadIdx.y;

   //block = blockIdx.x;
   float tmp = -8 * blockInd + 4 * (m-1) * (m) - 7;
   int b_i = sqrt(tmp) / 2.0 - 0.5;
   b_i = m - 2 - b_i;
   int b_j = blockInd + b_i - (m-1) * (m)/2 + (m-1 - b_i) * (( m-1 - b_i) + 1)/2;
  
   //b_i++;
   b_j++; 
 
   int leaf_id_g = blockIdx.y;
   
   //if (leaf_id_g == 1000 && i == 0 && j == 0) printf("ind %d -> (%d, %d) \n", blockInd, b_i, b_j);

   

   int g_rowId_I = leaf_id_g * ppl + b_i * m + i;
   int g_rowId_J = leaf_id_g * ppl + b_j * m + j;

   int perm_i = G_Id[g_rowId_I];
   int perm_j = G_Id[g_rowId_J];

   int ind0_i = R[perm_i];
   int ind1_i = R[perm_i+1];

   int ind0_j = R[perm_j];
   int ind1_j = R[perm_j+1];

   int nnz_i = ind1_i - ind0_i;
   int nnz_j = ind1_j - ind0_j;

   
   
   float norm_ij = Norms[g_rowId_I] + Norms[g_rowId_J];

   int shift_i = max_nnz * i;
   //int shift_j = max_nnz * j;

   //__shared__ int SM_col[4096];

   for (int n_i = j; n_i< nnz_i; n_i += m) SM[shift_i + n_i] = C[ind0_i + n_i];
   //for (int n_j = i; n_j< nnz_j; n_j += m) SM_col[shift_j + n_j] = C[ind0_j + n_j];

   __syncthreads();

    
		float c_tmp = 0.0;
		int tmp_0, tmp_1, ind_jk, k, ret, testInd;

		ret = 0;
		testInd = 0;

		// loop over the elements of j
		for (int pos_k = 0; pos_k < nnz_j; pos_k++){
			//k = SM_col[max_nnz * j + pos_k];
      k = C[ind0_j + pos_k];
			// Binary search
			for (int l = nnz_i - ret; l > 1; l /= 2){
				tmp_0 = ret + l;
				tmp_1 = nnz_i - 1;
				testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
				ret = (SM[testInd + shift_i] <= k) ? testInd : ret;
			}

			tmp_0 = ret + 1;
			tmp_1 = nnz_i - 1;
			testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;
			ret = (SM[testInd + shift_i] <= k) ? testInd : ret;
			ind_jk = (SM[ret + shift_i] == k) ? ret : -1;
			c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;
    }
		c_tmp = -2 * c_tmp + norm_ij;
		c_tmp = ( c_tmp > 0) ? sqrt(c_tmp) : 0.0;

    __syncthreads();
      
    
    // horizontal merge

    SM_dist[i * 2*m + j] = c_tmp;
    SM[i * 2*m + j] =  G_Id[g_rowId_J];

    SM_dist[i * 2 * m + j + m] = (j < k_nn) ? KNN_dist[leaf_id_g * ppl * k_nn + (b_i * m + i) * k_nn + j] : 1e30;
    SM[i * 2 * m + j + m] = (j < k_nn) ? KNN_Id[leaf_id_g * ppl * k_nn + (b_i * m + i) * k_nn + j] : 0;

    
    // bitonic sort

    float tmp_f;
    int tmp_i;
    int size = 2 * m;
    int ind = i * m + j;
    int row = ind / (2 * m);
    int col = ind - row * 2 * m;
    for (int batch = 0; batch < 2; batch ++){
         
      int j_tmp = (row + batch * m /2)  * 2 * m + col;
      
      for (int g = 2; g <= size; g *= 2){
        for (int l = g/2; l > 0; l /= 2){

          int ixj = col ^ l;
          int ixj_tmp = (row + batch * m / 2) * 2*m + ixj;

          if (ixj > j){
            if(( j & g) == 0){
              if (SM_dist[j_tmp] > SM_dist[ixj_tmp]){

                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[j_tmp];
                SM_dist[j_tmp] = tmp_f;

                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[j_tmp];
                SM[j_tmp] = tmp_i;
              }
           } else {
              if (SM_dist[j_tmp] < SM_dist[ixj_tmp]){

                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[j_tmp];
                SM_dist[j_tmp] = tmp_f;

                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[j_tmp];
                SM[j_tmp] = tmp_i;
              }
           }
         }
       __syncthreads();
       }
     }
   
   if (col < k_nn){
     int ind_write = leaf_id_g * ppl * k_nn + (b_i * m + row + batch * m /2)* k_nn + col;
     KNN_dist[ind_write] = SM_dist[(row + batch * m /2) * 2 *m + col];
     KNN_Id[ind_write] = SM[(row + batch * m /2) * 2 *m + col];
     //if (leaf_id_g == 1000 && b_i == 30 && b_j == 30 && i == 31) printf(" val = %.4f , in = %d \n", KNN_dist[ind_write], j);
   } 
   

   }
   
   __syncthreads();
  
    /*
    // vertical merge

    SM_dist[j * 2*m + i] = c_tmp;
    SM[j * 2*m + i] =  G_Id[g_rowId_I];

    SM_dist[j * 2 * m + i + m] = (i < k_nn) ? KNN_dist[leaf_id_g * ppl * k_nn + (b_j * m + j) * k_nn + i] : 1e30;
    SM[j * 2 * m + i + m] = (i < k_nn) ? KNN_Id[leaf_id_g * ppl * k_nn + (b_j * m + j) * k_nn + i] : 0;

   
    // bitonic sort

    //float tmp_f;
    //int tmp_i;
    //int size = 2 * m;
    ind = j * m + i;
    row = ind / (2 * m);
    col = ind - row * 2 * m;
    for (int batch = 0; batch < 2; batch ++){
         
      int j_tmp = (row + batch * m /2)  * 2 * m + col;
      
      for (int g = 2; g <= size; g *= 2){
        for (int l = g/2; l > 0; l /= 2){

          int ixj = col ^ l;
          int ixj_tmp = (row + batch * m / 2) * 2*m + ixj;

          if (ixj > j){
            if(( j & g) == 0){
              if (SM_dist[j_tmp] > SM_dist[ixj_tmp]){

                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[j_tmp];
                SM_dist[j_tmp] = tmp_f;

                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[j_tmp];
                SM[j_tmp] = tmp_i;
              }
           } else {
              if (SM_dist[j_tmp] < SM_dist[ixj_tmp]){

                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[j_tmp];
                SM_dist[j_tmp] = tmp_f;

                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[j_tmp];
                SM[j_tmp] = tmp_i;
              }
           }
         }
       __syncthreads();
       }
     }
   
   if (col < k_nn){
     int ind_write = leaf_id_g * ppl * k_nn + (b_j * m + row + batch * m /2) * k_nn + col;
     KNN_dist[ind_write] = SM_dist[(row + batch * m /2) * 2 *m + col];
     KNN_Id[ind_write] = SM[(row + batch * m /2) * 2 *m + col];

   }


   }


   */



 
   
    
}




void gen_sparse(int M, int tot_nnz, int d, int *R, int *C, float *V) {
 
    int nnz_row;
    int val;

    for (int i=0; i < M; i++){
      nnz_row = R[i+1] - R[i];
      for (int j=0; j < nnz_row; j++){
          int ind = R[i]+j; 
          val = rand()%d;
          //val = rand()%d;
          C[ind] = val;
          //V[ind] = (rand()%1000)/30;
          V[ind] = ((float) rand()) / (float) RAND_MAX;
        }    
      std::sort(C+R[i], C+(R[i+1]));
    }
}

void gen_R(int M, int nnzperrow, int *R, int *G_Id, int d) {  
  R[0] = 0;
  int tot_nnz = 0;
  int val;
  for (int m =1; m <= M; m++){ 
   //val = 1 + rand()%(2*nnzperrow);
   val = nnzperrow; //+ rand()%nnzperrow;
   if (val > d) val = 1; 
   tot_nnz += val;
   R[m] = tot_nnz;
   G_Id[m-1] = m-1;
  } 
  //std::random_shuffle(&G_Id[0], &G_Id[M]);
  /*
  for (int m = 0; m < M; m++){ 
  printf("G_Id[%d] = %d \n", m , G_Id[m]);
  } 
  */
}

void gpu_knn(int *R, int *C, float *V, int *G_Id, int M, int leaves, int k, float *knn, int *knn_Id, int max_nnz){
 
	int ppl = M/leaves;
	int m = 8192 / max_nnz;
  m = min(m, ppl);
  
  if (m > 32){ 
    m = 32; 
  } 


	int num_batch_I = (ppl + m - 1) / m;
	int num_batch_J = (ppl + m - 1) / m;

  size_t free, total;
  hipMemGetInfo(&free, &total);
  int log_size = log2(free / (sizeof(float)));
  double arr_len = pow(2, log_size); 

  int size_batch_leaves = arr_len / (ppl * ppl);
  if (size_batch_leaves > leaves) size_batch_leaves = leaves;
  int num_batch_leaves = (leaves + size_batch_leaves - 1) / size_batch_leaves;

  //printf("%d , %d  , %d \n", num_batch_I, num_batch_J, num_batch_leaves);
  float del_t1;
  hipEvent_t t0; 
  hipEvent_t t1;
  int blocks = m*m;
  
  int num_blocks_tri = ppl / m;
  dim3 dimBlock_tri(blocks, 1);	
  dim3 dimGrid_tri(num_blocks_tri, leaves); 
  
  int num_blocks_sq = m * (m-1) /2;
  dim3 dimBlock_sq(m, m);	
  dim3 dimGrid_sq(1, leaves); 
  
  dim3 dimBlock_norm(ppl);	
  dim3 dimGrid_norm(leaves); 
  
  float *d_Norms;
  
  printf("# leaves : %d \n", leaves);
  printf("# points/leaf : %d \n", ppl);
  printf("  max_nnz : %d \n", max_nnz); 
  printf(" block (tri) = (%d,%d) \n", blocks, 1);
  printf(" grid (tri) = (%d, %d) \n", num_blocks_tri, leaves);
  printf(" block (sq) = (%d,%d) \n", m, m);
  printf(" grid (sq) = (%d, %d) \n", 1, leaves);
  printf(" # points = %d \n" , M);
 
  checkCudaErrors(hipMalloc((void **) &d_Norms, sizeof(float) * ppl * size_batch_leaves));


  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));

  checkCudaErrors(hipEventRecord(t0, 0));
  compute_norm <<< dimGrid_norm, dimBlock_norm >>>(R, C, V, G_Id, d_Norms, ppl);
  
  for (int blockInd = 0; blockInd < num_blocks_sq; blockInd++){  
    checkCudaErrors(hipDeviceSynchronize());
    if (blockInd == 0) knn_kernel_tri <<< dimGrid_tri, dimBlock_tri >>>(R, C, V, G_Id, d_Norms, k, knn, knn_Id, ppl, max_nnz, m);
    
    checkCudaErrors(hipDeviceSynchronize());
    knn_kernel_sq <<< dimGrid_sq, dimBlock_sq >>>(R, C, V, G_Id, d_Norms, k, knn, knn_Id, ppl, max_nnz, m ,blockInd);
    checkCudaErrors(hipDeviceSynchronize());
  } 
  
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(t1, 0));
  checkCudaErrors(hipEventSynchronize(t1));
  checkCudaErrors(hipEventElapsedTime(&del_t1, t0, t1));
 
  printf("\n Elapsed time (s) : %.4f \n ", del_t1/1000);
 
  checkCudaErrors(hipFree(d_Norms));
  checkCudaErrors(hipEventDestroy(t0));
  checkCudaErrors(hipEventDestroy(t1));

}





int main(int argc, char **argv)
{

    checkCudaErrors(hipSetDevice(0));

    int d, nnzperrow;
    float *h_V, *d_V;
    int *h_C, *d_C;
    int *h_R, *d_R;
    int *h_G_Id, *d_G_Id;
    int M = 1024*2048;
    int leaves = 2048;
    d = 100000;
    int k = 32;
    nnzperrow = 8;
    int max_nnz = nnzperrow;
    int leaf_size = M / leaves; 
    

    bool print_pt = false;    
    bool print_res = true;    
    int test_leaf = 1000;    
    int test_pt = 1000;

    int *d_knn_Id, *h_knn_Id, *h_knn_Id_seq;
    float *d_knn, *h_knn, *h_knn_seq;

    h_R = (int *)malloc(sizeof(int)*(M+1));
    h_G_Id = (int *)malloc(sizeof(int)*(M));

    h_knn = (float *)malloc(sizeof(float) * M *k);
    h_knn_seq = (float *)malloc(sizeof(float) * M *k / leaves);
    h_knn_Id = (int *)malloc(sizeof(int) * M *k);
    h_knn_Id_seq = (int *)malloc(sizeof(int) * M *k / leaves);
    //memset(h_knn, 1000000.0, sizeof(float) * M * k);
    //memset(h_knn_Id, 0, sizeof(int) * M * k);
    // generate random data 
    gen_R(M, nnzperrow, h_R,h_G_Id, d);
    int tot_nnz = h_R[M];
		h_V = (float *)malloc(sizeof(float)*tot_nnz);
    h_C = (int *)malloc(sizeof(int)*tot_nnz);
    gen_sparse(M, tot_nnz, d , h_R, h_C, h_V);   
    if (print_pt){   
    for (int i = 0; i < M; i++){
        int nnz = h_R[i+1] - h_R[i];
        for (int j = 0; j < nnz; j++)
        printf("R[%d] = %d , C[%d] = %d , V[%d] = %.4f \n", i ,h_R[i], h_R[i]+j, h_C[h_R[i] + j], h_R[i]+j, h_V[h_R[i]+j]);
    }    
    }
    
    checkCudaErrors(hipMalloc((void **) &d_R, sizeof(int)*(M+1)));
    checkCudaErrors(hipMalloc((void **) &d_G_Id, sizeof(int)*(M)));
    checkCudaErrors(hipMalloc((void **) &d_C, sizeof(int)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_V, sizeof(float)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_knn_Id, sizeof(int)*M*k));
    checkCudaErrors(hipMalloc((void **) &d_knn, sizeof(float)*M*k));
 
    checkCudaErrors(hipMemcpy(d_C, h_C, sizeof(int)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_V, h_V, sizeof(float)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_R, h_R, sizeof(int)*(M+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_G_Id, h_G_Id, sizeof(int)*(M), hipMemcpyHostToDevice));
    //checkCudaErrors(hipMemcpy(d_knn, h_knn, sizeof(float)*(M * k), hipMemcpyHostToDevice));
    //checkCudaErrors(hipMemcpy(d_knn_Id, h_knn_Id, sizeof(int)*(M * k), hipMemcpyHostToDevice));
    //checkCudaErrors(hipMemset(d_knn, 1000, sizeof(float) * M * k));  
    //checkCudaErrors(hipMemset(d_knn_Id, 0, sizeof(int) * M * k));  

    printf("Random csr is generated  \n");

    gpu_knn(d_R, d_C, d_V, d_G_Id, M, leaves, k, d_knn, d_knn_Id, max_nnz);
    
    checkCudaErrors(hipMemcpy(h_knn, d_knn, sizeof(float) * M * k, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_knn_Id, d_knn_Id, sizeof(int) * M * k, hipMemcpyDeviceToHost));

    printf(" \n running Seq knn \n");
    printf("\n test for leaf %d , pt %d\n",test_leaf, test_pt);  

    f_knnSeq(h_R, h_C, h_V, h_G_Id, h_knn_seq, h_knn_Id_seq, k, test_leaf, test_pt, leaf_size);
    
    float acc= 0.0;  

    
    int ind;
    bool match;
    int counter = 0;
    int gpu_pt,seq_pt,ind_seq,ind_gpu, nnz_gpu,nnz_seq;
    int ind0_i = h_R[h_G_Id[test_leaf * leaf_size + test_pt]];
    int nnz_i = h_R[h_G_Id[test_leaf * leaf_size + test_pt] + 1] - ind0_i;
    //for (int i=0; i < nnz_i; i++) printf("[(%d, %d, %.4f)] \n", h_G_Id[test_leaf * leaf_size + test_pt], h_C[ind0_i + i], h_V[ind0_i + i]);

    for (int i = 0; i < k; i++){
      ind = test_leaf * k * leaf_size + test_pt * k + i;
      match = (h_knn_Id_seq[test_pt*k + i] == h_knn_Id[ind]);
      if (print_res){
      printf("seq ind %d,\t gpu_ind %d , \t match %d , \t v_seq %.4f, \t v_gpu %.4f , \t ind = %d\n", h_knn_Id_seq[test_pt*k + i], h_knn_Id[ind], match, h_knn_seq[test_pt*k + i], h_knn[ind], ind);
      }
      if (match) acc += 1.0;
      if (counter < 2 && match==0) {
        counter++;
		    gpu_pt = h_knn_Id[ind];
        seq_pt = h_knn_Id_seq[test_pt * k + i];
        ind_gpu = h_R[gpu_pt];
        ind_seq = h_R[seq_pt];
        nnz_gpu = h_R[gpu_pt + 1]  - h_R[gpu_pt];
        nnz_seq = h_R[seq_pt + 1]  - h_R[seq_pt]; 
        //printf("gpu pt %d \n", gpu_pt); 
       
        //for (int q=0; q < nnz_gpu; q++) printf("[(%d, %d, %.4f)] \n", gpu_pt, h_C[ind_gpu + q], h_V[ind_gpu + q]);
        //printf("\n seq pt %d \n", seq_pt); 
       
        //for (int q=0; q < nnz_seq; q++) printf("[(%d, %d, %.4f)] \n", seq_pt, h_C[ind_seq + q], h_V[ind_seq + q]);
        
		
		}
    }
    
    acc /= k;    
    printf("\n\naccuracy %.4f for leaf %d\n\n", acc*100, test_leaf);
    
    checkCudaErrors(hipFree(d_R));
    checkCudaErrors(hipFree(d_G_Id));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_V));
 
    free(h_R);
    free(h_C);
    free(h_V);
    free(h_G_Id);
    free(h_knn);
    free(h_knn_Id);
    free(h_knn_seq);
    free(h_knn_Id_seq);


}
