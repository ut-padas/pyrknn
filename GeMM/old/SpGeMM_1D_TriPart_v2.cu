#include "hip/hip_runtime.h"

#include <stdio.h> 
#include <stdlib.h>
//#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include "knn_seq.cpp"
#define shared_size  8192
#define MaxProcperBlock  1024 





__global__ void compute_norm(int* R, int* C, float* V, int* G_Id, float* K, int M_I, int leaf_batch_g) {

  int row = threadIdx.x;
  int leaf_id_g = leaf_batch_g * gridDim.x + blockIdx.x;

  int g_rowId = leaf_id_g * M_I + row;

  int g_Id = G_Id[g_rowId];
  int ind0_i = R[g_Id];

  int nnz = R[g_Id + 1] - ind0_i;
  float norm_i = 0.0;

  for (int n_i = 0; n_i < nnz; n_i++) norm_i += V[ind0_i + n_i] * V[ind0_i + n_i];
  int ind_write = blockIdx.x * M_I + row;
  K[ind_write] = norm_i;

}

__global__ void compute_dist(int* R, int* C, float* V, int* G_Id, float* Norms, float* K, int* K_Id, int m, int k_nn, int ppl, int leaf_batch_g, int max_nnz, int M){



    // Preproc for Triangular part. 

    int b_i = blockIdx.y;
    int b_j = blockIdx.x;
 
    int b_ind = b_i * gridDim.x + b_j;
    int num_blocks = ppl / m; 

    float tmp = num_blocks * num_blocks - b_ind - 1;
    int blockId_I = sqrt(tmp);
    blockId_I = num_blocks - 1 - blockId_I;

    int blockId_J_tmp = b_ind - num_blocks*num_blocks + (num_blocks - blockId_I)*(num_blocks - blockId_I) + 2*blockId_I;
    int blockId_J = (blockId_J_tmp+1)/2;

    int i = threadIdx.y;
    int j = threadIdx.x;

    int ind = i * (m + 1) + j;

    tmp = -8*ind + 4 * m * (m + 1) - 7;
    int row_Id = sqrt(tmp)/2.0 - 0.5;
    row_Id = m - 1 - row_Id;
    int col_Id = ind + row_Id - m * (m+1)/2 + (m - row_Id) * ((m - row_Id) + 1)/2;

    int tmp1;
    bool lower_block = false;

    // determine the lower block
    if (blockId_I % 2 == 0 && blockId_J_tmp % 2 != 0){
      tmp1 = row_Id;
      row_Id = col_Id;
      col_Id = tmp1;
      lower_block = true;
    }
    if (blockId_I % 2 != 0 && blockId_J_tmp % 2 != 0){
      tmp1 = row_Id;
      row_Id = col_Id;
      col_Id = tmp1;
      lower_block = true;
    }

    // end of partioning   


    //int len_ind = b_i * b_j; 

       
    int leaf_id_g = leaf_batch_g * gridDim.z + blockIdx.z;
    
    int g_rowId_I = leaf_id_g * ppl + blockId_I * m + row_Id;
    //int g_rowId_J = leaf_id_g * ppl + blockId_J * m + col_Id;
    
    if (g_rowId_I >= M) return;
    
    int g_Id_i = G_Id[g_rowId_I];
   

    int ind0_i = R[g_Id_i];
    int ind1_i = R[g_Id_i + 1];

 
    int nnz_i = ind1_i - ind0_i;


   
    //float norm_i = Norms[ind_read_norm_i];
    
    __shared__ int si[8192];
    //__shared__ float dist[1024];
    //__shared__ int dist_Id[1024];
    __shared__ float c_tmp[1024];
    float norm_ij;
    if (j == 0) c_tmp[i] = ind0_i;
    __syncthreads();
    int tmp3 = c_tmp[i];
    for (int n_i = j; n_i < nnz_i; n_i += blockDim.x) si[max_nnz * row_Id + n_i] = C[tmp3 + n_i];
    
    __syncthreads();
    
    int k; 
    //dist_Id[j_Id] = G_Id[leaf_id_g * ppl + blockId_j * blockDim.x + j_Id];
     
    int ind_tmp, row_tmp, col_tmp, read_j_Id, read_i_Id, ind_read_i, ind_read_j, nnz_j, ind_norm_i, ind_norm_j; 
    int tmp_0, tmp_1, ind_jk, ret, testInd;

    // for loop over the row points
    //for (int i_tmp = i; i_tmp < m/2; i_tmp += b_i){
    int i_tmp = i;
 
      // for loop over the column points
      for (int j_tmp = 0; j_tmp < m+1; j_tmp++){
        ind_tmp = i_tmp * (m +1) + j_tmp;
        tmp = -8*ind_tmp + 4 * m * (m + 1) - 7;
        row_tmp = sqrt(tmp)/2.0 - 0.5;
        row_tmp = m - 1 - row_tmp;
        col_tmp = ind_tmp + row_tmp - m * (m+1)/2 + (m - row_tmp) * ((m - row_tmp) + 1)/2;


        int shift_i = max_nnz * row_tmp;
        
        if (lower_block) {
          tmp1 = row_tmp;
          row_tmp = col_tmp;
          col_tmp = tmp1;
        }

        c_tmp[ind_tmp] = 0.0;
        ind_norm_i = blockIdx.z * ppl + blockId_I * m + row_tmp;
        ind_norm_j = blockIdx.z * ppl + blockId_J * m + col_tmp;

        //printf("i, j = (%d, %d) -> row,col = (%d, %d) \n", i_tmp, j_tmp, row_tmp, col_tmp);
        //printf("read j %d , read i %d \n", leaf_id_g * ppl + blockId_J * m + col_tmp, leaf_id_g * ppl + blockId_I * m + row_tmp);
        //printf("row,col = (%d, %d) \n", row_tmp, col_tmp);

        read_j_Id = G_Id[leaf_id_g * ppl + blockId_J * m + col_tmp];
        read_i_Id = G_Id[leaf_id_g * ppl + blockId_I * m + row_tmp];
        norm_ij = Norms[ind_norm_i] + Norms[ind_norm_j]; 
        
        ind_read_j = R[read_j_Id];
        ind_read_i = R[read_i_Id];
        nnz_j = R[read_j_Id + 1] - ind_read_j;
        nnz_i = R[read_i_Id + 1] - ind_read_i;
        // loop over the elements 
        
        
        for (int pos = j; pos < nnz_j; pos++){
       		if (j < nnz_i){
            k = C[ind_read_j + pos];
            ret = 0;
            testInd = 0;    
            // binary search 
            for (int l = nnz_i; l > 1; l/=2){
 							tmp_0 = ret + l;
              tmp_1 = nnz_i - 1;
              testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
              ret = (si[testInd + shift_i] == k) ? testInd : ret; 
            }
             
            tmp_0 = ret + 1;
            tmp_1 = nnz_i - 1;
            testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
            ret = (si[testInd + shift_i] <= k) ? testInd : ret;
            ind_jk = (si[ret + shift_i] == k) ? ret : -1;
            c_tmp[ind] += (ind_jk != -1) ? V[ind_read_j + pos] * V[ind_read_i + j] : 0.0;

        }
        }
         
        __syncthreads();
        
        // reduction 
        
        for (int size = nnz_i/2; size > 0; size /= 2) {
					if (j < size) c_tmp[ind] += c_tmp[ind + size];
          __syncthreads();
        }
        
        int col_write = blockId_J * m + col_tmp;
        int row_write = blockId_I * m + row_tmp;
        int ind_write = blockIdx.z * ppl * ppl + row_write * ppl + col_write;
        int ind_write_T = blockIdx.z * ppl * ppl + col_write * ppl + row_write;
        float val = sqrt(-2*c_tmp[0] + norm_ij);
        if (lower_block == 0 && i_tmp == i && j_tmp == j) K[ind_write] = val;
        if (lower_block == 0 && i_tmp == i && j_tmp == j) K[ind_write_T] = val;
        if (lower_block == 1 && row_tmp != col_tmp && i_tmp == i && j_tmp == j) K[ind_write] = val;
        if (lower_block == 1 && row_tmp != col_tmp && i_tmp == i && j_tmp == j) K[ind_write_T] = val;
        
        //printf("(%d, %d, %d) \n", leaf_id_g, i_tmp, j_tmp);
}

   
}

__global__ void find_neighbor(float* knn, int* knn_Id, float* K, int* K_Id, int* G_Id, int k, int ppl, int leaf_batch_g, int M){

    int col_Id = threadIdx.x; 
    int row_Id = blockIdx.x;

    if (row_Id >= M || col_Id >= M) return;
 
    __shared__ float Dist[4096];
    __shared__ int Dist_Id[4096];

    int size = blockDim.x;
    int leaf_id_g = leaf_batch_g * gridDim.y + blockIdx.y;
    
    int ind_K = blockIdx.z * ppl * (ppl)  + row_Id * (ppl) + col_Id; 
    int i = col_Id;
    //if (row_Id == 1000) printf("val = %.4f , ind = %d \n", K[ind_K], K_Id[ind_K]); 
    Dist[col_Id] = K[ind_K];
    Dist_Id[col_Id] = K_Id[ind_K];
    
    int ind_shared = col_Id;
     
    __syncthreads();

    // bitonic sort
    float tmp_f;
    int tmp_i;
    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
      int ixj = i ^ l;
      int ixj_tmp = ixj;
      if (ixj > i){
        if ((i & g) == 0){
          if (Dist[ind_shared] > Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        } else {
          if (Dist[ind_shared] < Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        }
        }
      
      __syncthreads();
      }
  }

  size = 2*k;

  int ind_knn = leaf_id_g * ppl * k + row_Id * k + col_Id;

  // should change to the given knn 
  //if (col_Id >= k && col_Id < size) Dist[col_Id] = 1e30;
  if (col_Id >= k && col_Id < size) Dist[col_Id] = 1e30;
  //if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = 0;
  if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = 0;

  __syncthreads();
	for (int g = 2; g <= size; g *= 2){
		for (int l = g/2; l>0; l /= 2){
		int ixj = i ^ l;
		int ixj_tmp =  ixj;
		if (ixj > i){
			if ((i & g) == 0){
				if (Dist[col_Id] > Dist[ixj_tmp]){
						 tmp_f = Dist[ixj_tmp];
						 Dist[ixj_tmp] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
						 tmp_i = Dist_Id[ixj_tmp];
						 Dist_Id[ixj_tmp] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			} else {
				if (Dist[col_Id] < Dist[ixj_tmp]){
						 tmp_f = Dist[ixj_tmp];
						 Dist[ixj_tmp] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
						 tmp_i = Dist_Id[ixj_tmp];
						 Dist_Id[ixj_tmp] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			  }
	    }
		
		__syncthreads();
		}
     
    if (col_Id < k){
      knn[ind_knn] = Dist[col_Id];
      knn_Id[ind_knn] = Dist_Id[col_Id];
    }
    
}

}



void gen_sparse(int M, int tot_nnz, int d, int *R, int *C, float *V) {
 
    int nnz_row;
    int val;

    for (int i=0; i < M; i++){
      nnz_row = R[i+1] - R[i];
      for (int j=0; j < nnz_row; j++){
          int ind = R[i]+j; 
          val = rand()%d;
          //val = rand()%d;
          C[ind] = val;
          //V[ind] = rand()%100;
          V[ind] = ((float) rand()) / (float) RAND_MAX;
        }    
      std::sort(C+R[i], C+(R[i+1]));
      /*
      printf("\n point %d\n", i);
      for (int j=R[i]; j<R[i+1]; j++) {
      printf("(%d ,%.4f) ",C[j], V[j]);
      }
      */
    }
}

void gen_R(int M, int nnzperrow, int *R, int *G_Id, int d) {  
  R[0] = 0;
  int tot_nnz = 0;
  int val;
  for (int m =1; m <= M; m++){ 
   //val = 1 + rand()%(2*nnzperrow);
   val = nnzperrow; //+ rand()%nnzperrow;
   if (val > d) val = 1; 
   tot_nnz += val;
   R[m] = tot_nnz;
   G_Id[m-1] = m-1;
  } 
  //std::random_shuffle(&G_Id[0], &G_Id[M]);
  /* 
  for (int m = 0; m < M; m++){ 
  printf("G_Id[%d] = %d \n", m , G_Id[m]);
  } 
  */
}

void gpu_knn(int *R, int *C, float *V, int *G_Id, int M, int leaves, int k, float *knn, int *knn_Id, int max_nnz){
 
	int ppl = M/leaves;
	
  size_t free, total;
  hipMemGetInfo(&free, &total);
  
  int log_size = log2(free / (sizeof(float)));
  double arr_len = pow(2, log_size);  

  int size_batch_leaves = arr_len / (ppl * ppl );
  
  if (size_batch_leaves > leaves) size_batch_leaves = leaves; 
  
  int num_batch_leaves = (leaves) / size_batch_leaves; 
   

	//int M_I = M/leaves;

  float del_t1;
  hipEvent_t t0; 
  hipEvent_t t1;
   
  //int m = min(ppl, 1024);
  int m = 32;
  m = (max_nnz > 32) ? min(m, max_nnz) : min(m, 32);
  int num_blocks = (ppl + m - 1) / m;
  
  int block_size_i = m / 2;
  int block_size_j = m + 1;

  int num_batch_I = (ppl + m - 1) / m;
  int num_batch_J = (ppl + m - 1) / m;

  dim3 dimBlock(block_size_j, block_size_i, 1);	
  dim3 dimGrid(num_batch_J, num_batch_I, size_batch_leaves);
 
  dim3 dimBlock_findk(ppl, 1);
  dim3 dimGrid_findk(ppl, size_batch_leaves);

  dim3 dimBlock_norm(ppl);
  dim3 dimGrid_norm(size_batch_leaves);

  hipMemGetInfo(&free, &total);
  printf("%d kB from  %d kB is free \n", free/1024, total/1024);
  

  float *d_K, *d_Norms;
  int *d_K_Id;
  checkCudaErrors(hipMalloc((void **) &d_K, sizeof(float) * arr_len));
  //checkCudaErrors(hipMalloc((void **) &d_K_Id, sizeof(int) * arr_len));
  checkCudaErrors(hipMalloc((void **) &d_Norms, sizeof(float) * size_batch_leaves * ppl));

  
  //size_t free, total; 

  hipMemGetInfo(&free, &total);
  printf("%d kB from  %d kB is free \n", free/1024, total/1024);



  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));

  checkCudaErrors(hipEventRecord(t0, 0));
  checkCudaErrors(hipProfilerStart());
  printf("# leaves : %d \n", leaves);
  printf("# points/leaf : %d \n", ppl);
  printf(" max_nnz : %d \n", max_nnz);
  printf(" blockDim (norms) : (%d) \n", ppl);
  printf(" blockGrid (norms) : (%d) \n", size_batch_leaves); 
  printf(" blockDim (distance) : (%d,%d,1) \n", block_size_j, block_size_i, 1);
  printf(" blockGrid (distance) : (%d,%d,%d) \n", num_batch_J, num_batch_I, size_batch_leaves);
  printf(" blockDim (find knn) : (%d,%d,1) \n", ppl, 1);
  printf(" blockGrid (find knn) : (%d,%d,1) \n", ppl, size_batch_leaves);
  printf(" num leaves per loop : %d \n",size_batch_leaves);
  printf(" # points = %d \n" , M);  
  

  for (int leaf_id_g = 0; leaf_id_g < num_batch_leaves; leaf_id_g++){
    compute_norm <<< dimGrid_norm, dimBlock_norm >>>(R, C, V, G_Id, d_Norms,ppl, leaf_id_g);
    checkCudaErrors(hipDeviceSynchronize()); 
    compute_dist <<< dimGrid, dimBlock >>>(R, C, V, G_Id, d_Norms, d_K, d_K_Id, m, k, ppl, leaf_id_g, max_nnz, M);
    checkCudaErrors(hipDeviceSynchronize());
    //find_neighbor <<< dimGrid_findk, dimBlock_findk >>>(knn, knn_Id, d_K, d_K_Id, G_Id, k, ppl, leaf_id_g, M);
  } 

  checkCudaErrors(hipProfilerStop());  
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(t1, 0));
  checkCudaErrors(hipEventSynchronize(t1));
  checkCudaErrors(hipEventElapsedTime(&del_t1, t0, t1));
  


   
  printf("\n Elapsed time (s) : %.4f \n ", del_t1/1000);
 
  checkCudaErrors(hipFree(d_K));
  checkCudaErrors(hipEventDestroy(t0));
  checkCudaErrors(hipEventDestroy(t1));

}





int main(int argc, char **argv)
{

  //, del_t2, del_t3;

    checkCudaErrors(hipSetDevice(0));

    int d, nnzperrow;
    float *h_V, *d_V;
    int *h_C, *d_C;
    int *h_R, *d_R;
    int *h_G_Id, *d_G_Id;
    int M = 1024;     // total number of points 
    int leaves = 1;     // number of leaves
    d = 10000;
    int k = 32;
    nnzperrow = 32;
    int max_nnz = nnzperrow;
    int leaf_size = M / leaves; 
    

    int *d_knn_Id, *h_knn_Id, *h_knn_Id_seq;
    float *d_knn, *h_knn, *h_knn_seq;

    h_R = (int *)malloc(sizeof(int)*(M+1));
    h_G_Id = (int *)malloc(sizeof(int)*(M));

    h_knn = (float *)malloc(sizeof(float) * M *k);
    h_knn_seq = (float *)malloc(sizeof(float) * M *k / leaves);
    h_knn_Id = (int *)malloc(sizeof(int) * M *k);
    h_knn_Id_seq = (int *)malloc(sizeof(int) * M *k / leaves);


    // generate random data 
    gen_R(M, nnzperrow, h_R,h_G_Id, d);
    int tot_nnz = h_R[M];
		h_V = (float *)malloc(sizeof(float)*tot_nnz);
    h_C = (int *)malloc(sizeof(int)*tot_nnz);
    gen_sparse(M, tot_nnz, d , h_R, h_C, h_V);   
    /* 
    for (int i = 0; i < M; i++){
        int nnz = h_R[i+1] - h_R[i];
        for (int j = 0; j < nnz; j++)
        printf("R[%d] = %d , C[%d] = %d , V[%d] = %.4f \n", i ,h_R[i], h_R[i]+j, h_C[h_R[i] + j], h_R[i]+j, h_V[h_R[i]+j]);
    }    
    */
    checkCudaErrors(hipMalloc((void **) &d_R, sizeof(int)*(M+1)));
    checkCudaErrors(hipMalloc((void **) &d_G_Id, sizeof(int)*(M)));
    checkCudaErrors(hipMalloc((void **) &d_C, sizeof(int)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_V, sizeof(float)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_knn_Id, sizeof(int)*M*k));
    checkCudaErrors(hipMalloc((void **) &d_knn, sizeof(float)*M*k));
 
    checkCudaErrors(hipMemcpy(d_C, h_C, sizeof(int)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_V, h_V, sizeof(float)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_R, h_R, sizeof(int)*(M+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_G_Id, h_G_Id, sizeof(int)*(M), hipMemcpyHostToDevice));

    printf("\nRandom csr is generated  \n");

    gpu_knn(d_R, d_C, d_V, d_G_Id, M, leaves, k, d_knn, d_knn_Id, max_nnz);
  
    checkCudaErrors(hipMemcpy(h_knn, d_knn, sizeof(float) * M * k, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_knn_Id, d_knn_Id, sizeof(int) * M * k, hipMemcpyDeviceToHost));
 
    int test_leaf = 0;
    int test_pt = 1000;
    //printf(" \n running Seq knn \n");
    //printf("\n test for leaf %d \n",test_leaf);

    f_knnSeq(h_R, h_C, h_V, h_G_Id, h_knn_seq, h_knn_Id_seq, k, test_leaf, test_pt, leaf_size);

    float acc= 0.0;

    int ind;
    bool match;
    int counter = 0;
    int gpu_pt,seq_pt,ind_seq,ind_gpu, nnz_gpu,nnz_seq;
    int ind0_i = h_R[h_G_Id[test_leaf * leaf_size + test_pt]];
    int nnz_i = h_R[h_G_Id[test_leaf * leaf_size + test_pt] + 1] - ind0_i;
    //for (int i=0; i < nnz_i; i++) printf("[(%d, %d, %.4f)] \n", h_G_Id[test_leaf * leaf_size + test_pt], h_C[ind0_i + i], h_V[ind0_i + i]);

    for (int i = 0; i < k; i++){
      ind = test_leaf * k * leaf_size + test_pt * k + i;
      match = (h_knn_Id_seq[test_pt*k + i] == h_knn_Id[ind]);
      //printf("seq val %d,\t gpu_val %d , \t match %d , \t v_seq %.4f, \t v_gpu %.4f , \t ind = %d\n", h_knn_Id_seq[test_pt*k + i], h_knn_Id[ind], match, h_knn_seq[test_pt*k + i], h_knn[ind], ind);
      if (match) acc += 1.0;
      if (counter < 2 && match==0) {
        counter++;
        gpu_pt = h_knn_Id[ind];
        seq_pt = h_knn_Id_seq[test_pt * k + i];
        ind_gpu = h_R[gpu_pt];
        ind_seq = h_R[seq_pt];
        nnz_gpu = h_R[gpu_pt + 1]  - h_R[gpu_pt];
        nnz_seq = h_R[seq_pt + 1]  - h_R[seq_pt];
        //printf("gpu pt %d \n", gpu_pt);

        //for (int q=0; q < nnz_gpu; q++) printf("[(%d, %d, %.4f)] \n", gpu_pt, h_C[ind_gpu + q], h_V[ind_gpu + q]);
        //printf("\n seq pt %d \n", seq_pt);

        //for (int q=0; q < nnz_seq; q++) printf("[(%d, %d, %.4f)] \n", seq_pt, h_C[ind_seq + q], h_V[ind_seq + q]);


    }
    }


    acc /= k;

    printf("accuracy %.2f %% for leaf %d\n\n", acc*100, test_leaf);







 
    printf("\n\n");
    checkCudaErrors(hipFree(d_R));
    checkCudaErrors(hipFree(d_G_Id));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_V));
    free(h_R);
    free(h_C);
    free(h_V);
    free(h_G_Id);


}







