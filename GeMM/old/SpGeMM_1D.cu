#include "hip/hip_runtime.h"

#include <stdio.h> 
#include <stdlib.h>
//#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include "knn_seq.cpp"
#define shared_size  8192
#define MaxProcperBlock  1024 





__global__ void compute_norm(int* R, int* C, float* V, int* G_Id, float* Norms, int M_I, int leaf_batch_g) {

  int row = threadIdx.x;
  int leaf_id_g = leaf_batch_g * gridDim.x + blockIdx.x;

  int g_rowId = leaf_id_g * M_I + row;

  int g_Id = G_Id[g_rowId];
  int ind0_i = R[g_Id];

  int nnz = R[g_Id + 1] - ind0_i;
  float norm_i = 0.0;

  for (int n_i = 0; n_i < nnz; n_i++) norm_i += V[ind0_i + n_i] * V[ind0_i + n_i];
  int ind_write = blockIdx.x * M_I + row;
  Norms[ind_write] = norm_i;

}

__global__ void compute_dist(int* R, int* C, float* V, int* G_Id, float* Norms, float* K, int* K_Id, int m, int k_nn, int ppl, int leaf_batch_g, int max_nnz, int M){

    int j_Id = threadIdx.x;
    int blockId_j = blockIdx.x;

    int row_Id_leaf = blockIdx.y;
       
    int leaf_id_g = leaf_batch_g * gridDim.z + blockIdx.z;
    
    int g_rowId_I = leaf_id_g * ppl + row_Id_leaf;
    
    if (g_rowId_I >= M) return;
    
    int g_Id_i = G_Id[g_rowId_I]; 
   

    int ind0_i = R[g_Id_i];
    int ind1_i = R[g_Id_i + 1];

 
    int nnz_i = ind1_i - ind0_i;


    int ind_read_norm_i = blockIdx.z * ppl + row_Id_leaf;
   
    float norm_i = Norms[ind_read_norm_i];
    
    __shared__ int si[1024];
    __shared__ float dist[1024];
    __shared__ int dist_Id[1024];
    __shared__ float c_tmp[1024];

    //c_tmp[1] = 0.0;
    for (int n_i = j_Id; n_i < nnz_i; n_i += blockDim.x) si[n_i] = C[ind0_i + n_i];
     
    __syncthreads();
    
    int k; 
    dist_Id[j_Id] = G_Id[leaf_id_g * ppl + blockId_j * blockDim.x + j_Id];
    //dist_Id[j_Id] = blockId_j * blockDim.x + j_Id;
   
    // for loop over the points    
    for (int col_Id = 0; col_Id < blockDim.x; col_Id++) {
       
      int col_Id_leaf = blockId_j * blockDim.x + col_Id;
      int g_Id_J = leaf_id_g * ppl + col_Id_leaf;
      int g_Id_j = G_Id[g_Id_J]; 
      int ind0_j = R[g_Id_j];
      int ind1_j = R[g_Id_j + 1];
      
      int nnz_j = ind1_j - ind0_j;
      int ind_read_norm_j = blockIdx.z * ppl + col_Id_leaf; 
      float norm_ij = norm_i + Norms[ind_read_norm_j];
      c_tmp[j_Id] = 0.0;   
      // for loop over the elements
      for (int pos = 0; pos < nnz_j; pos++){
      if (j_Id < nnz_i) {
            k = C[ind0_j + pos];
            c_tmp[j_Id] += (k == si[j_Id]) * V[ind0_j + pos] * V[ind0_i + j_Id];
            //if (k == si[j_Id]) c_tmp[] += V[ind0_j + pos_k] * V[ind0_i + j_Id];
      }
      __syncthreads(); 
      }

    
      // reduction among the inner product 
      for (int size = nnz_i/2; size >0; size /= 2) {
        if (j_Id < size) c_tmp[j_Id] += c_tmp[j_Id + size];
        __syncthreads();
      }
      float c;
      if (col_Id == j_Id) c = c_tmp[0];
      //if (col_Id_leaf == 1000 && row_Id_leaf == 1000 && j_Id == 0) printf("val =%.4f , norm = %.4f \n", c_tmp[0], norm_ij);
      if (col_Id == j_Id) c = -2*c + norm_ij;
      //if (j_Id == 0) c_tmp[0] = norm_ij;
      if (col_Id == j_Id) c = (c > 0) ? sqrt(c) : 0.0;
      //if (col_Id_leaf == 1000 && row_Id_leaf == 1000 && j_Id == col_Id) printf("val =%.4f , norm = %.4f \n", c_tmp[0], norm_ij);
      if (col_Id == j_Id) dist[col_Id] = c;
      __syncthreads();   
    }
    
    // bitonic sort 

    if (m > k_nn){ 
 
    int size = blockDim.x;
    // bitonic sort  
    float tmp_f;
    int tmp_i;
    
    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
	    int ixj = j_Id ^ l;
	    if (ixj > j_Id){
		    if ((j_Id & g) == 0){
			    if (dist[j_Id] > dist[ixj]){ 
               tmp_f = dist[ixj]; 
               dist[ixj] = dist[j_Id]; 
               dist[j_Id] = tmp_f;
    
               tmp_i = dist_Id[ixj]; 
               dist_Id[ixj] = dist_Id[j_Id]; 
               dist_Id[j_Id] = tmp_i;
                }
		    } else {
			    if (dist[j_Id] < dist[ixj]){ 
               tmp_f = dist[ixj]; 
               dist[ixj] = dist[j_Id]; 
               dist[j_Id] = tmp_f;
    
               tmp_i = dist_Id[ixj]; 
               dist_Id[ixj] = dist_Id[j_Id]; 
               dist_Id[j_Id] = tmp_i;
               } 
		    }
	      }
	    __syncthreads();
      }
    }
    }

    if (j_Id < k_nn) {
	    int col_write = blockIdx.x * k_nn + j_Id; 
	    int row_write = row_Id_leaf;
	    //int ind_write = leaf_id_g * ppl * k_nn + row_write * k_nn + col_write;
	    int ind_write = blockIdx.z * ppl * (ppl / m) * k_nn  + row_write * (ppl * k_nn / m) + col_write;
      
      //if (row_Id_leaf == 1000) printf("Id = %d , val = %.4f , ind = %d , m = %d, \n", dist_Id[j_Id], dist[j_Id] , ind_write);  
	    K[ind_write] = dist[j_Id];
	    //K_Id[ind_write] = dist_Id[j_Id];
	    K_Id[ind_write] = dist_Id[j_Id];
    }
    
}

__global__ void find_neighbor(float* knn, int* knn_Id, float* K, int* K_Id, int* G_Id, int k, int ppl, int leaf_batch_g, int M){

    int col_Id = threadIdx.x; 
    int row_Id = blockIdx.x;

    if (row_Id >= M || col_Id >= M) return;
 
    __shared__ float Dist[4096];
    __shared__ int Dist_Id[4096];

    int size = blockDim.x;
    int leaf_id_g = leaf_batch_g * gridDim.y + blockIdx.y;
    
    int ind_K = blockIdx.z * ppl * (ppl)  + row_Id * (ppl) + col_Id; 
    int i = col_Id;
    //if (row_Id == 1000) printf("val = %.4f , ind = %d \n", K[ind_K], K_Id[ind_K]); 
    Dist[col_Id] = K[ind_K];
    Dist_Id[col_Id] = K_Id[ind_K];
    
    int ind_shared = col_Id;
     
    __syncthreads();

    // bitonic sort
    float tmp_f;
    int tmp_i;
    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
      int ixj = i ^ l;
      int ixj_tmp = ixj;
      if (ixj > i){
        if ((i & g) == 0){
          if (Dist[ind_shared] > Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        } else {
          if (Dist[ind_shared] < Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        }
        }
      
      __syncthreads();
      }
  }

  size = 2*k;

  int ind_knn = leaf_id_g * ppl * k + row_Id * k + col_Id;

  // should change to the given knn 
  //if (col_Id >= k && col_Id < size) Dist[col_Id] = 1e30;
  if (col_Id >= k && col_Id < size) Dist[col_Id] = 1e30;
  //if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = 0;
  if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = 0;

  __syncthreads();
	for (int g = 2; g <= size; g *= 2){
		for (int l = g/2; l>0; l /= 2){
		int ixj = i ^ l;
		int ixj_tmp =  ixj;
		if (ixj > i){
			if ((i & g) == 0){
				if (Dist[col_Id] > Dist[ixj_tmp]){
						 tmp_f = Dist[ixj_tmp];
						 Dist[ixj_tmp] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
						 tmp_i = Dist_Id[ixj_tmp];
						 Dist_Id[ixj_tmp] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			} else {
				if (Dist[col_Id] < Dist[ixj_tmp]){
						 tmp_f = Dist[ixj_tmp];
						 Dist[ixj_tmp] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
						 tmp_i = Dist_Id[ixj_tmp];
						 Dist_Id[ixj_tmp] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			  }
	    }
		
		__syncthreads();
		}
     
    if (col_Id < k){
      knn[ind_knn] = Dist[col_Id];
      knn_Id[ind_knn] = Dist_Id[col_Id];
    }
    
}

}



void gen_sparse(int M, int tot_nnz, int d, int *R, int *C, float *V) {
 
    int nnz_row;
    int val;

    for (int i=0; i < M; i++){
      nnz_row = R[i+1] - R[i];
      for (int j=0; j < nnz_row; j++){
          int ind = R[i]+j; 
          val = rand()%d;
          //val = rand()%d;
          C[ind] = val;
          //V[ind] = rand()%100;
          V[ind] = ((float) rand()) / (float) RAND_MAX;
        }    
      std::sort(C+R[i], C+(R[i+1]));
      /*
      printf("\n point %d\n", i);
      for (int j=R[i]; j<R[i+1]; j++) {
      printf("(%d ,%.4f) ",C[j], V[j]);
      }
      */
    }
}

void gen_R(int M, int nnzperrow, int *R, int *G_Id, int d) {  
  R[0] = 0;
  int tot_nnz = 0;
  int val;
  for (int m =1; m <= M; m++){ 
   //val = 1 + rand()%(2*nnzperrow);
   val = nnzperrow; //+ rand()%nnzperrow;
   if (val > d) val = 1; 
   tot_nnz += val;
   R[m] = tot_nnz;
   G_Id[m-1] = m-1;
  } 
  //std::random_shuffle(&G_Id[0], &G_Id[M]);
  /* 
  for (int m = 0; m < M; m++){ 
  printf("G_Id[%d] = %d \n", m , G_Id[m]);
  } 
  */
}

void gpu_knn(int *R, int *C, float *V, int *G_Id, int M, int leaves, int k, float *knn, int *knn_Id, int max_nnz){
 
	int ppl = M/leaves;
	
  size_t free, total;
  hipMemGetInfo(&free, &total);
  
  int log_size = log2(free / (2*sizeof(float)));
  double arr_len = pow(2, log_size);  

  int size_batch_leaves = arr_len / (ppl * ppl );
  
  if (size_batch_leaves > leaves) size_batch_leaves = leaves; 
  
  int num_batch_leaves = (leaves) / size_batch_leaves; 
   


	//int M_I = M/leaves;

  float del_t1;
  hipEvent_t t0; 
  hipEvent_t t1;
   
  int m = min(ppl, 1024);
  m = (max_nnz > 32) ? min(m, max_nnz) : min(m, 32);
  int num_blocks = (ppl + m - 1) / m;
  
  dim3 dimBlock(m, 1, 1);	
  dim3 dimGrid(num_blocks, ppl, size_batch_leaves);
 
  dim3 dimBlock_findk(ppl, 1);
  dim3 dimGrid_findk(ppl, size_batch_leaves);

  dim3 dimBlock_norm(ppl);
  dim3 dimGrid_norm(size_batch_leaves);

  hipMemGetInfo(&free, &total);
  printf("%d kB from  %d kB is free \n", free/1024, total/1024);
  

  float *d_K, *d_Norms;
  int *d_K_Id;
  checkCudaErrors(hipMalloc((void **) &d_K, sizeof(float) * arr_len));
  checkCudaErrors(hipMalloc((void **) &d_K_Id, sizeof(int) * arr_len));
  checkCudaErrors(hipMalloc((void **) &d_Norms, sizeof(float) * size_batch_leaves * ppl));

  
  //size_t free, total; 

  hipMemGetInfo(&free, &total);
  printf("%d kB from  %d kB is free \n", free/1024, total/1024);



  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));

  checkCudaErrors(hipEventRecord(t0, 0));
  checkCudaErrors(hipProfilerStart());
  printf("# leaves : %d \n", leaves);
  printf("# points/leaf : %d \n", ppl);
  printf(" max_nnz : %d \n", max_nnz);
  printf(" blockDim (norms) : (%d) \n", ppl);
  printf(" blockGrid (norms) : (%d) \n", size_batch_leaves); 
  printf(" blockDim (distance) : (%d,%d,1) \n", m, 1);
  printf(" blockGrid (distance) : (%d,%d,%d) \n", num_blocks, ppl, size_batch_leaves);
  printf(" blockDim (find knn) : (%d,%d,1) \n", ppl, 1);
  printf(" blockGrid (find knn) : (%d,%d,1) \n", ppl, size_batch_leaves);
  printf(" num leaves per loop : %d \n",size_batch_leaves);
  printf(" # points = %d \n" , M);  
  

  for (int leaf_id_g = 0; leaf_id_g < num_batch_leaves; leaf_id_g++){
    compute_norm <<< dimGrid_norm, dimBlock_norm >>>(R, C, V, G_Id, d_Norms,ppl, leaf_id_g);
    checkCudaErrors(hipDeviceSynchronize()); 
    compute_dist <<< dimGrid, dimBlock >>>(R, C, V, G_Id, d_Norms, d_K, d_K_Id, m, k, ppl, leaf_id_g, max_nnz, M);
    checkCudaErrors(hipDeviceSynchronize());
    find_neighbor <<< dimGrid_findk, dimBlock_findk >>>(knn, knn_Id, d_K, d_K_Id, G_Id, k, ppl, leaf_id_g, M);
  } 

  checkCudaErrors(hipProfilerStop());  
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(t1, 0));
  checkCudaErrors(hipEventSynchronize(t1));
  checkCudaErrors(hipEventElapsedTime(&del_t1, t0, t1));
  


   
  printf("\n Elapsed time (s) : %.4f \n ", del_t1/1000);
 
  checkCudaErrors(hipFree(d_K));
  checkCudaErrors(hipEventDestroy(t0));
  checkCudaErrors(hipEventDestroy(t1));

}





int main(int argc, char **argv)
{

  //, del_t2, del_t3;

    checkCudaErrors(hipSetDevice(0));

    int d, nnzperrow;
    float *h_V, *d_V;
    int *h_C, *d_C;
    int *h_R, *d_R;
    int *h_G_Id, *d_G_Id;
    int M = 1024;     // total number of points 
    int leaves = 1;     // number of leaves
    d = 10000;
    int k = 32;
    nnzperrow = 32;
    int max_nnz = nnzperrow;
    int leaf_size = M / leaves; 
    

    int *d_knn_Id, *h_knn_Id, *h_knn_Id_seq;
    float *d_knn, *h_knn, *h_knn_seq;

    h_R = (int *)malloc(sizeof(int)*(M+1));
    h_G_Id = (int *)malloc(sizeof(int)*(M));

    h_knn = (float *)malloc(sizeof(float) * M *k);
    h_knn_seq = (float *)malloc(sizeof(float) * M *k / leaves);
    h_knn_Id = (int *)malloc(sizeof(int) * M *k);
    h_knn_Id_seq = (int *)malloc(sizeof(int) * M *k / leaves);


    // generate random data 
    gen_R(M, nnzperrow, h_R,h_G_Id, d);
    int tot_nnz = h_R[M];
		h_V = (float *)malloc(sizeof(float)*tot_nnz);
    h_C = (int *)malloc(sizeof(int)*tot_nnz);
    gen_sparse(M, tot_nnz, d , h_R, h_C, h_V);   
    /* 
    for (int i = 0; i < M; i++){
        int nnz = h_R[i+1] - h_R[i];
        for (int j = 0; j < nnz; j++)
        printf("R[%d] = %d , C[%d] = %d , V[%d] = %.4f \n", i ,h_R[i], h_R[i]+j, h_C[h_R[i] + j], h_R[i]+j, h_V[h_R[i]+j]);
    }    
    */
    checkCudaErrors(hipMalloc((void **) &d_R, sizeof(int)*(M+1)));
    checkCudaErrors(hipMalloc((void **) &d_G_Id, sizeof(int)*(M)));
    checkCudaErrors(hipMalloc((void **) &d_C, sizeof(int)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_V, sizeof(float)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_knn_Id, sizeof(int)*M*k));
    checkCudaErrors(hipMalloc((void **) &d_knn, sizeof(float)*M*k));
 
    checkCudaErrors(hipMemcpy(d_C, h_C, sizeof(int)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_V, h_V, sizeof(float)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_R, h_R, sizeof(int)*(M+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_G_Id, h_G_Id, sizeof(int)*(M), hipMemcpyHostToDevice));

    printf("\nRandom csr is generated  \n");

    gpu_knn(d_R, d_C, d_V, d_G_Id, M, leaves, k, d_knn, d_knn_Id, max_nnz);
  
    checkCudaErrors(hipMemcpy(h_knn, d_knn, sizeof(float) * M * k, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_knn_Id, d_knn_Id, sizeof(int) * M * k, hipMemcpyDeviceToHost));
 
    int test_leaf = 0;
    int test_pt = 1000;
    //printf(" \n running Seq knn \n");
    //printf("\n test for leaf %d \n",test_leaf);

    f_knnSeq(h_R, h_C, h_V, h_G_Id, h_knn_seq, h_knn_Id_seq, k, test_leaf, test_pt, leaf_size);

    float acc= 0.0;

    int ind;
    bool match;
    int counter = 0;
    int gpu_pt,seq_pt,ind_seq,ind_gpu, nnz_gpu,nnz_seq;
    int ind0_i = h_R[h_G_Id[test_leaf * leaf_size + test_pt]];
    int nnz_i = h_R[h_G_Id[test_leaf * leaf_size + test_pt] + 1] - ind0_i;
    //for (int i=0; i < nnz_i; i++) printf("[(%d, %d, %.4f)] \n", h_G_Id[test_leaf * leaf_size + test_pt], h_C[ind0_i + i], h_V[ind0_i + i]);

    for (int i = 0; i < k; i++){
      ind = test_leaf * k * leaf_size + test_pt * k + i;
      match = (h_knn_Id_seq[test_pt*k + i] == h_knn_Id[ind]);
      //printf("seq val %d,\t gpu_val %d , \t match %d , \t v_seq %.4f, \t v_gpu %.4f , \t ind = %d\n", h_knn_Id_seq[test_pt*k + i], h_knn_Id[ind], match, h_knn_seq[test_pt*k + i], h_knn[ind], ind);
      if (match) acc += 1.0;
      if (counter < 2 && match==0) {
        counter++;
        gpu_pt = h_knn_Id[ind];
        seq_pt = h_knn_Id_seq[test_pt * k + i];
        ind_gpu = h_R[gpu_pt];
        ind_seq = h_R[seq_pt];
        nnz_gpu = h_R[gpu_pt + 1]  - h_R[gpu_pt];
        nnz_seq = h_R[seq_pt + 1]  - h_R[seq_pt];
        //printf("gpu pt %d \n", gpu_pt);

        //for (int q=0; q < nnz_gpu; q++) printf("[(%d, %d, %.4f)] \n", gpu_pt, h_C[ind_gpu + q], h_V[ind_gpu + q]);
        //printf("\n seq pt %d \n", seq_pt);

        //for (int q=0; q < nnz_seq; q++) printf("[(%d, %d, %.4f)] \n", seq_pt, h_C[ind_seq + q], h_V[ind_seq + q]);


    }
    }


    acc /= k;

    printf("accuracy %.2f %% for leaf %d\n\n", acc*100, test_leaf);







 
    printf("\n\n");
    checkCudaErrors(hipFree(d_R));
    checkCudaErrors(hipFree(d_G_Id));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_V));
    free(h_R);
    free(h_C);
    free(h_V);
    free(h_G_Id);


}







