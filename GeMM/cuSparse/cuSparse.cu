
#include <stdio.h> 
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>








/*
int gen_dense(int m, int d, float *outA)
{
   
  for (i=0; i < N; i++)
  { 
    for (j=0; j < N; j++)
    {
       int r = rand();
*/
       
    
int gen_Spdata(int m, int d, int Nnzperrow, float* V, int* I, int* J)
{
  int i, j;
  //double rMax = (double)RAND_MAX;
  //float *values = (float *)malloc(sizeof(float) * Nnzperrow * m);
  //int *rowptr = (int *)malloc(sizeof(int) *(m+1));
  //int *colInd = (int *)malloc(sizeof(int) * Nnzperrow * m);
  int totalNnz = 0;
    
  I[0] = 0;      
  for (i=0; i<m; i++)
  {
    I[i+1] = I[i] + Nnzperrow;
    for (j=0; j < Nnzperrow; j++) 
    {
      int ind = I[i]+j;
      J[ind] = rand()%d;
      V[ind] = rand();
    }
  }
  
  //V = &
  //*values = V;
  //*rowptr = I;
  //*colInd = J;
  totalNnz += m*Nnzperrow;

  return totalNnz;
}


#define CHECK_CUSPARSE(func) \
{ \
 hipsparseStatus_t status = (func); \
 if (status != HIPSPARSE_STATUS_SUCCESS) { \
 printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
 __LINE__, hipsparseGetErrorString(status), status); \
 return EXIT_FAILURE; \
 } \
}



int main(int argc, char **argv)
{




  float t_1,t_2,t_3,wtime;


  t_1 = 0;
  t_2 = 0;
  t_3 = 0;
  wtime = 0;



  for (int l=0; l<3000; l++){
  printf(" l = %d , tot_t = %.2f \n", l, wtime);

	//float *dA;
  float *C, *dC;
  //int *dANnzPerRow = 2; 
  float *dCsrValA;
  int *dCsrRowPtrA;
  int *dCsrColIndA;
  //float *dCsrValB;
  //int *dCsrRowPtrB;
  //int *dCsrColIndB;
  float *dCsrValC;
  int *dCsrRowPtrC=0;
  int *dCsrColIndC=0;
  //float *tmp_val;
  //int *tmp_col;
  //int *tmp_row;



  //int *dCNnzPerRow; 
  //int *dCtotalNnz; 
  
  //int totalAnnz_feed;
  //int trueTotalAnnz;

  
  float del_t1;
  float del_t2;
  float del_t3;
  
  checkCudaErrors(hipSetDevice(0));

  hipsparseHandle_t handle = 0; 
  hipsparseMatDescr_t Adescr = 0; 
  //hipsparseMatDescr_t Bdescr = 0; 
  hipsparseMatDescr_t Cdescr = 0; 
  
  hipEvent_t t0; 
  hipEvent_t t1;
  hipEvent_t t2;
  hipEvent_t t3;

  // trueTotalAnnz = generate_random_dense_matrix(m, d, dANnzPerRow, &A);
  int nnz;
  int m = 300;
  int d = 10000;
  int nnzperrow = 600;
  float *V; 
  int *J; 
  int *I;


  V = (float *)malloc(sizeof(float)*nnzperrow*m);
  J = (int *)malloc(sizeof(int)*nnzperrow*m);
  I = (int *)malloc(sizeof(int)*(m+1));


  int i, j;
  //double rMax = (double)RAND_MAX;
  //float *values = (float *)malloc(sizeof(float) * Nnzperrow * m);
  //int *rowptr = (int *)malloc(sizeof(int) *(m+1));
  //int *colInd = (int *)malloc(sizeof(int) * Nnzperrow * m);

  I[0] = 0;
  for (i=0; i<m; i++)
  {
    I[i+1] = I[i] + nnzperrow;
    for (j=0; j < nnzperrow; j++)
    {
      int ind = I[i]+j;
      J[ind] = rand()%d;
      //float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX); 
      V[ind] = (float)rand()/(float)(RAND_MAX); 
      //V[ind] = r;
      //printf("J[%d] = %d , V[%d] = %f \n", ind, J[ind], ind, V[ind]);
    }
  }

  //V = &
  //*values = V;
  //*rowptr = I;
  //*colInd = J;
  nnz = m*nnzperrow;








  //nnz = gen_Spdata(m, d, nnzperrow, V, I, J);

  /*
  float values[] = {1, 2, 3, 4, 5, 6};
  int colIdx[] = {0, 2, 2, 0, 1, 2};
  int rowPtr[] = {0, 2, 3, 6};
  int nnz = 6;
  */

 //int size = 6;
  
  //tmp_val = (float *)malloc(nnz*sizeof(float));
  int nnzC;
  //tmp_col = (int *)malloc(nnz*nnz*sizeof(int));
  //tmp_row = (int *)malloc((m+1)*sizeof(int));
  
  C = (float *)malloc(sizeof(float) * m * m); 
  
  
  checkCudaErrors(hipsparseCreate(&handle));
 
  // init device arrays

  //checkCudaErrors(hipMalloc((void **)&dA, sizeof(float) * m * d));
  checkCudaErrors(hipMalloc((void **)&dC, sizeof(float) * m * m));

  // descriptors
  checkCudaErrors(hipsparseCreateMatDescr(&Adescr));
  //checkCudaErrors(hipsparseCreateMatDescr(&Bdescr));
  checkCudaErrors(hipsparseCreateMatDescr(&Cdescr));
  
  checkCudaErrors(hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  //checkCudaErrors(hipsparseSetMatType(Bdescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  checkCudaErrors(hipsparseSetMatType(Cdescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  
  checkCudaErrors(hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO));
  //checkCudaErrors(hipsparseSetMatIndexBase(Bdescr, HIPSPARSE_INDEX_BASE_ZERO));
  checkCudaErrors(hipsparseSetMatIndexBase(Cdescr, HIPSPARSE_INDEX_BASE_ZERO));

  // init csr format for input A and output C
  checkCudaErrors(hipMalloc((void **)&dCsrValA, sizeof(float) * nnz));
  checkCudaErrors(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (m+1)));
  checkCudaErrors(hipMalloc((void **)&dCsrColIndA, sizeof(int) * nnz));

  //checkCudaErrors(hipMalloc((void **)&dCsrValB, sizeof(float) * nnz));
  //checkCudaErrors(hipMalloc((void **)&dCsrRowPtrB, sizeof(int) * (m+1)));
  //checkCudaErrors(hipMalloc((void **)&dCsrColIndB, sizeof(int) * nnz));

  checkCudaErrors(hipMalloc((void **)&dCsrRowPtrC, sizeof(int) * (m+1)));
  //checkCudaErrors(hipMalloc((void **)&dC, sizeof(float) * m * m));

  // copy input to device
  //checkCudaErrors(hipMemcpy(&dA, A, sizeof(float) * m * d, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dCsrValA, V, sizeof(float) * nnz, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dCsrRowPtrA, I, sizeof(int) * (m+1), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dCsrColIndA, J, sizeof(int) * nnz, hipMemcpyHostToDevice));
  
  //checkCudaErrors(hipMemcpy(dCsrValB, values, sizeof(float) * nnz, hipMemcpyHostToDevice));
  //checkCudaErrors(hipMemcpy(dCsrRowPtrB, rowPtr, sizeof(int) * (m+1), hipMemcpyHostToDevice));
  //checkCudaErrors(hipMemcpy(dCsrColIndB, colIdx, sizeof(int) * nnz, hipMemcpyHostToDevice));

  // timer  
  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));
  checkCudaErrors(hipEventCreate(&t2)); 
  checkCudaErrors(hipEventCreate(&t3)); 
  


  // set the device output array init value 
  //checkCudaErrors(hipMemset(dC, 0, sizeof(float) * m * m));

  // get the number of nonzeros for dense2csr
  //checkCudaErrors(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, m, m, Adescr, dA, m, dANnzPerRow, &trueTotalAnnz)); 

  int *nnzTotalDevHostPtr = &nnzC;
  checkCudaErrors(hipEventRecord(t0, 0));
  
  checkCudaErrors(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, m, m, d,
                                      Adescr, nnz, dCsrRowPtrA, dCsrColIndA, 
                                      Adescr, nnz, dCsrRowPtrA, dCsrColIndA,
                                      Cdescr, dCsrRowPtrC, nnzTotalDevHostPtr));

  nnzC = *nnzTotalDevHostPtr;
  
  
  checkCudaErrors(hipMalloc((void **)&dCsrValC, sizeof(float) * nnzC));
  checkCudaErrors(hipMalloc((void **)&dCsrColIndC, sizeof(int) * nnzC));
                                       
                                   
  // transfer the dense 2 csr
  //checkCudaErrors(hipsparseSdense2csr(handle, m, d, Adescr, dA, m, dANnzPerRow, dCsrValA, dCsrRowPtrA, dCsrColIndA));
 
  // timer
  checkCudaErrors(hipEventRecord(t1, 0));
  checkCudaErrors(hipEventSynchronize(t1)); 
  checkCudaErrors(hipEventElapsedTime(&del_t1, t0, t1));
  
  //checkCudaErrors(hipMalloc((void **)&dCsrRowPtrC, sizeof(int) * (m+1)));
  checkCudaErrors(hipMemcpy(dC, C, sizeof(float) * m *m, hipMemcpyHostToDevice));
  // SpGeMM 
  checkCudaErrors(hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, m, m, d, 
                                  Adescr, nnz, dCsrValA, dCsrRowPtrA, dCsrColIndA, 
                                  Adescr, nnz, dCsrValA, dCsrRowPtrA, dCsrColIndA, 
                                  Cdescr, dCsrValC, dCsrRowPtrC, dCsrColIndC)); 
 
  // timer
  checkCudaErrors(hipEventRecord(t2, 0));
  checkCudaErrors(hipEventSynchronize(t2)); 
  hipDeviceSynchronize();
  checkCudaErrors(hipEventElapsedTime(&del_t2, t1, t2));


  //checkCudaErrors(hipMemcpy(tmp_col, dCsrRowPtrC, sizeof(int) * (m+1), hipMemcpyDeviceToHost)); 
  //tmp_val = (float *)malloc(sizeof(float)*nnzC);
  //checkCudaErrors(hipMemcpy(tmp_val, dCsrValC, sizeof(float)*nnzC, hipMemcpyDeviceToHost)); 
  
     

  //checkCudaErrors(hipMemcpy(tmp_row, dCsrRowPtrC, sizeof(int) * (m+1), hipMemcpyDeviceToHost)); 



  // csr 2 dense 
  checkCudaErrors(hipsparseScsr2dense(handle, m, m, Cdescr, dCsrValC, dCsrRowPtrC, dCsrColIndC, dC, m));
  
  checkCudaErrors(hipEventRecord(t3, 0));
  checkCudaErrors(hipEventSynchronize(t3)); 
  checkCudaErrors(hipEventElapsedTime(&del_t3, t2, t3));
  

  //measure nonzeros
	//checkCudaErrors(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, m,m, Cdescr, dC, m, dCNnzPerRow, dCtotalNnz)); 
  
  // device to host                    
  checkCudaErrors(hipMemcpy(C, dC, sizeof(int)* m * m, hipMemcpyDeviceToHost));


  // measure time 

  // output
  float tot_t = del_t1 + del_t2 + del_t3;
  
  t_1 += del_t1;
  t_2 += del_t2;
  t_3 += del_t3;
  wtime += tot_t;


  /*
  for (int i = 0; i < nnz; i++)
  { 
      printf(" %4f ", V[i]);
  }
  printf(" \n\n");

  for (int i = 0; i < m; i++)
  { 
    for (int j= 0; j < m; j++)
    {
      printf(" %f ", C[i*m + j]);
    }
    printf(" \n");
  }
  
  */

  free(C);
  free(V);
  free(I);
  free(J);
  //free(A);
  //checkCudaErrors(hipFree(dA));
  //free(values);
  //free(rowPtr);
  //free(colIdx);
  checkCudaErrors(hipFree(dCsrValA));
  checkCudaErrors(hipFree(dCsrRowPtrA));
  checkCudaErrors(hipFree(dCsrColIndA));
  /* 
  checkCudaErrors(hipFree(dCsrValB));
  checkCudaErrors(hipFree(dCsrRowPtrB));
  checkCudaErrors(hipFree(dCsrColIndB));
  */
  checkCudaErrors(hipFree(dCsrValC));
  checkCudaErrors(hipFree(dCsrRowPtrC));
  checkCudaErrors(hipFree(dCsrColIndC));
  checkCudaErrors(hipFree(dC));
  //checkCudaErrors(hipEventDestroy(t0));
  //checkCudaErrors(hipEventDestroy(t1));
  //checkCudaErrors(hipEventDestroy(t2));
  checkCudaErrors(hipsparseDestroyMatDescr(Adescr));
  checkCudaErrors(hipsparseDestroyMatDescr(Cdescr));
	checkCudaErrors(hipsparseDestroy(handle));

  }
  printf("Get nnz of output : %.2f \n", t_1); 
  printf("SpGeMM : %.2f \n", t_2); 
  printf("csr to dense : %.2f \n", t_3); 
  printf("total time : %.2f \n", wtime);
  return 0;
}

