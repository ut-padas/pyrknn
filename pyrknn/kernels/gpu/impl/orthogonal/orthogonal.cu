#include "util_gpu.hpp"
#include <hipsolver.h>

#define CHECK_SOLVER(func) {                \
  hipsolverStatus_t stat = (func);           \
  assert(HIPSOLVER_STATUS_SUCCESS == stat);  \
}


void orthogonal(fvec &A, int m, int n) {

  assert(n <= m);
  float *d_A = thrust::raw_pointer_cast(A.data());

  hipsolverHandle_t cusolverH = NULL;
  CHECK_SOLVER( hipsolverDnCreate(&cusolverH) );
 
  int lwork_geqrf = 0;
  CHECK_SOLVER( hipsolverDnSgeqrf_bufferSize(
        cusolverH,
        m,
        n,
        d_A,
        m,
        &lwork_geqrf) );

  int lwork_orgqr = 0;
  float *d_tau = NULL;
  CHECK_CUDA( hipMalloc((void**)&d_tau, sizeof(float)*n) );
  CHECK_SOLVER( hipsolverDnSorgqr_bufferSize(
        cusolverH,
        m,
        n,
        n,
        d_A,
        m,
        d_tau,
        &lwork_orgqr) );

  int lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
  float *d_work = NULL;
  CHECK_CUDA( hipMalloc((void**)&d_work, sizeof(float)*lwork) );

  int *devInfo = NULL;
  CHECK_CUDA( hipMalloc((void**)&devInfo, sizeof(int)) );
  CHECK_SOLVER( hipsolverDnSgeqrf(
        cusolverH,
        m,
        n,
        d_A,
        m,
        d_tau,
        d_work,
        lwork,
        devInfo) );

  // check if QR is successful or not
  int info_gpu = 0;
  CHECK_CUDA( hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
  assert(0 == info_gpu);

  CHECK_SOLVER( hipsolverDnSorgqr(
        cusolverH,
        m,
        n,
        n,
        d_A,
        m,
        d_tau,
        d_work,
        lwork,
        devInfo) );
  CHECK_CUDA( hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
  assert(0 == info_gpu);

  // free resource
  if (d_tau  ) CHECK_CUDA( hipFree(d_tau) );
  if (devInfo) CHECK_CUDA( hipFree(devInfo) );
  if (d_work ) CHECK_CUDA( hipFree(d_work) );

  if (cusolverH) CHECK_SOLVER( hipsolverDnDestroy(cusolverH) );
}


void orthogonal_gpu(float *hA, int m, int n) {
  fvec dA(hA, hA+m*n);
  orthogonal(dA, m, n);
  thrust::copy_n(dA.begin(), m*n, hA);
}


