#include "hip/hip_runtime.h"

#include <stdio.h> 
#include <stdlib.h>
//#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "knn_seq.cpp"



__global__ void compute_norm(int* R, int* C, float* V, int* G_Id, float* Norms, int ppl, int leaf_batch_g) {

  int row = threadIdx.x;
  int leaf_id_g = leaf_batch_g * gridDim.x + blockIdx.x;
  
  int g_rowId = leaf_id_g * ppl + row;

  int g_Id = G_Id[g_rowId]; 
  int ind0_i = R[g_Id];
 
  int nnz = R[g_Id + 1] - ind0_i;
  float norm_i = 0.0;
  
  for (int n_i = 0; n_i < nnz; n_i++) norm_i += V[ind0_i + n_i] * V[ind0_i + n_i];
  int ind_write = blockIdx.x * ppl + row;
  Norms[ind_write] = norm_i;

}

__global__ void knn_kernel(int* R, int* C, float* V, int* G_Id, float* Norms , int k_nn, float* KNN_dist, int* KNN_ID, int ppl, int leaf_batch_g, int max_nnz, int m, bool tri_part, int blockInd){

    if (tri_part){

      if (ind < m*(m+1)/2){
      int leaf_id_g = leaf_batch_g * gridDim.y + blockIdx.y;
      
      int ind = threadIdx.x;
      int block = blockIdx.x;

      float tmp = -8 * ind + 4 * m * (m + 1) - 7;
      int i = sqrt(tmp)/2.0 - 0.5;
      i = m - 1 - i;
      int j = ind + i - m * (m+1)/2 + (m - i) * ((m - i) + 1)/2;
      
      int g_rowId = leaf_id_g * ppl + block * m + i;
      int g_colId = leaf_id_g * ppl + block * m + j;

      int perm_i = G_Id[g_rowId];
      int perm_j = G_Id[g_colId];

      int ind0_i = R[perm_i];
      int ind1_i = R[perm_i + 1];

      int ind0_j = R[perm_j];
      int ind1_j = R[perm_j + 1];
     
      int nnz_i = ind1_i - ind0_i;
      int nnz_j = ind1_j - ind0_j;
     
      
      float norm_ij = 0.0;
      if (nnz_i > 256 || nnz_j > 256) printf("Exceeding the max nnz/pt \n");  
      //__shared__ int si[4096];
      __shared__ int SM[8192];
      __shared__ float SM_dist[4096];

      int ind_read_norm_I = blockIdx.y * ppl + block * m + i;
      int ind_read_norm_J = blockIdx.y * ppl + block * m + j;

      norm_ij += Norms[ind_read_norm_I] + Norms[ind_read_norm_J];

      int shift_i = max_nnz * i;
      //int read_pt = ind / m;
      //int read_pt = ind - read_pt * m;     


      // TODO: this reading should be balanced among the threads
      for (int n_i = j - i; n_i < nnz_i; n_i += m - i) SM[shift_i + n_i] = C[ind0_i + n_i];

      __syncthreads();

      float c_tmp = 0.0;
      float c;
      int tmp_0, tmp_1, ind_jk, k, ret, testInd;
 
      ret = 0;
      testInd = 0;

      // loop over the elements of j

      for (int pos_k = 0; pos_k < nnz_j; pos_k++)
        k = SM[max_nnz * j + pos_k];

        // Binary search
        for (int l = nnz_i - ret; l > 1; l /= 2){
          tmp_0 = ret + l;
          tmp_1 = nnz_i - 1;
          testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
          ret = (SM[testInd + shift_i] <= k) ? testInd : ret;
        }
        
        tmp_0 = ret + 1;
        tmp_1 = nnz_i - 1;
        testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;
        ret = (SM[testInd + shift_i] <= k) ? testInd : ret; 
        ind_jk = (SM[ret + shift_i] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;
       
      c_tmp = -2 * c_tmp + norm_ij;
      c_tmp = ( c > 0) ? sqrt(c) : 0.0;
      
      __syncthreads();

      SM_dist[i * 2 * m + j] = c_tmp;
      SM[i * 2 * m + j] = G_Id[leaf_id_g * ppl + j];
      
      SM_dist[j * 2 * m + i] = c_tmp;
      SM[j * 2 * m + i] = G_Id[leaf_id_g * ppl + i];

       
      }
      
      int row = ind / m;
      int col = ind - m * row;
 
      SM_dist[row * 2 * m + col + m] = (col >= m && col < m+k_nn) ? KNN_dist[leaf_id_g * ppl * k_nn + block * m * k_nn + col] : if (col >= m+k_nn ) 1e30;
      SM_Id[row * 2 * m + col + m] = (col >= m && col < m+k_nn) ? KNN_Id[leaf_id_g * ppl * k_nn + block * m * k_nn + col] : if (col >= m+k_nn )  0;
      


      // bitonic sort 


      float tmp_f; 
      int tmp_i;
      int size = 2 *m;
      for (int g = 2; g <= size; g *= 2){
        for (int l = g/2; l > 0; l /= 2){
          int ixj = col ^ l;
               
          if (ixj > col){
            if(( col & g) == 0){
              if (SM_dist[col] > SM_dist[ixj]){
                
                tmp_f = SM_dist[ixj];
                SM_dist[ixj] = SM_dist[col];
                SM_dist[col] = tmp_f;
                
                tmp_i = SM[ixj];
                SM[ixj] = SM[col];
                SM[col] = tmp_i;
              }
           } else {
              if (SM_dist[col] < SM_dist[ixj]){
                
                tmp_f = SM_dist[ixj];
                SM_dist[ixj] = SM_dist[col];
                SM_dist[col] = tmp_f;
                
                tmp_i = SM[ixj];
                SM[ixj] = SM[col];
                SM[col] = tmp_i;
              }
           }
         }
       __syncthreads();
       }
     }



   __syncthreads();
   if (col < k_nn){
     KNN_dist[leaf_id_g * ppl * k_nn + block * m * k_nn + col] = SM_dist[col] 
     KNN_Id[leaf_id_g * ppl * k_nn + block * m * k_nn + col] = SM_Id[col] 

   }




   } else {

   int i = threadIdx.x;
   int j = threadIdx.y;

   //block = blockIdx.x;
   float tmp = -8 * blockInd + 4 * m * (m+1) - 7;
   int b_i = sqrt(tmp) / 2.0 - 0.5;
   b_i = m - 1 - b_i;
   int b_j = blockInd + b_i - m * (m+1)/2 + (m - b_i) * (( m - b_i) + 1)/2;
   
   int leaf_id_g = leaf_batch_g * gridDim.y + blockIdx.y;
   
   int g_rowId_I = leaf_id_g * ppl + b_i * m + i;
   int g_rowId_J = leaf_id_g * ppl + b_j * m + j;

   int perm_i = G_Id[g_rowId_I];
   int perm_j = G_Id[g_rowId_J];

   int ind0_i = R[perm_i];
   int ind1_i = R[perm_i+1];

   int ind0_j = R[perm_j];
   int ind1_j = R[perm_j+1];

   int nnz_i = ind1_i - ind0_i;
   int nnz_j = ind1_j - ind0_j;

   
   float norm_ij = 0.0;
   
   __shared__ int SM[8192];
   __shared__ float SM_dist[4096];

   norm_ij += Norms[g_rowId_I] + [g_rowId_J];

   int shift_i = max_nnz * i;

   for (int n_i = j; j< nnz_i; n_i += m) SM[shift_i + n_i] = C[ind0_i + n_i];

   __syncthreads();

    
		float c_tmp = 0.0;
		float c;
		int tmp_0, tmp_1, ind_jk, k, ret, testInd;

		ret = 0;
		testInd = 0;

		// loop over the elements of j

		for (int pos_k = 0; pos_k < nnz_j; pos_k++)
			k = SM[max_nnz * j + pos_k];

			// Binary search
			for (int l = nnz_i - ret; l > 1; l /= 2){
				tmp_0 = ret + l;
				tmp_1 = nnz_i - 1;
				testInd = (tmp_0 < tmp_1) ? tmp_0 : tmp_1;
				ret = (SM[testInd + shift_i] <= k) ? testInd : ret;
			}

			tmp_0 = ret + 1;
			tmp_1 = nnz_i - 1;
			testInd = (tmp_0 < tmp_1 ) ? tmp_0 : tmp_1;
			ret = (SM[testInd + shift_i] <= k) ? testInd : ret;
			ind_jk = (SM[ret + shift_i] == k) ? ret : -1;
			c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k] * V[ind0_i + ind_jk] : 0;

		c_tmp = -2 * c_tmp + norm_ij;
		c_tmp = ( c > 0) ? sqrt(c) : 0.0;

    __syncthreads();
      
    // horizontal merge

    SM_dist[i * 2*m + j] = c_tmp;
    SM[i * 2*m + j] =  G_Id[g_rowId_J];

    SM_dist[i * 2 * m + j + m] = (j < k_nn) ? KNN_dist[leaf_id_g * ppl * k_nn + b_j * m * k_nn + j] : 1e30;
    SM[i * 2 * m + j + m] = (j < k_nn) ? KNN_dist[leaf_id_g * ppl * k_nn + b_j * m * k_nn + j] : 1e30;

    // bitonic sort


    float tmp_f;
    int tmp_i;
    int size = 2 * m;
      int j_tmp = i * 2 * m + j;

      for (int g = 2; g <= size; g *= 2){
        for (int l = g/2; l > 0; l /= 2){

          int ixj = j ^ l;
          int ixj_tmp = i * 2*m + ixj;

          if (ixj > j){
            if(( j & g) == 0){
              if (SM_dist[j_tmp] > SM_dist[ixj_tmp]){

                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[j_tmp];
                SM_dist[j_tmp] = tmp_f;

                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[j_tmp];
                SM[j_tmp] = tmp_i;
              }
           } else {
              if (SM_dist[j_tmp] < SM_dist[ixj_tmp]){

                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[j_tmp];
                SM_dist[j] = tmp_f;

                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[j_tmp];
                SM[j_tmp] = tmp_i;
              }
           }
         }
       __syncthreads();
       }
     }



   __syncthreads();
   if (j < k_nn){
     KNN_dist[leaf_id_g * ppl * k_nn + b_j * m * k_nn + j] = SM_dist[j_tmp]
     KNN_Id[leaf_id_g * ppl * k_nn + b_j * m * k_nn + j] = SM_Id[j_tmp]
   } 

   // vertical merge

   SM_dist[j * 2*m + i] = c_tmp;
   SM[j * 2*m + i] =  G_Id[g_rowId_I];

   SM_dist[j * 2 * m + i + m] = (i < k_nn) ? KNN_dist[leaf_id_g * ppl * k_nn + b_i * m * k_nn + i] : 1e30;
   SM[j * 2 * m + i + m] = (i < k_nn) ? KNN_dist[leaf_id_g * ppl * k_nn + b_i * m * k_nn + i] : 1e30;

   
   // bitonic sort

    float tmp_f;
    int tmp_i;
    int size = 2 * m;

      int i_tmp = j * 2 * m + i;
      
      for (int g = 2; g <= size; g *= 2){
        for (int l = g/2; l > 0; l /= 2){

          int ixj = i ^ l;
          int ixj_tmp = j * 2 * m + ixj;

          if (ixj > i){
            if(( i & g) == 0){
              if (SM_dist[i_tmp] > SM_dist[ixj_tmp]){

                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[i_tmp];
                SM_dist[i_tmp] = tmp_f;

                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[i_tmp];
                SM[i_tmp] = tmp_i;
              }
           } else {
              if (SM_dist[i_tmp] < SM_dist[ixj_tmp]){

                tmp_f = SM_dist[ixj_tmp];
                SM_dist[ixj_tmp] = SM_dist[i_tmp];
                SM_dist[i_tmp] = tmp_f;

                tmp_i = SM[ixj_tmp];
                SM[ixj_tmp] = SM[i_tmp];
                SM[i_tmp] = tmp_i;
              }
           }
         }
       __syncthreads();
       }
     }



   __syncthreads();
   if (i < k_nn){
     KNN_dist[leaf_id_g * ppl * k_nn + b_j * m * k_nn + i] = SM_dist[i_tmp]
     KNN_Id[leaf_id_g * ppl * k_nn + b_j * m * k_nn + i] = SM_Id[i_tmp]
   } 
   

   }



    
}

__global__ void find_neighbor(float* knn, int* knn_Id, float* K, int* G_Id, int k, int ppl, int m, int leaf_batch_g, int M){
    int col_Id = threadIdx.x; 
    int row_Id = blockIdx.x;

    if (row_Id >= M || col_Id >= M) return;
 
    __shared__ float Dist[2048];
    __shared__ int Dist_Id[2048];

    int size = blockDim.x;
    int leaf_id_g = leaf_batch_g * gridDim.y + blockIdx.y; 
    int ind_K = blockIdx.y * ppl * ppl + row_Id * ppl + col_Id; 
    int i = col_Id;
         
    Dist[col_Id] = K[ind_K];
    Dist_Id[col_Id] = G_Id[leaf_id_g * ppl + col_Id];
    int ind_shared = col_Id;
    //printf("leaf = %d, (%d,%d) , val = %.4f, ind = %d \n" , leaf_id_g, row_Id, leaf_id_g*ppl + col_Id, Dist[col_Id], ind_K);
    __syncthreads();
    //if (leaf_id_g == 0 && row_Id == 10 && col_Id < k) printf("(%d, %d) , val = %.4f , id = %d, ind_read = %d\n", row_Id, col_Id, Dist[col_Id], Dist_Id[col_Id], ind_K);

    // bitonic sort
    float tmp_f;
    int tmp_i;
    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
      int ixj = i ^ l;
      int ixj_tmp = ixj;
      if (ixj > i){
        if ((i & g) == 0){
          if (Dist[ind_shared] > Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        } else {
          if (Dist[ind_shared] < Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        }
        }
      
      __syncthreads();
      }
    }


    size = 2*k;
 
    int ind_knn = leaf_id_g * ppl * k + row_Id * k + col_Id;

    if (col_Id >= k && col_Id < size) Dist[col_Id] = 1.0e30;
    if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = 0;
    
    // should be replaced for the correct knn
    //if (col_Id >= k && col_Id < size) Dist[col_Id] = knn[ind_knn];
    //if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = knn_Id[ind_knn];

  __syncthreads();
  //if (col_Id >= k && col_Id < size) printf("k = %d , size = %d, leaf = %d, row = %d , col = %d , val = %.4f, \n" , k, size, leaf_id_g, row_Id, col_Id, Dist[col_Id]);
	for (int g = 2; g <= size; g *= 2){
		for (int l = g/2; l>0; l /= 2){
		int ixj = i ^ l;
		if (ixj > i){
			if ((i & g) == 0){
				if (Dist[col_Id] > Dist[ixj]){

						 tmp_f = Dist[ixj];
						 Dist[ixj] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
 
						 tmp_i = Dist_Id[ixj];
						 Dist_Id[ixj] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			} else {
				if (Dist[col_Id] < Dist[ixj]){

						 tmp_f = Dist[ixj];
						 Dist[ixj] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;

						 tmp_i = Dist_Id[ixj];
						 Dist_Id[ixj] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			}
			}
	  //if (leaf_id_g == 1 && row_Id == 2) printf("C[%d] = %.4f \n", col_Id, Dist[col_Id]);
    
		__syncthreads();
		//if (row_Id == 2 && col_Id == 0) printf("\n");
		__syncthreads();
    }
    
}
    //if (col_Id < size) printf("leaf = %d, row = %d , col = %d , val = %.4f, \n" , leaf_id_g, row_Id, col_Id, Dist[col_Id]);

    if (col_Id < k){
      knn[ind_knn] = Dist[col_Id];
      knn_Id[ind_knn] = Dist_Id[col_Id];
      //printf("leaf = %d, row = %d , col = %d , val = %.4f, ind_knn = %d \n" , leaf_id_g, row_Id, col_Id, Dist[col_Id], ind_knn );
    }

}



void gen_sparse(int M, int tot_nnz, int d, int *R, int *C, float *V) {
 
    int nnz_row;
    int val;

    for (int i=0; i < M; i++){
      nnz_row = R[i+1] - R[i];
      for (int j=0; j < nnz_row; j++){
          int ind = R[i]+j; 
          val = rand()%d;
          //val = rand()%d;
          C[ind] = val;
          //V[ind] = (rand()%1000)/30;
          V[ind] = ((float) rand()) / (float) RAND_MAX;
        }    
      std::sort(C+R[i], C+(R[i+1]));
    }
}

void gen_R(int M, int nnzperrow, int *R, int *G_Id, int d) {  
  R[0] = 0;
  int tot_nnz = 0;
  int val;
  for (int m =1; m <= M; m++){ 
   //val = 1 + rand()%(2*nnzperrow);
   val = nnzperrow; //+ rand()%nnzperrow;
   if (val > d) val = 1; 
   tot_nnz += val;
   R[m] = tot_nnz;
   G_Id[m-1] = m-1;
  } 
  std::random_shuffle(&G_Id[0], &G_Id[M]);
  /*
  for (int m = 0; m < M; m++){ 
  printf("G_Id[%d] = %d \n", m , G_Id[m]);
  } 
  */
}

void gpu_knn(int *R, int *C, float *V, int *G_Id, int M, int leaves, int k, float *knn, int *knn_Id, int max_nnz){
 
	int ppl = M/leaves;
	int m = 8192 / max_nnz;
  m = min(m, ppl);
  
  if (m > 32){ 
    m = 32; 
  } 


	int num_batch_I = (ppl + m - 1) / m;
	int num_batch_J = (ppl + m - 1) / m;

  size_t free, total;
  hipMemGetInfo(&free, &total);
  int log_size = log2(free / (sizeof(float)));
  double arr_len = pow(2, log_size); 

  int size_batch_leaves = arr_len / (ppl * ppl);
  if (size_batch_leaves > leaves) size_batch_leaves = leaves;
  int num_batch_leaves = (leaves + size_batch_leaves - 1) / size_batch_leaves;

  //printf("%d , %d  , %d \n", num_batch_I, num_batch_J, num_batch_leaves);
  float del_t1;
  hipEvent_t t0; 
  hipEvent_t t1;
  int blocks = m*2;
  int num_blocks = ppl/m;
  dim3 dimBlock_tri(blocks, 1);	
  dim3 dimGrid_tri(num_blocks, leaves); 
  
  num_blocks = (m-1)*(m)/2;
  dim3 dimBlock_sq(m, m);	
  dim3 dimGrid_sq(1, leaves); 
  
  dim3 dimBlock_norm(ppl);	
  dim3 dimGrid_norm(leaves); 
  
  float *d_Norms;
  
  checkCudaErrors(hipMalloc((void **) &d_Norms, sizeof(float) * ppl * size_batch_leaves));


  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));

  checkCudaErrors(hipEventRecord(t0, 0));
  compute_norm <<< dimGrid_norm, dimBlock_norm >>>(R, C, V, G_Id, d_Norms, ppl, leaf_id_g);
  
  for (int blockInd = 0; blockInd < num_blocks; blockInd++){  
    checkCudaErrors(hipDeviceSynchronize());
    if (blockInd == 0) knn_iter <<< dimGrid, dimBlock >>>(R, C, V, G_Id, d_Norms, k, knn, knn_Id, ppl, 0, max_nnz, m , true, blockInd);
    checkCudaErrors(hipDeviceSynchronize());
    knn_iter <<< dimGrid, dimBlock >>>(R, C, V, G_Id, d_Norms, k, knn, knn_Id, ppl, 0, max_nnz, m , false, blockInd);
  } 
  
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(t1, 0));
  checkCudaErrors(hipEventSynchronize(t1));
  checkCudaErrors(hipEventElapsedTime(&del_t1, t0, t1));
 
  printf("# leaves : %d \n", leaves);
  printf("# points/leaf : %d \n", ppl);
  printf("  max_nnz : %d \n", max_nnz);
  
  printf("\n Elapsed time (s) : %.4f \n ", del_t1/1000);
  printf(" # points = %d" , M);
 
  checkCudaErrors(hipFree(d_K));
  checkCudaErrors(hipEventDestroy(t0));
  checkCudaErrors(hipEventDestroy(t1));

}





int main(int argc, char **argv)
{

    checkCudaErrors(hipSetDevice(0));

    int d, nnzperrow;
    float *h_V, *d_V;
    int *h_C, *d_C;
    int *h_R, *d_R;
    int *h_G_Id, *d_G_Id;
    int M = 1024*2048;
    int leaves = 2048;
    d = 100000;
    int k = 32;
    nnzperrow = 256;
    int max_nnz = nnzperrow;
    int leaf_size = M / leaves; 
    

    bool print_pt = false;    
    bool print_res = false;    
    int test_leaf = 1000;    
    int test_pt = 1000;

    int *d_knn_Id, *h_knn_Id, *h_knn_Id_seq;
    float *d_knn, *h_knn, *h_knn_seq;

    h_R = (int *)malloc(sizeof(int)*(M+1));
    h_G_Id = (int *)malloc(sizeof(int)*(M));

    h_knn = (float *)malloc(sizeof(float) * M *k);
    h_knn_seq = (float *)malloc(sizeof(float) * M *k / leaves);
    h_knn_Id = (int *)malloc(sizeof(int) * M *k);
    h_knn_Id_seq = (int *)malloc(sizeof(int) * M *k / leaves);

    // generate random data 
    gen_R(M, nnzperrow, h_R,h_G_Id, d);
    int tot_nnz = h_R[M];
		h_V = (float *)malloc(sizeof(float)*tot_nnz);
    h_C = (int *)malloc(sizeof(int)*tot_nnz);
    gen_sparse(M, tot_nnz, d , h_R, h_C, h_V);   
    if (print_pt){   
    for (int i = 0; i < M; i++){
        int nnz = h_R[i+1] - h_R[i];
        for (int j = 0; j < nnz; j++)
        printf("R[%d] = %d , C[%d] = %d , V[%d] = %.4f \n", i ,h_R[i], h_R[i]+j, h_C[h_R[i] + j], h_R[i]+j, h_V[h_R[i]+j]);
    }    
    }
    checkCudaErrors(hipMalloc((void **) &d_R, sizeof(int)*(M+1)));
    checkCudaErrors(hipMalloc((void **) &d_G_Id, sizeof(int)*(M)));
    checkCudaErrors(hipMalloc((void **) &d_C, sizeof(int)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_V, sizeof(float)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_knn_Id, sizeof(int)*M*k));
    checkCudaErrors(hipMalloc((void **) &d_knn, sizeof(float)*M*k));
 
    checkCudaErrors(hipMemcpy(d_C, h_C, sizeof(int)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_V, h_V, sizeof(float)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_R, h_R, sizeof(int)*(M+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_G_Id, h_G_Id, sizeof(int)*(M), hipMemcpyHostToDevice));

    printf("Random csr is generated  \n");

    gpu_knn(d_R, d_C, d_V, d_G_Id, M, leaves, k, d_knn, d_knn_Id, max_nnz);
    
    checkCudaErrors(hipMemcpy(h_knn, d_knn, sizeof(float) * M * k, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_knn_Id, d_knn_Id, sizeof(int) * M * k, hipMemcpyDeviceToHost));
  

    printf(" \n running Seq knn \n");
    printf("\n test for leaf %d , pt %d\n",test_leaf, test_pt);  

    f_knnSeq(h_R, h_C, h_V, h_G_Id, h_knn_seq, h_knn_Id_seq, k, test_leaf, test_pt, leaf_size);
    
    float acc= 0.0;  

    
    int ind;
    bool match;
    int counter = 0;
    int gpu_pt,seq_pt,ind_seq,ind_gpu, nnz_gpu,nnz_seq;
    int ind0_i = h_R[h_G_Id[test_leaf * leaf_size + test_pt]];
    int nnz_i = h_R[h_G_Id[test_leaf * leaf_size + test_pt] + 1] - ind0_i;
    //for (int i=0; i < nnz_i; i++) printf("[(%d, %d, %.4f)] \n", h_G_Id[test_leaf * leaf_size + test_pt], h_C[ind0_i + i], h_V[ind0_i + i]);

    for (int i = 0; i < k; i++){
      ind = test_leaf * k * leaf_size + test_pt * k + i;
      match = (h_knn_Id_seq[test_pt*k + i] == h_knn_Id[ind]);
      if (print_res){
      printf("seq ind %d,\t gpu_ind %d , \t match %d , \t v_seq %.4f, \t v_gpu %.4f , \t ind = %d\n", h_knn_Id_seq[test_pt*k + i], h_knn_Id[ind], match, h_knn_seq[test_pt*k + i], h_knn[ind], ind);
      }
      if (match) acc += 1.0;
      if (counter < 2 && match==0) {
        counter++;
		    gpu_pt = h_knn_Id[ind];
        seq_pt = h_knn_Id_seq[test_pt * k + i];
        ind_gpu = h_R[gpu_pt];
        ind_seq = h_R[seq_pt];
        nnz_gpu = h_R[gpu_pt + 1]  - h_R[gpu_pt];
        nnz_seq = h_R[seq_pt + 1]  - h_R[seq_pt]; 
        //printf("gpu pt %d \n", gpu_pt); 
       
        //for (int q=0; q < nnz_gpu; q++) printf("[(%d, %d, %.4f)] \n", gpu_pt, h_C[ind_gpu + q], h_V[ind_gpu + q]);
        //printf("\n seq pt %d \n", seq_pt); 
       
        //for (int q=0; q < nnz_seq; q++) printf("[(%d, %d, %.4f)] \n", seq_pt, h_C[ind_seq + q], h_V[ind_seq + q]);
        
		
		}
    }
    
    acc /= k;    
    printf("\n\naccuracy %.4f for leaf %d\n\n", acc*100, test_leaf);
    
    checkCudaErrors(hipFree(d_R));
    checkCudaErrors(hipFree(d_G_Id));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_V));
 
    free(h_R);
    free(h_C);
    free(h_V);
    free(h_G_Id);
    free(h_knn);
    free(h_knn_Id);
    free(h_knn_seq);
    free(h_knn_Id_seq);


}
