#include "hip/hip_runtime.h"

#include <stdio.h> 
#include <stdlib.h>
//#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>






__global__ void compute_dist(int* R, int* C, float* V, float* K, int m_i, int m_j , int k_nn, int M_I, int leaf_id_g, int max_nnz){

    int col_Id = threadIdx.x;
    //int row_Id = blockIdx.x;
    int row_Id = threadIdx.y;
    
    //int blockId_J = threadIdx.y;
    int blockId_J = blockIdx.x;
    int blockId_I = blockIdx.y;
   
   
    //int leaf_id_g = threadIdx.z +blockIdx.z * blockDim.z; 
    //int leaf_id_g = blockIdx.z; 

    //int binsearch_thread = threadIdx.z;
    //int num_thread = blockDim.z; 

    int g_rowId_I = leaf_id_g * M_I + blockId_I * m_i + row_Id;
    int g_rowId_J = leaf_id_g * M_I + blockId_J * m_j + col_Id;
     
    int ind0_i = R[g_rowId_I];
    int ind1_i = R[g_rowId_I + 1];

    int ind0_j = R[g_rowId_J];
    int ind1_j = R[g_rowId_J + 1];
 
    int nnz_i = ind1_i - ind0_i;
    int nnz_j = ind1_j - ind0_j;


    float norm_ij = 0;

    

    //__shared__ int sj[512];
    //__shared__ float vj[512];
    
    //int si[256];
    __shared__ int si[4096];
    //float vi[256];
    //int sj[256];
    //float vj[256];
    int shift = max_nnz*threadIdx.y;
    //if (leaf_id_g == 0) printf("Id_y = %d , shift = %d \n", threadIdx.y , shift);
    
    for (int n_i = 0; n_i < nnz_i; n_i++){
      si[n_i + shift] = C[ind0_i + n_i];
      //vi[n_i] = V[ind0_i + n_i];
      //norm_ij += vi[n_i]*vi[n_i];
      norm_ij += V[ind0_i + n_i]*V[ind0_i + n_i];
    }
    for (int n_j = 0; n_j < nnz_j; n_j++){
      //sj[n_j] = C[ind0_j + n_j];
      //vj[n_j] = V[ind0_j + n_j];
      //norm_ij += vj[n_j]*vj[n_j];
      norm_ij += V[ind0_j + n_j]*V[ind0_j + n_j];
    }
     
     
     
    float c_tmp = 0;
    float c;
    
    int tmp_0, tmp_1, ind_jk, k, ret, testInd; 
    
    ret=0; 
    testInd = 0;

    
    
    for (int pos_k=0; pos_k<nnz_j;pos_k++){       
        //k = sj[pos_k];
        k = C[ind0_j + pos_k];
        //ret = testInd;
           
        // Binary search 
        for (int l=nnz_i-ret; l > 1; l/=2){
            tmp_0 = ret+l;
            tmp_1 = nnz_i-1;
            testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
            ret = (si[testInd+ shift] <= k) ? testInd : ret ;
        }
        tmp_0 = ret+1;
        tmp_1 = nnz_i-1;
        testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
        ret = (si[testInd+ shift] <= k) ? testInd : ret;
        ind_jk = (si[ret+ shift] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k]*V[ind0_i + ind_jk] : 0;
        
    }
    /* 
    for (int pos_k=0; pos_k<nnz_i;pos_k++){       
        k = si[pos_k];
        
        ret=0; 
        testInd = 0;
        
        // Binary search 
        for (int l=nnz_j; l > 1; l/=2){
            tmp_0 = ret+l;
            tmp_1 = nnz_j-1;
            testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
            ret = (sj[testInd] <= k) ? testInd : ret ;
        }
        tmp_0 = ret+1;
        tmp_1 = nnz_j-1;
        testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
        ret = (sj[testInd] <= k) ? testInd : ret;
        ind_jk = (sj[ret] == k) ? ret : -1;
        
        //c_tmp += (ind_jk != -1) ? vi[pos_k]*vj[pos_k] : 0;
        //c_tmp +=  vi[pos_k]*vi[pos_k] ;
        
    }
    */
    c = -2*c_tmp + norm_ij;
    c_tmp = (c > 0) ? c : 0.0;
    c_tmp = sqrt(c_tmp);
    
	  int col_write = blockId_J * m_j + col_Id; 
	  int row_write = blockId_I * m_i + row_Id;
	  int ind_write = row_write * M_I + col_write;
    K[ind_write] = c_tmp;
    //if (leaf_id_g == 2 && row_write == 0) printf("leaf = %d , row = %d , col = %d , val = %.2f \n", leaf_id_g, row_write, col_write , c_tmp);

    /*
    // bitonic sort 
    __shared__ float kvals[2048];
    //__shared__ int id_k[2048];

    int ind_s = threadIdx.y * blockDim.x + threadIdx.x; 
    kvals[ind_s] = c_tmp; 
    //id_k[ind_s] = g_rowId_J;
    si[ind_s] = g_rowId_J;
    
    
    __syncthreads();
     
    int log_size = 0;
    int m_j_tmp = m_j;
    while (m_j_tmp >>= 1) ++log_size;

    //int log_size = log2(m_j);
    int size = (pow(2,log_size) < m_j) ? pow(2, log_size+1) : m_j;
    // bitonic sort  
    float tmp_f;
    int tmp_i;
    //if (g_rowId_I == 13) printf("col = %d , c_tmp = %.2f , ind_s = %d , m_j = %d , m_i = %d \n", g_rowId_J, c_tmp, ind_s, threadIdx.x, threadIdx.y); 
    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
	    int ixj = i ^ l;
      int ixj_tmp = threadIdx.y * blockDim.x + ixj;
	    if (ixj > i){
		    if ((i & g) == 0){
			    if (kvals[ind_s] > kvals[ixj_tmp]){ 
               tmp_f = kvals[ixj_tmp]; 
               kvals[ixj_tmp] = kvals[ind_s]; 
               kvals[ind_s] = tmp_f;
               //tmp_i = id_k[ixj_tmp]; 
               tmp_i = si[ixj_tmp]; 
               si[ixj_tmp] = si[ind_s]; 
               //id_k[ixj_tmp] = id_k[ind_s]; 
               si[ind_s] = tmp_i;
               //id_k[ind_s] = tmp_i;
                }
		    } else {
			    if (kvals[ind_s] < kvals[ixj_tmp]){ 
               tmp_f = kvals[ixj_tmp]; 
               kvals[ixj_tmp] = kvals[ind_s]; 
               kvals[ind_s] = tmp_f;
               //tmp_i = id_k[ixj_tmp]; 
               //id_k[ixj_tmp] = id_k[ind_s]; 
               //id_k[ind_s] = tmp_i;
               tmp_i = si[ixj_tmp]; 
               si[ixj_tmp] = si[ind_s]; 
               si[ind_s] = tmp_i;
                } 
		    }
	      }
	    __syncthreads();
      }
    }
    if (col_Id < k_nn){
	    int col_write = blockId_J * k_nn + col_Id; 
	    int row_write = blockId_I * m_i + row_Id;
	    int ind_write = leaf_id_g * M_I * k_nn + row_write * k_nn + col_write; 
      //printf("leaf_id = %d , row_write = %d , col_write = %d , ind_write = %d \n", leaf_id_g, row_write, col_write , ind_write);
	    K[ind_write] = kvals[ind_s];
	    //K_ID[ind_write] = id_k[ind_s];
	    K_ID[ind_write] = si[ind_s];
    }
    */ 
    
}


//__global__ void find_neighbor(float* knn, int* knn_Id, float* K, int* K_Id, int k, int M_I){
__global__ void find_neighbor(float* knn, int* knn_Id, float* K, int k, int M_I, int m_j, int leaf_id_g){

    int col_Id = threadIdx.x;
    
    int row_Id = blockIdx.x;

    //int blockId_J = threadIdx.y;
    //int blockId_J = blockIdx.x;
    //int blockId_I = blockIdx.y;
    

    //int leaf_id_g = threadIdx.z +blockIdx.z * blockDim.z;

    __shared__ float Dist[2048];
    __shared__ int Dist_Id[2048];

		//int col_write = blockId_J * m_j + col_Id;
		//int row_write = blockId_I * blockDim.y + row_Id;
		//int ind_read = leaf_id_g * M_I * k + row_write * k + col_write;
		//int ind_read = row_write * M_I + col_write;
		int ind_read = row_Id * M_I + col_Id;
    //int ind_shared = row_Id * blockDim.x + col_Id;
    //int ind_knn = leaf_id_g * M_I * k + blockId_I * blockDim.y * k + row_Id * k + col_Id;
    
    int size = blockDim.x;
    
    
    int true_size = size;
    //while (tmp >>= 1) ++log_size;
    size = size - 1;
    size |= size >> 1;    
    size |= size >> 2;    
    size |= size >> 4;    
    size |= size >> 8;    
    size |= size >> 16;    
    //size |= size >> 1;
    size++;    
    
    
    int i = col_Id;
    //if (leaf_id_g == 2047) printf("row = %d , col = %d 0 , ind_shared = %d \n", row_write, col_write, ind_shared);
    //Dist[ind_shared] = (col_Id < k) ? knn[ind_knn] : (col_Id < size) ? K[ind_read] : 1e30;
    Dist[col_Id] = K[ind_read];
    //Dist_Id[ind_shared] = (col_Id < k) ? knn_Id[ind_knn] : (col_Id < size) ? K_Id[ind_read] : 0;
    //Dist_Id[ind_shared] = (col_Id < k) ? 0 : (col_Id < size) ? K_Id[ind_read] : 0;
    Dist_Id[col_Id] = col_Id;
    int ind_shared = col_Id;
    //if (leaf_id_g == 2047) printf("row = %d , col = %d 1 , ind_shared = %d , \n", row_write, col_write, ind_shared);
     
    __syncthreads();

    // bitonic sort
    float tmp_f;
    int tmp_i;
    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
      int ixj = i ^ l;
      int ixj_tmp = ixj;
      if (ixj > i){
        if ((i & g) == 0){
          if (Dist[ind_shared] > Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        } else {
          if (Dist[ind_shared] < Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        }
        }
      
      __syncthreads();
      //if (leaf_id_g == 0 && row_write == 0) printf("i = %d , ixj = %d , Dist[%d] = %.2f \n", i , ixj, i, Dist[i]);
      }
    }
    //if (leaf_id_g == 2 && row_write == 0) printf("i = %d , col = %d , Dist[%d] = %.2f \n", i , col_Id, i, Dist[i]);
    size = 2*k;
    
    //if (col_Id >= k && cold_Id < size) Dist[ind_shared] = knn[ind_knn];
    if (col_Id >= k && col_Id < size) Dist[col_Id] = 1e30;
    if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = 0;

    // merge with knn

    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
      int ixj = i ^ l;
      int ixj_tmp =  ixj;
      if (ixj > i){
        if ((i & g) == 0){
          if (Dist[ind_shared] > Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        } else {
          if (Dist[ind_shared] < Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        }
        }
      
      __syncthreads();
      }
    }
    


    //if (leaf_id_g == 2 && row_write == 0) printf("i = %d , col = %d , Dist[%d] = %.2f \n", i , col_Id, i, Dist[i]);
    if (col_Id < k){
      //col_write = row_Id * k + col_Id;
      //row_write = blockId_I * blockDim.y + row_Id;
      int ind_write = leaf_id_g * M_I * k + row_Id * k + col_Id;
      knn[ind_write] = Dist[col_Id];
      knn_Id[ind_write] = Dist_Id[col_Id];
    }
     
    
}



void gen_sparse(int M, int tot_nnz, int d, int *R, int *C, float *V) {
 
    int nnz_row;
    int val;

    for (int i=0; i < M; i++){
      nnz_row = R[i+1] - R[i];
      for (int j=0; j < nnz_row; j++){
          int ind = R[i]+j; 
          val = rand()%d;
          //val = rand()%d;
          C[ind] = val;
          V[ind] = rand()%100;
        }    
      std::sort(C+R[i], C+(R[i+1]));
    }
}

void gen_R(int M, int nnzperrow, int *R, int d) {  
  R[0] = 0;
  int tot_nnz = 0;
  int val;
  for (int m =1; m <= M; m++){ 
   //val = nnzperrow/2 + rand()%nnzperrow;
   val = nnzperrow; //+ rand()%nnzperrow;
   if (val > d) val = 1; 
   tot_nnz += val;
   R[m] = tot_nnz;
  } 
  
}

void gpu_knn(int *R, int *C, float *V, int M, int leaves, int k, float *knn, int *knn_Id){

  

	int pointsperleaf = M/leaves;
	int m_j = min(64, pointsperleaf);
	int m_i = 16;
	int max_nnz = 200;
	int size_batch_I = (pointsperleaf)/m_i;
	int size_batch_J = (pointsperleaf + m_j - 1) / m_j;
	int M_I = M/leaves;
  float del_t1;
  hipEvent_t t0; 
  hipEvent_t t1;
  
  dim3 dimBlock(m_j, m_i);	
  dim3 dimGrid(size_batch_J, size_batch_I); 
  dim3 dimBlock_n(M_I);
  dim3 dimGrid_n(M_I);


  float *d_K;
  checkCudaErrors(hipMalloc((void **) &d_K, sizeof(int)*pointsperleaf*pointsperleaf));


  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));

  checkCudaErrors(hipEventRecord(t0, 0));
  for (int leaf_id_g = 0; leaf_id_g < leaves; leaf_id_g++){
    compute_dist <<< dimGrid, dimBlock >>>(R, C, V, d_K, m_i, m_j, k, M_I, leaf_id_g, max_nnz);
    checkCudaErrors(hipDeviceSynchronize());
    find_neighbor <<< dimGrid_n, dimBlock_n >>>(knn, knn_Id, d_K, k, M_I, m_j, leaf_id_g);
  } 
  
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(t1, 0));
  checkCudaErrors(hipEventSynchronize(t1));
  checkCudaErrors(hipEventElapsedTime(&del_t1, t0, t1));
  printf("\n Elapsed time (ms) : %.4f \n ", del_t1);

  checkCudaErrors(hipFree(d_K));
  checkCudaErrors(hipEventDestroy(t0));
  checkCudaErrors(hipEventDestroy(t1));

}





int main(int argc, char **argv)
{

  //, del_t2, del_t3;

    checkCudaErrors(hipSetDevice(0));

    int d, nnzperrow;
    float *h_V, *d_V;
    int *h_C, *d_C;
    int *h_R, *d_R;

    int M = 1024*2048;
    int leaves = 2048;
    d = 10000;
    int k = 32;
    nnzperrow = 128;
    
    
    

    int *d_knn_Id;
    float *d_knn;

    h_R = (int *)malloc(sizeof(int)*(M+1));

    // generate random data 
    gen_R(M, nnzperrow, h_R, d);
    int tot_nnz = h_R[M];
		h_V = (float *)malloc(sizeof(float)*tot_nnz);
    h_C = (int *)malloc(sizeof(int)*tot_nnz);
    gen_sparse(M, tot_nnz, d , h_R, h_C, h_V);   
    /*   
    for (int i = 0; i < M; i++){
        int nnz = h_R[i+1] - h_R[i];
        for (int j = 0; j < nnz; j++)
        printf("R[%d] = %d , C[%d] = %d , V[%d] = %.4f \n", i ,h_R[i], h_R[i]+j, h_C[h_R[i] + j], h_R[i]+j, h_V[h_R[i]+j]);
    }    
    */
    checkCudaErrors(hipMalloc((void **) &d_R, sizeof(int)*(M+1)));
    checkCudaErrors(hipMalloc((void **) &d_C, sizeof(int)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_V, sizeof(float)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_knn_Id, sizeof(int)*M*k));
    checkCudaErrors(hipMalloc((void **) &d_knn, sizeof(float)*M*k));
 
    checkCudaErrors(hipMemcpy(d_C, h_C, sizeof(int)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_V, h_V, sizeof(float)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_R, h_R, sizeof(int)*(M+1), hipMemcpyHostToDevice));

    printf("Gen rnd mat finished \n");
    //printf("m_j = %d , size_batch_J = %d , m_i = %d , size_batch_I = %d , blocksize_leaf = %d , batch_leavs = %d \n", 
           //m_j , size_batch_J , m_i , size_batch_I , blocksize_leaf, batch_leaves);

    gpu_knn(d_R, d_C, d_V, M, leaves, k, d_knn, d_knn_Id);
    
    printf("\n\n");
    checkCudaErrors(hipFree(d_R));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_V));
    free(h_R);
    free(h_C);
    free(h_V);


}
