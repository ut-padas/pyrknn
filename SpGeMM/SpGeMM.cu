#include "hip/hip_runtime.h"

#include <stdio.h> 
#include <stdlib.h>
//#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hip/hip_runtime_api.h>





__global__ void compute_dist(int* R, int* C, float* V, int* G_Id,  float* K, int m_i, int m_j , int k_nn, int M_I, int leaf_batch_g, int max_nnz, int M){

    int col_Id = threadIdx.x;
    //int row_Id = blockIdx.x;
    int row_Id = threadIdx.y;
    
    //int blockId_J = threadIdx.y;
    int blockId_J = blockIdx.x;
    int blockId_I = blockIdx.y;
    int leaf_id_g = leaf_batch_g * gridDim.z + blockIdx.z;
    //if (col_Id == 0 && row_Id == 0) printf("Id z = %d ,  gridDim = %d ,  leaf_batch = %d , leaf_id_g = %d \n", blockIdx.z, gridDim.z, leaf_batch_g , leaf_id_g);
    int g_rowId_I = leaf_id_g * M_I + blockId_I * m_i + row_Id;
    int g_rowId_J = leaf_id_g * M_I + blockId_J * m_j + col_Id;
    
    if (g_rowId_I >= M || g_rowId_J >= M) return;
    //if (g_rowId_I >= M || g_rowId_J >= M) printf(" Illegal access for I = %d , J = %d \n", g_rowId_I , g_rowId_J);
    //if (g_rowId_I > M_I*2048 || g_rowId_J > M_I*2048) printf("g_rowID_I = %d , g_row_ID_J = %d , leaf_id_g = %d , M_I = %d , blockId_I = %d , blockID_J = %d , row_Id = %d , col_Id = %d \n", g_rowId_I , g_rowId_J, leaf_id_g , M_I , blockId_I, blockId_J, row_Id , col_Id);
    int g_Id_i = G_Id[g_rowId_I]; 
    int g_Id_j = G_Id[g_rowId_J];     

    //int g_Id_i = g_rowId_I;
    //int g_Id_j = g_rowId_J;

    int ind0_i = R[g_Id_i];
    int ind1_i = R[g_Id_i + 1];

    int ind0_j = R[g_Id_j];
    int ind1_j = R[g_Id_j + 1];
 
    int nnz_i = ind1_i - ind0_i;
    int nnz_j = ind1_j - ind0_j;


    float norm_ij = 0.0;    

    __shared__ int si[8192];

    int shift = max_nnz*threadIdx.y;

    //if (leaf_id_g == 0) printf("Id_y = %d , shift = %d \n", threadIdx.y , shift);
    /*
    for (int n_i = 0; n_i < nnz_i; n_i++){
      if (col_Id == 0) si[n_i + shift] = C[ind0_i + n_i];
      norm_ij += V[ind0_i + n_i]*V[ind0_i + n_i];
    }
    */
    for (int n_j = 0; n_j < nnz_j; n_j++) norm_ij += V[ind0_j + n_j]*V[ind0_j + n_j];
    for (int n_i = 0; n_i < nnz_i; n_i++) norm_ij += V[ind0_i + n_i]*V[ind0_i + n_i];
    for (int n_i = threadIdx.x; n_i < nnz_i; n_i += blockDim.x) si[shift + n_i] = C[ind0_i + n_i];
     
    __syncthreads();
    float c_tmp = 0.0;
    
    int tmp_0, tmp_1, ind_jk, k, ret, testInd; 
    
    ret=0; 
    testInd = 0;

    
    
    for (int pos_k=0; pos_k<nnz_j;pos_k++){       
        k = C[ind0_j + pos_k];
           
        // Binary search 
        for (int l=nnz_i-ret; l > 1; l/=2){
            tmp_0 = ret+l;
            tmp_1 = nnz_i-1;
            testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
            ret = (si[testInd+ shift] <= k) ? testInd : ret ;
        }
        tmp_0 = ret+1;
        tmp_1 = nnz_i-1;
        testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
        ret = (si[testInd+ shift] <= k) ? testInd : ret;
        ind_jk = (si[ret+ shift] == k) ? ret : -1;
        c_tmp += (ind_jk != -1) ? V[ind0_j + pos_k]*V[ind0_i + ind_jk] : 0;
        
    }
    /* 
    for (int pos_k=0; pos_k<nnz_i;pos_k++){       
        k = si[pos_k];
        
        ret=0; 
        testInd = 0;
        
        // Binary search 
        for (int l=nnz_j; l > 1; l/=2){
            tmp_0 = ret+l;
            tmp_1 = nnz_j-1;
            testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
            ret = (sj[testInd] <= k) ? testInd : ret ;
        }
        tmp_0 = ret+1;
        tmp_1 = nnz_j-1;
        testInd = (tmp_0 < tmp_1) ? tmp_0: tmp_1;
        ret = (sj[testInd] <= k) ? testInd : ret;
        ind_jk = (sj[ret] == k) ? ret : -1;
        
        //c_tmp += (ind_jk != -1) ? vi[pos_k]*vj[pos_k] : 0;
        //c_tmp +=  vi[pos_k]*vi[pos_k] ;
        
    }
    */
    c_tmp = -2*c_tmp + norm_ij;
    c_tmp = (c_tmp > 0) ? sqrt(c_tmp) : 0.0;
    //c_tmp = sqrt(c_tmp);
    
	  int col_write = blockId_J * m_j + col_Id; 
	  int row_write = blockId_I * m_i + row_Id;
	  //int ind_write = row_write * M_I + col_write;
	  int ind_write = blockIdx.z * M_I * M_I + row_write * M_I + col_write;
    K[ind_write] = c_tmp;

    
    /*
    // bitonic sort 
    __shared__ float kvals[4096];
    //__shared__ int id_k[2048];

    int ind_s = threadIdx.y * blockDim.x + threadIdx.x; 
    kvals[ind_s] = c_tmp;
    //id_k[ind_s] = g_rowId_J;
    si[ind_s] = col_write;
    
    
    __syncthreads();
     
    int log_size = 0;
    int m_j_tmp = m_j;
    while (m_j_tmp >>= 1) ++log_size;

    //int log_size = log2(m_j);
    int size = (pow(2,log_size) < m_j) ? pow(2, log_size+1) : m_j;
    // bitonic sort  
    float tmp_f;
    int tmp_i, i;
    i = col_Id;
    //if (g_rowId_I == 13) printf("col = %d , c_tmp = %.2f , ind_s = %d , m_j = %d , m_i = %d \n", g_rowId_J, c_tmp, ind_s, threadIdx.x, threadIdx.y); 
    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
	    int ixj = i ^ l;
      int ixj_tmp = threadIdx.y * blockDim.x + ixj;
	    if (ixj > i){
		    if ((i & g) == 0){
			    if (kvals[ind_s] > kvals[ixj_tmp]){ 
               tmp_f = kvals[ixj_tmp]; 
               kvals[ixj_tmp] = kvals[ind_s]; 
               kvals[ind_s] = tmp_f;
               //tmp_i = id_k[ixj_tmp]; 
               tmp_i = si[ixj_tmp]; 
               si[ixj_tmp] = si[ind_s]; 
               //id_k[ixj_tmp] = id_k[ind_s]; 
               si[ind_s] = tmp_i;
               //id_k[ind_s] = tmp_i;
                }
		    } else {
			    if (kvals[ind_s] < kvals[ixj_tmp]){ 
               tmp_f = kvals[ixj_tmp]; 
               kvals[ixj_tmp] = kvals[ind_s]; 
               kvals[ind_s] = tmp_f;
               //tmp_i = id_k[ixj_tmp]; 
               //id_k[ixj_tmp] = id_k[ind_s]; 
               //id_k[ind_s] = tmp_i;
               tmp_i = si[ixj_tmp]; 
               si[ixj_tmp] = si[ind_s]; 
               si[ind_s] = tmp_i;
                } 
		    }
	      }
	    __syncthreads();
      }
    }
    if (col_Id < k_nn){
	    int col_write = blockId_J * k_nn + col_Id; 
	    int row_write = blockId_I * m_i + row_Id;
	    //int ind_write = leaf_id_g * M_I * k_nn + row_write * k_nn + col_write; 
	    int ind_write = row_write * k_nn + col_write; 
      //printf("leaf_id = %d , row_write = %d , col_write = %d , ind_write = %d \n", leaf_id_g, row_write, col_write , ind_write);
	    K[ind_write] = kvals[ind_s];
	    //K_ID[ind_write] = id_k[ind_s];
	    //K_ID[ind_write] = si[ind_s];
    }
    */ 
    
}


//__global__ void find_neighbor(float* knn, int* knn_Id, float* K, int* K_Id, int k, int M_I){
__global__ void find_neighbor(float* knn, int* knn_Id, float* K, int* G_Id, int k, int M_I, int m_j, int leaf_batch_g, int M){

    int col_Id = threadIdx.x; 
    int row_Id = blockIdx.x;

    if (row_Id >= M || col_Id >= M) return;
 
    __shared__ float Dist[2048];
    __shared__ int Dist_Id[2048];

    int size = blockDim.x;
    int leaf_id_g = leaf_batch_g * gridDim.y + blockIdx.y;
    
    //int ind_K = row_Id * M_I + col_Id; 
    int ind_K = blockIdx.z * M_I * M_I + row_Id * M_I + col_Id; 
    int i = col_Id;
    //if (leaf_id_g == 2047) printf("row = %d , col = %d 0 , ind_shared = %d \n", row_write, col_write, ind_shared);
    //Dist[ind_shared] = (col_Id < k) ? knn[ind_knn] : (col_Id < size) ? K[ind_read] : 1e30;
    Dist[col_Id] = K[ind_K];
    //Dist_Id[ind_shared] = (col_Id < k) ? knn_Id[ind_knn] : (col_Id < size) ? K_Id[ind_read] : 0;
    //Dist_Id[ind_shared] = (col_Id < k) ? 0 : (col_Id < size) ? K_Id[ind_read] : 0;
    //Dist_Id[col_Id] = col_Id;
    Dist_Id[col_Id] = G_Id[leaf_id_g * M_I + col_Id];
    int ind_shared = col_Id;
    //if (leaf_id_g == 2047) printf("row = %d , col = %d 1 , ind_shared = %d , \n", row_write, col_write, ind_shared);
     
    __syncthreads();

    // bitonic sort
    float tmp_f;
    int tmp_i;
    for (int g = 2; g <= size; g *= 2){
      for (int l = g/2; l>0; l /= 2){
      int ixj = i ^ l;
      int ixj_tmp = ixj;
      if (ixj > i){
        if ((i & g) == 0){
          if (Dist[ind_shared] > Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        } else {
          if (Dist[ind_shared] < Dist[ixj_tmp]){
               tmp_f = Dist[ixj_tmp];
               Dist[ixj_tmp] = Dist[ind_shared];
               Dist[ind_shared] = tmp_f;
               tmp_i = Dist_Id[ixj_tmp];
               Dist_Id[ixj_tmp] = Dist_Id[ind_shared];
               Dist_Id[ind_shared] = tmp_i;
                }
        }
        }
      
      __syncthreads();
      //if (leaf_id_g == 0 && row_write == 0) printf("i = %d , ixj = %d , Dist[%d] = %.2f \n", i , ixj, i, Dist[i]);
      }
    }

    /*
    if (col_Id < k) {
      K[ind_K] = Dist[col_Id];
      K_Id[ind_K] = Dist_Id[col_Id];
    }
    */

    size = 2*k;


    int ind_knn = leaf_id_g * M_I * k + row_Id * k + col_Id;

    if (col_Id >= k && col_Id < size) Dist[col_Id] = 1e30;
    //if (col_Id >= k && col_Id < size) Dist[col_Id] = knn[ind_knn];
    if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = 0;
    //if (col_Id >= k && col_Id < size) Dist_Id[col_Id] = knn_Id[ind_knn];

  __syncthreads();
	for (int g = 2; g <= size; g *= 2){
		for (int l = g/2; l>0; l /= 2){
		int ixj = i ^ l;
		int ixj_tmp =  ixj;
		if (ixj > i){
			if ((i & g) == 0){
				if (Dist[col_Id] > Dist[ixj_tmp]){
						 tmp_f = Dist[ixj_tmp];
						 Dist[ixj_tmp] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
						 tmp_i = Dist_Id[ixj_tmp];
						 Dist_Id[ixj_tmp] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			} else {
				if (Dist[col_Id] < Dist[ixj_tmp]){
						 tmp_f = Dist[ixj_tmp];
						 Dist[ixj_tmp] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
						 tmp_i = Dist_Id[ixj_tmp];
						 Dist_Id[ixj_tmp] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			}
			}
		
		__syncthreads();
		}
    
    if (col_Id < k){
      knn[ind_knn] = Dist[col_Id];
      knn_Id[ind_knn] = Dist_Id[col_Id];
    }
    
}

}



__global__ void merge(float* knn, int* knn_Id, float* K, int* K_Id, int k, int M_I, int m_j, int leaf_id_g){

  int col_Id = threadIdx.x; 
  int row_Id = blockIdx.x;
  //int leaf_id_g = threadIdx.z +blockIdx.z * blockDim.z;

  __shared__ float Dist[2048];
  __shared__ int Dist_Id[2048];

  int size = 2*k;


  int ind_knn = leaf_id_g * M_I * k + row_Id * k + col_Id;
  int ind_K = ind_knn - k;

  //Dist[col_Id] = (col_Id < k) ? knn[ind_knn] : K[ind_K]; 
  //Dist_Id[col_Id] = (col_Id < k) ? knn_Id[ind_knn] : K_Id[ind_K]; 

  Dist[col_Id] = (col_Id < k) ? 1e30 : K[ind_K]; 
  Dist_Id[col_Id] = (col_Id < k) ? 0 : K_Id[ind_K]; 


  // merge with knn
  int i = col_Id;
  float tmp_f;
  int tmp_i;  
	for (int g = 2; g <= size; g *= 2){
		for (int l = g/2; l>0; l /= 2){
		int ixj = i ^ l;
		int ixj_tmp =  ixj;
		if (ixj > i){
			if ((i & g) == 0){
				if (Dist[col_Id] > Dist[ixj_tmp]){
						 tmp_f = Dist[ixj_tmp];
						 Dist[ixj_tmp] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
						 tmp_i = Dist_Id[ixj_tmp];
						 Dist_Id[ixj_tmp] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			} else {
				if (Dist[col_Id] < Dist[ixj_tmp]){
						 tmp_f = Dist[ixj_tmp];
						 Dist[ixj_tmp] = Dist[col_Id];
						 Dist[col_Id] = tmp_f;
						 tmp_i = Dist_Id[ixj_tmp];
						 Dist_Id[ixj_tmp] = Dist_Id[col_Id];
						 Dist_Id[col_Id] = tmp_i;
							}
			}
			}
		
		__syncthreads();
		}
    }
    
    if (col_Id < k){
      knn[ind_knn] = Dist[col_Id];
      knn_Id[ind_knn] = Dist_Id[col_Id];
    }
     
}

void gen_sparse(int M, int tot_nnz, int d, int *R, int *C, float *V) {
 
    int nnz_row;
    int val;

    for (int i=0; i < M; i++){
      nnz_row = R[i+1] - R[i];
      for (int j=0; j < nnz_row; j++){
          int ind = R[i]+j; 
          val = rand()%d;
          //val = rand()%d;
          C[ind] = val;
          V[ind] = rand()%100;
        }    
      std::sort(C+R[i], C+(R[i+1]));
    }
}

void gen_R(int M, int nnzperrow, int *R, int *G_Id, int d) {  
  R[0] = 0;
  int tot_nnz = 0;
  int val;
  for (int m =1; m <= M; m++){ 
   val = 1 + rand()%(2*nnzperrow);
   //val = nnzperrow; //+ rand()%nnzperrow;
   if (val > d) val = 1; 
   tot_nnz += val;
   R[m] = tot_nnz;
   G_Id[m-1] = m-1;
  } 
  std::random_shuffle(&G_Id[0], &G_Id[M]);
  /*
  for (int m = 0; m < M; m++){ 
  printf("G_Id[%d] = %d \n", m , G_Id[m]);
  } 
  */
}

void gpu_knn(int *R, int *C, float *V, int *G_Id, int M, int leaves, int k, float *knn, int *knn_Id, int max_nnz){
 
	int pointsperleaf = M/leaves;
	int m_i = 8192 / max_nnz;
	int m_j = 8192 / max_nnz;
  m_i = min(m_i, pointsperleaf);
  m_i = min(m_i , 1024);
  //int m_j = 1024 / m_i;
  //int m_j = 1024 / m_i;
  m_j = min(m_j, pointsperleaf);
  if (m_i*m_j > 1024){
    m_j = 1024/m_i;
  }
 
  


  if (m_i*max_nnz > 8192) printf("Exceeds the shared memory size \n"); 
	int size_batch_I = (pointsperleaf + m_i - 1)/m_i;
	int size_batch_J = (pointsperleaf + m_j - 1) / m_j;
  int size_batch_leaves = (pow(2, 33)) / (4 * pointsperleaf * pointsperleaf ); 
  int num_batch_leaves = (leaves) / size_batch_leaves; 
   


	int M_I = M/leaves;

  printf("m_i = %d , m_j = %d , size_batch_I = %d , size_batch_J = %d , M_I = %d \n", m_i , m_j , size_batch_I , size_batch_J, M_I);

  float del_t1;
  hipEvent_t t0; 
  hipEvent_t t1;
  
  
  dim3 dimBlock(m_j, m_i, 1);	
  dim3 dimGrid(size_batch_J, size_batch_I, size_batch_leaves); 
  dim3 dimBlock_n(M_I, 1);
  dim3 dimGrid_n(M_I, size_batch_leaves);

  float *d_K;
  checkCudaErrors(hipMalloc((void **) &d_K, sizeof(float) * pointsperleaf * pointsperleaf * size_batch_leaves));
  //checkCudaErrors(hipMalloc((void **) &d_K_Id, sizeof(int)* pointsperleaf * k));
  //checkCudaErrors(hipMalloc((void **) &d_K, sizeof(int)*pointsperleaf*pointsperleaf));


  checkCudaErrors(hipEventCreate(&t0));
  checkCudaErrors(hipEventCreate(&t1));

  checkCudaErrors(hipEventRecord(t0, 0));
  //for (int leaf_id_g = 0; leaf_id_g < leaves; leaf_id_g++){
  checkCudaErrors(hipProfilerStart());
  for (int leaf_id_g = 0; leaf_id_g < num_batch_leaves; leaf_id_g++){
    compute_dist <<< dimGrid, dimBlock >>>(R, C, V, G_Id, d_K, m_i, m_j, k, M_I, leaf_id_g, max_nnz, M);
    checkCudaErrors(hipDeviceSynchronize());
    find_neighbor <<< dimGrid_n, dimBlock_n >>>(knn, knn_Id, d_K, G_Id, k, M_I, m_j, leaf_id_g, M);
    //checkCudaErrors(hipDeviceSynchronize());
    //merge <<< dimGrid_merge, dimBlock_merge >>>(knn, knn_Id, d_K, d_K_Id, k, M_I, m_j, leaf_id_g);
  } 
  checkCudaErrors(hipProfilerStop());
  
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(t1, 0));
  checkCudaErrors(hipEventSynchronize(t1));
  checkCudaErrors(hipEventElapsedTime(&del_t1, t0, t1));
  printf("\n Elapsed time (s) : %.4f \n ", del_t1/1000);
  printf(" # points = %d" , M);
  checkCudaErrors(hipFree(d_K));
  checkCudaErrors(hipEventDestroy(t0));
  checkCudaErrors(hipEventDestroy(t1));

}





int main(int argc, char **argv)
{

  //, del_t2, del_t3;

    checkCudaErrors(hipSetDevice(0));

    int d, nnzperrow;
    float *h_V, *d_V;
    int *h_C, *d_C;
    int *h_R, *d_R;
    int *h_G_Id, *d_G_Id;
    int M = 1024*2048;
    int leaves = 2048;
    d = 10000;
    int k = 32;
    nnzperrow = 32;
    int max_nnz = 2*nnzperrow;
    
    

    int *d_knn_Id;
    float *d_knn;

    h_R = (int *)malloc(sizeof(int)*(M+1));
    h_G_Id = (int *)malloc(sizeof(int)*(M));

    // generate random data 
    gen_R(M, nnzperrow, h_R,h_G_Id, d);
    int tot_nnz = h_R[M];
		h_V = (float *)malloc(sizeof(float)*tot_nnz);
    h_C = (int *)malloc(sizeof(int)*tot_nnz);
    gen_sparse(M, tot_nnz, d , h_R, h_C, h_V);   
    /*   
    for (int i = 0; i < M; i++){
        int nnz = h_R[i+1] - h_R[i];
        for (int j = 0; j < nnz; j++)
        printf("R[%d] = %d , C[%d] = %d , V[%d] = %.4f \n", i ,h_R[i], h_R[i]+j, h_C[h_R[i] + j], h_R[i]+j, h_V[h_R[i]+j]);
    }    
    */
    checkCudaErrors(hipMalloc((void **) &d_R, sizeof(int)*(M+1)));
    checkCudaErrors(hipMalloc((void **) &d_G_Id, sizeof(int)*(M)));
    checkCudaErrors(hipMalloc((void **) &d_C, sizeof(int)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_V, sizeof(float)*tot_nnz));
    checkCudaErrors(hipMalloc((void **) &d_knn_Id, sizeof(int)*M*k));
    checkCudaErrors(hipMalloc((void **) &d_knn, sizeof(float)*M*k));
 
    checkCudaErrors(hipMemcpy(d_C, h_C, sizeof(int)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_V, h_V, sizeof(float)*tot_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_R, h_R, sizeof(int)*(M+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_G_Id, h_G_Id, sizeof(int)*(M), hipMemcpyHostToDevice));

    printf("Random csr is generated  \n");

    gpu_knn(d_R, d_C, d_V, d_G_Id, M, leaves, k, d_knn, d_knn_Id, max_nnz);
    
    printf("\n\n");
    checkCudaErrors(hipFree(d_R));
    checkCudaErrors(hipFree(d_G_Id));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_V));
    free(h_R);
    free(h_C);
    free(h_V);
    free(h_G_Id);


}
