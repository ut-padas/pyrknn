#include "timer_gpu.hpp"

#include <sys/time.h>
#include <sstream>
#include <iostream>

//#include 

// Return time in seconds since the Unix epoch
static double timer() {
  double time;
  struct timeval tv;
  gettimeofday(&tv, NULL);
  time = (double)tv.tv_sec + (double)tv.tv_usec/1.e6;
  return time;
}

void TimerGPU::start() {
  hipDeviceSynchronize();
  tStart = timer();
}

void TimerGPU::stop() {
  hipDeviceSynchronize();
  tStop = timer();
}

double TimerGPU::elapsed_time() {
  return tStop-tStart;
}

void TimerGPU::show_elapsed_time() {
  std::cout << "Elapsed time : " << tStop-tStart << ".\n";
}

void TimerGPU::show_elapsed_time(const char* msg) {
  std::cout << msg << " : "
	    << tStop-tStart << " seconds." << std::endl;
}

std::string TimerGPU::get_elapsed_time(const char* msg) {
  std::stringstream ss;
  ss << msg << " : " << tStop-tStart << " seconds." << std::endl;
  return ss.str();
}
