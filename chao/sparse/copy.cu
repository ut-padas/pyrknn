#include "util.hpp"

#include <cassert>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        assert(false);                                                         \
    }                                                                          \
}

void copy_spmat_d2h(int m, int n, int nnz, int *dRowPtr, int *dColIdx, float *dVal, 
    int *hRowPtr, int *hColIdx, float *hVal) {
  CHECK_CUDA( hipMemcpy(hRowPtr, dRowPtr, (m+1)*sizeof(int), hipMemcpyDeviceToHost) )
  CHECK_CUDA( hipMemcpy(hColIdx, dColIdx, nnz*sizeof(int),   hipMemcpyDeviceToHost) )
  CHECK_CUDA( hipMemcpy(hVal,    dVal,    nnz*sizeof(float), hipMemcpyDeviceToHost) )
}

void copy(int m, int *dvec, int *hvec) {
  CHECK_CUDA( hipMemcpy(hvec, dvec, m*sizeof(int), hipMemcpyDeviceToHost) )
}

void copy(int m, float *dvec, float *hvec) {
  CHECK_CUDA( hipMemcpy(hvec, dvec, m*sizeof(float), hipMemcpyDeviceToHost) )
}

