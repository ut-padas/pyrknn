#include "util/timer.hpp"
#include "sort/sort_gpu.hpp"
#include "knn_gpu.hpp"

#include <vector>
#include <algorithm> 

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>

#include "hipblas.h"
#include <hip/hip_runtime.h>

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static const char *cudaGetErrorEnum(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "<unknown>";
}

#define cublasCheck(ans) { cublasAssert((ans), __FILE__, __LINE__); }
inline void cublasAssert(hipblasStatus_t code, const char *file, int line, bool abort=true) {
   if (code != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr,"CUBLAS assert: %s %s %d\n", cudaGetErrorEnum(code), file, line);
      if (abort) exit(code);
   }
}

template <typename T>
using dvec = thrust::device_vector<T>;

struct square: public thrust::unary_function<float,float>
{
  __host__ __device__
  float operator()(float x) const
  {
    return x*x;
  }
};

void compute_row_norms(const float *R, const float *Q, 
    dvec<float> &R2, dvec<float> &Q2, int nRow, int nCol) {

  dvec<int> Rrow(nRow), Qrow(nRow); // temporary storage for row indices
  auto zero = thrust::make_counting_iterator<int>(0);
  auto countItr = thrust::make_transform_iterator(zero, rowIdx(nCol));
 
  typedef thrust::device_ptr<const float> dptr;
  dptr dR(R), dQ(Q);
  thrust::transform_iterator<square, dptr> R_iter(dR, square());
  thrust::transform_iterator<square, dptr> Q_iter(dQ, square());
  thrust::reduce_by_key(countItr, countItr+nRow*nCol, R_iter, Rrow.begin(), R2.begin());
  thrust::reduce_by_key(countItr, countItr+nRow*nCol, Q_iter, Qrow.begin(), Q2.begin());
}


void compute_distance(const float *R, const float *Q, 
    const dvec<float> &R2, const dvec<float> &Q2, const dvec<float> &ones,
    dvec<float> &Dist, int nLeaf, int N, int d, int r, int m, hipblasHandle_t &handle) {
  
  float *ptrDist = thrust::raw_pointer_cast(Dist.data());

  // compute the distance (transpose) (cublas assumes column-major ordering)
  const float alpha = -2;
  const float beta = 0;
  const float one = 1;
  cublasCheck( hipblasSgemmStridedBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, m, d, &alpha,
          R, d, N*d, Q+r*m*d, d, N*d, &beta, ptrDist, N, m*N, nLeaf) );

  
  // rank-1 updates
  const int oneInt = 1;
  const float *ptrR2 = thrust::raw_pointer_cast(R2.data()),
      *ptrQ2 = thrust::raw_pointer_cast(Q2.data()),
      *ptrOne = thrust::raw_pointer_cast(ones.data());

  cublasCheck( hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, m, oneInt, &one, 
        ptrR2, N, N,
        ptrOne, m, m,
        &one, ptrDist, N, m*N, nLeaf) );
  
  cublasCheck( hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, m, oneInt, &one, 
        ptrOne, N, N,
        ptrQ2+r*m, m, N,
        &one, ptrDist, N, m*N, nLeaf) );
}


struct firstKCols : public thrust::unary_function<int, int> {
  int k, N;

  __host__ __device__
    firstKCols(int k_, int N_): k(k_), N(N_)  {}

  __host__ __device__
    int operator()(int i) {
      return i/k*N+i%k;
    }
};


struct strideBlock : public thrust::unary_function<int, int> {
  int mk, Nk, r;

  __host__ __device__
    strideBlock(int mk_, int Nk_, int r_): mk(mk_), Nk(Nk_), r(r_) {}

  __host__ __device__
    int operator()(int i) {
      return i/mk*Nk + r*mk + i%mk;
    }
};


void get_kcols(const dvec<float> &D, float *Dk, int nLeaf, int m, int N, int k, int r) {
  auto zero  = thrust::make_counting_iterator<int>(0);
  auto iterD = thrust::make_transform_iterator(zero, firstKCols(k, N));
  auto permD = thrust::make_permutation_iterator(D.begin(), iterD);
  auto iterK = thrust::make_transform_iterator(zero, strideBlock(m*k, N*k, r));
  auto permK = thrust::make_permutation_iterator(thrust::device_ptr<float>(Dk), iterK);
  thrust::copy(permD, permD+nLeaf*m*k, permK);
}


struct firstKVals : public thrust::unary_function<int, int> {
  int k, N, m;
  const int* vals;

  __host__ __device__
    firstKVals(int k_, int N_, int m_, const int *val_): 
      k(k_), N(N_), m(m_), vals(val_)  {}

  __host__ __device__
    int operator()(int i) {
      return vals[i/k*N+i%k]%N + i/(m*k)*N;
    }
};


void get_kcols(const dvec<int> &idx, const int *ID,
    int *IDk, int nLeaf, int m, int N, int k, int r) {
  const int* vals  = thrust::raw_pointer_cast(idx.data());
  auto zero  = thrust::make_counting_iterator<int>(0);
  auto iterD = thrust::make_transform_iterator(zero, firstKVals(k, N, m, vals));
  auto permD = thrust::make_permutation_iterator(thrust::device_ptr<const int>(ID), iterD);
  auto iterK = thrust::make_transform_iterator(zero, strideBlock(m*k, N*k, r));
  auto permK = thrust::make_permutation_iterator(thrust::device_ptr<int>(IDk), iterK);
  thrust::copy(permD, permD+nLeaf*m*k, permK);
}


void find_knn(dvec<float> &Dist, const int *ID, 
    float *nborDist, int *nborID, int nLeaf, int m, int N, int k, int r) {
  
  dvec<int> idx(m*nLeaf*N); // no need to initialize for mgpu
  sortGPU::sort_matrix_rows_mgpu(Dist, idx, m*nLeaf, N);
  get_kcols(Dist, nborDist, nLeaf, m, N, k, r);
  get_kcols(idx, ID, nborID, nLeaf, m, N, k, r);
}


void knn_gpu(const float *R, const float *Q, const int *ID, float *nborDist, int *nborID,
    int nLeaf, int N, int d, int k, int m, hipblasHandle_t &handle,
    float &t_dist, float &t_sort, float &t_kernel) {

  const dvec<float> ones(N*nLeaf, 1.0);

  Timer t, t1;
  cudaCheck( hipDeviceSynchronize() ); t1.start();


  // compute row norms  
  dvec<float> R2(N*nLeaf), Q2(N*nLeaf);
  cudaCheck( hipDeviceSynchronize() ); t.start();
  compute_row_norms(R, Q, R2, Q2, N*nLeaf, d);
  cudaCheck( hipDeviceSynchronize() );
  t.stop(); t_dist += t.elapsed_time();


  // blocking
  assert(N%m==0); // m is block size
  int M = N/m; // number of blocks
  dvec<float> Dist(m*N*nLeaf); // block/partial results 

  for (int r=0; r<M; r++) {

    cudaCheck( hipDeviceSynchronize() ); t.start();
    compute_distance(R, Q, R2, Q2, ones, Dist, nLeaf, N, d, r, m, handle);
    cudaCheck( hipDeviceSynchronize() );
    t.stop(); t_dist += t.elapsed_time();


    cudaCheck( hipDeviceSynchronize() ); t.start();
    find_knn(Dist, ID, nborDist, nborID, nLeaf, m, N, k, r);
    cudaCheck( hipDeviceSynchronize() );
    t.stop(); t_sort += t.elapsed_time();
  }
  
  cudaCheck( hipDeviceSynchronize() );
  t1.stop(); t_kernel += t1.elapsed_time();
}


void gemm_kselect_opt(int nLeaf, float *ptrR[], float *ptrQ[], int *ptrID[], int N, int d, 
		     float *ptrNborDist[], int *ptrNborID[], int k, int m,
         float &t_dist, float &t_sort, float &t_kernel) {

  // copy data to device
  dvec<float> R(N*d*nLeaf), Q(N*d*nLeaf);
  dvec<int>   ID(N*nLeaf); // ID of reference points
  for (int i=0; i<nLeaf; i++) {
    thrust::copy(ptrR[i], ptrR[i]+N*d, R.begin()+i*N*d);
    thrust::copy(ptrQ[i], ptrQ[i]+N*d, Q.begin()+i*N*d);
    thrust::copy(ptrID[i], ptrID[i]+N, ID.begin()+i*N);
  }
  
  // output
  dvec<float> nborDist(N*k*nLeaf);
  dvec<int>   nborID(N*k*nLeaf);

  // initialize for CUBLAS and MGPU
  hipblasHandle_t handle;
  cublasCheck( hipblasCreate(&handle) );  
  sortGPU::init_mgpu();
 

  // run kernel
  knn_gpu(
      thrust::raw_pointer_cast(R.data()),
      thrust::raw_pointer_cast(Q.data()),
      thrust::raw_pointer_cast(ID.data()),
      thrust::raw_pointer_cast(nborDist.data()),
      thrust::raw_pointer_cast(nborID.data()),
      nLeaf, N, d, k, m, handle,
      t_dist, t_sort, t_kernel);


  // copy results back to host
  for (int i=0; i<nLeaf; i++) {
    thrust::copy(nborDist.begin()+i*N*k, nborDist.begin()+(i+1)*N*k, ptrNborDist[i]);
    thrust::copy(nborID.begin()+i*N*k, nborID.begin()+(i+1)*N*k, ptrNborID[i]);
  }

  // clean up resouce
  hipblasDestroy(handle);
  sortGPU::final_mgpu();
}

